/*
Copyright (C) 2009 Scott Grauer-Gray, Chandra Kambhamettu, and Kannappan Palaniappan

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307 USA
*/

//Defines the functions to store the resulting disparity map

#include "saveResultingDisparityMapHeader.cuh"

//save the output disparity map using the scale defined in scaleDisparityInOutput at each pixel to the file at disparityMapSaveImagePath
//also takes in the timer to time the implementation including the transfer time from the device to the host
void saveResultingDisparityMap(const char* disparityMapSaveImagePath,
		float*& disparityMapFromImage1To2Device, float scaleDisparityInOutput,
		unsigned int widthImages, unsigned int heightImages,
		std::chrono::time_point<std::chrono::system_clock>& timeWithTransferStart,
		double& totalTimeIncludeTransfer) {
	//allocate the space on the host for and x and y movement between images
	float* disparityMapFromImage1To2Host = new float[widthImages * heightImages];

	//transfer the disparity map estimation on the device to the host for output
	(hipMemcpy(disparityMapFromImage1To2Host, disparityMapFromImage1To2Device, widthImages*heightImages*sizeof(float),
						  hipMemcpyDeviceToHost) );

	auto timeWithTransferEnd = std::chrono::system_clock::now();

	//printf("Running time including transfer time: %.10lf seconds\n", timeEnd-timeStart);
	std::chrono::duration<double> diff = timeWithTransferEnd-timeWithTransferStart;
	totalTimeIncludeTransfer += diff.count();
	//stop the timer and print the total time of the BP implementation including the device-host transfer time
	//printf("Time to retrieve movement on host (including transfer): %f (ms) \n", totalTimeIncludeTransfer);

	//save the resulting disparity map images to a file
	saveDisparityImageToPGM(disparityMapSaveImagePath, scaleDisparityInOutput, disparityMapFromImage1To2Host, widthImages, heightImages);

	delete [] disparityMapFromImage1To2Host;
}


#include "hip/hip_runtime.h"
/*
Copyright (C) 2009 Scott Grauer-Gray, Chandra Kambhamettu, and Kannappan Palaniappan

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307 USA
*/

//Defines the functions to run the CUDA implementation of 2-D Stereo estimation using BP

#include "runBpStereoHostHeader.cuh"
#include <chrono>

#define RUN_DETAILED_TIMING

double timeCopyDataKernelTotalTime = 0.0;
double timeBpItersKernelTotalTime = 0.0;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//functions directed related to running BP to retrieve the movement between the images

//set the current BP settings in the host in constant memory on the device
__host__ void setBPSettingInConstMem(BPsettings& currentBPSettings)
{
	//write BP settings to constant memory on the GPU
	(hipMemcpyToSymbol(HIP_SYMBOL(BPSettingsConstMemStereo), &currentBPSettings, sizeof(BPsettings)));
}

//run the given number of iterations of BP at the current level using the given message values in global device memory
__host__ void runBPAtCurrentLevel(int& numIterationsAtLevel, int& widthLevelActualIntegerSize, int& heightLevelActualIntegerSize, size_t& dataTexOffset,
	float*& messageUDeviceCheckerboard1, float*& messageDDeviceCheckerboard1, float*& messageLDeviceCheckerboard1, 
	float*& messageRDeviceCheckerboard1, float*& messageUDeviceCheckerboard2, float*& messageDDeviceCheckerboard2, float*& messageLDeviceCheckerboard2, 
	float*& messageRDeviceCheckerboard2, dim3& grid, dim3& threads, int& numBytesDataAndMessageSetInCheckerboardAtLevel, float* dataCostDeviceCheckerboard1,
	float* dataCostDeviceCheckerboard2)

{
	//at each level, run BP for numIterations, alternating between updating the messages between the two "checkerboards"
	for (int iterationNum = 0; iterationNum < numIterationsAtLevel; iterationNum++)
	{
		if ((iterationNum % 2) == 0)
		{
			(hipDeviceSynchronize());

#ifdef RUN_DETAILED_TIMING
			auto timeBpItersKernelStart = std::chrono::system_clock::now();
#endif

			runBPIterationUsingCheckerboardUpdatesNoTextures<<<grid, threads>>>(
					dataCostDeviceCheckerboard1, dataCostDeviceCheckerboard2,
					messageUDeviceCheckerboard1, messageDDeviceCheckerboard1,
					messageLDeviceCheckerboard1, messageRDeviceCheckerboard1,
					messageUDeviceCheckerboard2, messageDDeviceCheckerboard2,
					messageLDeviceCheckerboard2, messageRDeviceCheckerboard2,
					widthLevelActualIntegerSize, heightLevelActualIntegerSize,
					CHECKERBOARD_PART_2, ((int) dataTexOffset / sizeof(float)));

			(hipDeviceSynchronize());

#ifdef RUN_DETAILED_TIMING

			auto timeBpItersKernelEnd = std::chrono::system_clock::now();
			std::chrono::duration<double> diff = timeBpItersKernelEnd-timeBpItersKernelStart;

			timeBpItersKernelTotalTime += diff.count();

#endif
		}
		else
		{
			(hipDeviceSynchronize());

#ifdef RUN_DETAILED_TIMING

			auto timeBpItersKernelStart = std::chrono::system_clock::now();

#endif

			runBPIterationUsingCheckerboardUpdatesNoTextures<<<grid, threads>>>(
					dataCostDeviceCheckerboard1, dataCostDeviceCheckerboard2,
					messageUDeviceCheckerboard1, messageDDeviceCheckerboard1,
					messageLDeviceCheckerboard1, messageRDeviceCheckerboard1,
					messageUDeviceCheckerboard2, messageDDeviceCheckerboard2,
					messageLDeviceCheckerboard2, messageRDeviceCheckerboard2,
					widthLevelActualIntegerSize, heightLevelActualIntegerSize,
					CHECKERBOARD_PART_1, ((int) dataTexOffset / sizeof(float)));

			(hipDeviceSynchronize());

#ifdef RUN_DETAILED_TIMING

			auto timeBpItersKernelEnd = std::chrono::system_clock::now();
			std::chrono::duration<double> diff = timeBpItersKernelEnd-timeBpItersKernelStart;

			timeBpItersKernelTotalTime += diff.count();

#endif
		}
	}
}



//copy the computed BP message values from the current now-completed level to the corresponding slots in the next level "down" in the computation
//pyramid; the next level down is double the width and height of the current level so each message in the current level is copied into four "slots"
//in the next level down
//need two different "sets" of message values to avoid read-write conflicts
__host__ void copyMessageValuesToNextLevelDown(int& widthLevelActualIntegerSizePrevLevel, int& heightLevelActualIntegerSizePrevLevel,
	int& widthLevelActualIntegerSizeNextLevel, int& heightLevelActualIntegerSizeNextLevel,
	float*& messageUDeviceCheckerboard1CopyFrom, float*& messageDDeviceCheckerboard1CopyFrom, float*& messageLDeviceCheckerboard1CopyFrom, 
	float*& messageRDeviceCheckerboard1CopyFrom, float*& messageUDeviceCheckerboard2CopyFrom, float*& messageDDeviceCheckerboard2CopyFrom, 
	float*& messageLDeviceCheckerboard2CopyFrom, float*& messageRDeviceCheckerboard2CopyFrom, float*& messageUDeviceCheckerboard1CopyTo, 
	float*& messageDDeviceCheckerboard1CopyTo, float*& messageLDeviceCheckerboard1CopyTo, float*& messageRDeviceCheckerboard1CopyTo, 
	float*& messageUDeviceCheckerboard2CopyTo, float*& messageDDeviceCheckerboard2CopyTo, float*& messageLDeviceCheckerboard2CopyTo, 
	float*& messageRDeviceCheckerboard2CopyTo, int& numBytesDataAndMessageSetInCheckerboardAtLevel, dim3& grid, dim3& threads)
{

#if !defined(USE_SAME_ARRAY_FOR_ALL_LEVEL_MESSAGE_VALS) && !defined(USE_SAME_ARRAY_FOR_ALL_ALLOC)

	//allocate space in the GPU for the message values in the checkerboard set to copy to
	(hipMalloc((void**) &messageUDeviceCheckerboard1CopyTo, numBytesDataAndMessageSetInCheckerboardAtLevel));
	(hipMalloc((void**) &messageDDeviceCheckerboard1CopyTo, numBytesDataAndMessageSetInCheckerboardAtLevel));
	(hipMalloc((void**) &messageLDeviceCheckerboard1CopyTo, numBytesDataAndMessageSetInCheckerboardAtLevel));
	(hipMalloc((void**) &messageRDeviceCheckerboard1CopyTo, numBytesDataAndMessageSetInCheckerboardAtLevel));

	(hipMalloc((void**) &messageUDeviceCheckerboard2CopyTo, numBytesDataAndMessageSetInCheckerboardAtLevel));
	(hipMalloc((void**) &messageDDeviceCheckerboard2CopyTo, numBytesDataAndMessageSetInCheckerboardAtLevel));
	(hipMalloc((void**) &messageLDeviceCheckerboard2CopyTo, numBytesDataAndMessageSetInCheckerboardAtLevel));
	(hipMalloc((void**) &messageRDeviceCheckerboard2CopyTo, numBytesDataAndMessageSetInCheckerboardAtLevel));

#endif

	( hipDeviceSynchronize() );

#ifdef RUN_DETAILED_TIMING

	auto timeCopyDataKernelStart = std::chrono::system_clock::now();

#endif

	//call the kernal to copy the computed BP message data to the next level down in parallel in each of the two "checkerboards"
	//storing the current message values
	copyPrevLevelToNextLevelBPCheckerboardStereoNoTextures <<< grid, threads >>> (messageUDeviceCheckerboard1CopyFrom, messageDDeviceCheckerboard1CopyFrom,
			messageLDeviceCheckerboard1CopyFrom, messageRDeviceCheckerboard1CopyFrom, messageUDeviceCheckerboard2CopyFrom,
			messageDDeviceCheckerboard2CopyFrom, messageLDeviceCheckerboard2CopyFrom, messageRDeviceCheckerboard2CopyFrom,
			messageUDeviceCheckerboard1CopyTo, messageDDeviceCheckerboard1CopyTo, messageLDeviceCheckerboard1CopyTo,
			messageRDeviceCheckerboard1CopyTo, messageUDeviceCheckerboard2CopyTo, messageDDeviceCheckerboard2CopyTo, messageLDeviceCheckerboard2CopyTo,
			messageRDeviceCheckerboard2CopyTo, (widthLevelActualIntegerSizePrevLevel), (heightLevelActualIntegerSizePrevLevel),
			widthLevelActualIntegerSizeNextLevel, heightLevelActualIntegerSizeNextLevel, CHECKERBOARD_PART_1);

	( hipDeviceSynchronize() );

	copyPrevLevelToNextLevelBPCheckerboardStereoNoTextures <<< grid, threads >>> (messageUDeviceCheckerboard1CopyFrom, messageDDeviceCheckerboard1CopyFrom,
			messageLDeviceCheckerboard1CopyFrom, messageRDeviceCheckerboard1CopyFrom, messageUDeviceCheckerboard2CopyFrom,
			messageDDeviceCheckerboard2CopyFrom, messageLDeviceCheckerboard2CopyFrom, messageRDeviceCheckerboard2CopyFrom,
			messageUDeviceCheckerboard1CopyTo, messageDDeviceCheckerboard1CopyTo, messageLDeviceCheckerboard1CopyTo,
			messageRDeviceCheckerboard1CopyTo, messageUDeviceCheckerboard2CopyTo, messageDDeviceCheckerboard2CopyTo, messageLDeviceCheckerboard2CopyTo,
			messageRDeviceCheckerboard2CopyTo, (widthLevelActualIntegerSizePrevLevel), (heightLevelActualIntegerSizePrevLevel),
			widthLevelActualIntegerSizeNextLevel, heightLevelActualIntegerSizeNextLevel, CHECKERBOARD_PART_2);

	( hipDeviceSynchronize() );

#ifdef RUN_DETAILED_TIMING

	auto timeCopyDataKernelEnd = std::chrono::system_clock::now();
	std::chrono::duration<double> diff = timeCopyDataKernelEnd-timeCopyDataKernelStart;

	timeCopyDataKernelTotalTime += diff.count();

#endif

#if !defined(USE_SAME_ARRAY_FOR_ALL_LEVEL_MESSAGE_VALS) && !defined(USE_SAME_ARRAY_FOR_ALL_ALLOC)

	//free the now-copied from computed data of the completed level
	hipFree(messageUDeviceCheckerboard1CopyFrom);
	hipFree(messageDDeviceCheckerboard1CopyFrom);
	hipFree(messageLDeviceCheckerboard1CopyFrom);
	hipFree(messageRDeviceCheckerboard1CopyFrom);

	hipFree(messageUDeviceCheckerboard2CopyFrom);
	hipFree(messageDDeviceCheckerboard2CopyFrom);
	hipFree(messageLDeviceCheckerboard2CopyFrom);
	hipFree(messageRDeviceCheckerboard2CopyFrom);

#endif
}

//initialize the data cost at each pixel with no estimated Stereo values...only the data and discontinuity costs are used
__host__ void initializeDataCosts(float*& image1PixelsDevice, float*& image2PixelsDevice, float*& dataCostDeviceCheckerboard1, float*& dataCostDeviceCheckerboard2, BPsettings& algSettings)
{
	//allocate array and copy image data
	//data is in the single-float value format
	hipChannelFormatDesc channelDescImages = hipCreateChannelDesc<float>();

	//store the two image pixels in the GPU in a CUDA array
	hipArray* cu_arrayImage1BP;
	hipArray* cu_arrayImage2BP;

	//allocate and then copy the image pixel data for the two images on the GPU
	( hipMallocArray( &cu_arrayImage1BP, &channelDescImages, algSettings.widthImages, algSettings.heightImages )); 
	( hipMallocArray( &cu_arrayImage2BP, &channelDescImages, algSettings.widthImages, algSettings.heightImages )); 

	( hipMemcpyToArray( cu_arrayImage1BP, 0, 0, image1PixelsDevice, algSettings.widthImages*algSettings.heightImages*sizeof(float), hipMemcpyDeviceToDevice));
	( hipMemcpyToArray( cu_arrayImage2BP, 0, 0, image2PixelsDevice, algSettings.widthImages*algSettings.heightImages*sizeof(float), hipMemcpyDeviceToDevice));

	// set texture parameters for the CUDA arrays to hold the input images
	image1PixelsTextureBPStereo.addressMode[0] = hipAddressModeClamp;
	image1PixelsTextureBPStereo.addressMode[1] = hipAddressModeClamp;
	image1PixelsTextureBPStereo.filterMode = hipFilterModePoint;
	image1PixelsTextureBPStereo.normalized = false;    // access with normalized texture coordinates

	image2PixelsTextureBPStereo.addressMode[0] = hipAddressModeClamp;
	image2PixelsTextureBPStereo.addressMode[1] = hipAddressModeClamp;
	image2PixelsTextureBPStereo.filterMode = hipFilterModePoint;
	image2PixelsTextureBPStereo.normalized = false;    // access with normalized texture coordinates

	//Bind the CUDA Arrays holding the input image pixel arrays to the appropriate texture
	( hipBindTextureToArray( image1PixelsTextureBPStereo, cu_arrayImage1BP, channelDescImages));
	( hipBindTextureToArray( image2PixelsTextureBPStereo, cu_arrayImage2BP, channelDescImages));

	//setup execution parameters
	//the thread size remains constant throughout but the grid size is adjusted based on the current level/kernal to run
	dim3 threads(BLOCK_SIZE_WIDTH_BP, BLOCK_SIZE_HEIGHT_BP);
	dim3 grid;


	//kernal run on full-sized image to retrieve data costs at the "bottom" level of the pyramid
	grid.x = (unsigned int)ceil((float)algSettings.widthImages / (float)threads.x);
	grid.y = (unsigned int)ceil((float)algSettings.heightImages / (float)threads.y);

	//initialize the data the the "bottom" of the image pyramid
	initializeBottomLevelDataStereo <<< grid, threads >>> (dataCostDeviceCheckerboard1, dataCostDeviceCheckerboard2);

	( hipDeviceSynchronize() );

	//unbind the texture attached to the image pixel values
	hipUnbindTexture( image1PixelsTextureBPStereo);
	hipUnbindTexture( image2PixelsTextureBPStereo);

	//image data no longer needed after data costs are computed
	(hipFreeArray(cu_arrayImage1BP));
	(hipFreeArray(cu_arrayImage2BP));
}



//initialize the message values with no previous message values...all message values are set to 0
__host__ void initializeMessageValsToDefault(float*& messageUDeviceSet0Checkerboard1, float*& messageDDeviceSet0Checkerboard1, float*& messageLDeviceSet0Checkerboard1, float*& messageRDeviceSet0Checkerboard1,
												  float*& messageUDeviceSet0Checkerboard2, float*& messageDDeviceSet0Checkerboard2, float*& messageLDeviceSet0Checkerboard2, float*& messageRDeviceSet0Checkerboard2,
												  int widthOfCheckerboard, int heightOfCheckerboard, int numPossibleMovements)
{
	dim3 threads(BLOCK_SIZE_WIDTH_BP, BLOCK_SIZE_HEIGHT_BP);
	dim3 grid((unsigned int)ceil((float)widthOfCheckerboard / (float)threads.x), (unsigned int)ceil((float)heightOfCheckerboard / (float)threads.y));

	//initialize all the message values for each pixel at each possible movement to the default value in the kernal
	initializeMessageValsToDefault <<< grid, threads >>> (messageUDeviceSet0Checkerboard1, messageDDeviceSet0Checkerboard1, messageLDeviceSet0Checkerboard1, 
												messageRDeviceSet0Checkerboard1, messageUDeviceSet0Checkerboard2, messageDDeviceSet0Checkerboard2, 
												messageLDeviceSet0Checkerboard2, messageRDeviceSet0Checkerboard2, widthOfCheckerboard, heightOfCheckerboard);

	hipDeviceSynchronize();
}




//run the belief propagation algorithm with on a set of stereo images to generate a disparity map
//the input images image1PixelsDevice and image2PixelsDevice are stored in the global memory of the GPU
//the output movements resultingDisparityMapDevice is stored in the global memory of the GPU
__host__ void runBeliefPropStereoCUDA(float*& image1PixelsDevice, float*& image2PixelsDevice, float*& resultingDisparityMapDevice, BPsettings& algSettings, DetailedTimings& timings)
{	
#ifdef RUN_DETAILED_TIMING

	timeCopyDataKernelTotalTime = 0.0;
	timeBpItersKernelTotalTime = 0.0;
#endif

	//retrieve the total number of possible movements; this is equal to the number of disparity values 
	int totalPossibleMovements = NUM_POSSIBLE_DISPARITY_VALUES;

#ifdef RUN_DETAILED_TIMING

	auto timeInitSettingsConstMemStart = std::chrono::system_clock::now();

#endif

	//set the BP algorithm and extension settings on the device
	setBPSettingInConstMem(algSettings);

	( hipDeviceSynchronize() );

#ifdef RUN_DETAILED_TIMING

	auto timeInitSettingsConstMemEnd = std::chrono::system_clock::now();

	std::chrono::duration<double> diff = timeInitSettingsConstMemEnd-timeInitSettingsConstMemStart;
	double totalTimeInitSettingsConstMem = diff.count();

#endif

	//setup execution parameters
	//the thread size remains constant throughout but the grid size is adjusted based on the current level/kernal to run
	dim3 threads(BLOCK_SIZE_WIDTH_BP, BLOCK_SIZE_HEIGHT_BP);
	dim3 grid;

	//start at the "bottom level" and word way up to determine amount of space needed to store data costs
	float widthLevel = (float)algSettings.widthImages;
	float heightLevel = (float)algSettings.heightImages;

	//store the "actual" integer size of the width and height of the level since it's not actually
	//possible to work with level with a decimal sizes...the portion of the last row/column is truncated
	//if the width/level size has a decimal
	int widthLevelActualIntegerSize = (int)roundf(widthLevel);
	int heightLevelActualIntegerSize = (int)roundf(heightLevel);

	int halfTotalDataAllLevels = 0;

	//compute "half" the total number of pixels in including every level of the "pyramid"
	//using "half" because the data is split in two using the checkerboard scheme
	for (int levelNum = 0; levelNum < algSettings.numLevels; levelNum++)
	{
		halfTotalDataAllLevels += (int)(ceil(widthLevelActualIntegerSize/2.0f))*(heightLevelActualIntegerSize);
		widthLevel /= 2.0f;
		heightLevel /= 2.0f;

		widthLevelActualIntegerSize = (int)ceil(widthLevel);
		heightLevelActualIntegerSize = (int)ceil(heightLevel);
	}

	//declare and then allocate the space on the device to store the data cost component at each possible movement at each level of the "pyramid"
	//each checkboard holds half of the data
	float* dataCostDeviceCheckerboard1; //checkerboard 1 includes the pixel in slot (0, 0)
	float* dataCostDeviceCheckerboard2;

	float* messageUDeviceCheckerboard1;
	float* messageDDeviceCheckerboard1;
	float* messageLDeviceCheckerboard1;
	float* messageRDeviceCheckerboard1;

	float* messageUDeviceCheckerboard2;
	float* messageDDeviceCheckerboard2;
	float* messageLDeviceCheckerboard2;
	float* messageRDeviceCheckerboard2;

#ifdef RUN_DETAILED_TIMING

	auto timeInitSettingsMallocStart = std::chrono::system_clock::now();

#endif

#ifdef USE_SAME_ARRAY_FOR_ALL_ALLOC

	printf("ALLOC ALL MEMORY\n");
	(hipMalloc((void**) &dataCostDeviceCheckerboard1, 10*(halfTotalDataAllLevels)*totalPossibleMovements*sizeof(float)));
	dataCostDeviceCheckerboard2 = &(dataCostDeviceCheckerboard1[1*(halfTotalDataAllLevels)*totalPossibleMovements]);

	messageUDeviceCheckerboard1 = &(dataCostDeviceCheckerboard1[2*(halfTotalDataAllLevels)*totalPossibleMovements]);
	messageDDeviceCheckerboard1 = &(dataCostDeviceCheckerboard1[3*(halfTotalDataAllLevels)*totalPossibleMovements]);
	messageLDeviceCheckerboard1 = &(dataCostDeviceCheckerboard1[4*(halfTotalDataAllLevels)*totalPossibleMovements]);
	messageRDeviceCheckerboard1 = &(dataCostDeviceCheckerboard1[5*(halfTotalDataAllLevels)*totalPossibleMovements]);

	messageUDeviceCheckerboard2 = &(dataCostDeviceCheckerboard1[6*(halfTotalDataAllLevels)*totalPossibleMovements]);
	messageDDeviceCheckerboard2 = &(dataCostDeviceCheckerboard1[7*(halfTotalDataAllLevels)*totalPossibleMovements]);
	messageLDeviceCheckerboard2 = &(dataCostDeviceCheckerboard1[8*(halfTotalDataAllLevels)*totalPossibleMovements]);
	messageRDeviceCheckerboard2 = &(dataCostDeviceCheckerboard1[9*(halfTotalDataAllLevels)*totalPossibleMovements]);

#else

	(hipMalloc((void**) &dataCostDeviceCheckerboard1, (halfTotalDataAllLevels)*totalPossibleMovements*sizeof(float)));
	(hipMalloc((void**) &dataCostDeviceCheckerboard2, (halfTotalDataAllLevels)*totalPossibleMovements*sizeof(float)));

#ifdef USE_SAME_ARRAY_FOR_ALL_LEVEL_MESSAGE_VALS

	(hipMalloc((void**) &messageUDeviceCheckerboard1, (halfTotalDataAllLevels)*totalPossibleMovements*sizeof(float)));
	(hipMalloc((void**) &messageDDeviceCheckerboard1, (halfTotalDataAllLevels)*totalPossibleMovements*sizeof(float)));
	(hipMalloc((void**) &messageLDeviceCheckerboard1, (halfTotalDataAllLevels)*totalPossibleMovements*sizeof(float)));
	(hipMalloc((void**) &messageRDeviceCheckerboard1, (halfTotalDataAllLevels)*totalPossibleMovements*sizeof(float)));

	(hipMalloc((void**) &messageUDeviceCheckerboard2, (halfTotalDataAllLevels)*totalPossibleMovements*sizeof(float)));
	(hipMalloc((void**) &messageDDeviceCheckerboard2, (halfTotalDataAllLevels)*totalPossibleMovements*sizeof(float)));
	(hipMalloc((void**) &messageLDeviceCheckerboard2, (halfTotalDataAllLevels)*totalPossibleMovements*sizeof(float)));
	(hipMalloc((void**) &messageRDeviceCheckerboard2, (halfTotalDataAllLevels)*totalPossibleMovements*sizeof(float)));

#endif

#endif

	( hipDeviceSynchronize() );

#ifdef RUN_DETAILED_TIMING

	auto timeInitSettingsMallocEnd = std::chrono::system_clock::now();

	diff = timeInitSettingsMallocEnd-timeInitSettingsMallocStart;
	double totalTimeInitSettingsMallocStart = diff.count();

	auto timeInitDataCostsStart = std::chrono::system_clock::now();

#endif

	//now go "back to" the bottom level to initialize the data costs starting at the bottom level and going up the pyramid
	widthLevel = (float)algSettings.widthImages;
	heightLevel = (float)algSettings.heightImages;

	widthLevelActualIntegerSize = (int)roundf(widthLevel);
	heightLevelActualIntegerSize = (int)roundf(heightLevel);

	//initialize the data cost at the bottom level 
	initializeDataCosts(image1PixelsDevice, image2PixelsDevice, dataCostDeviceCheckerboard1, dataCostDeviceCheckerboard2, algSettings);

#ifdef RUN_DETAILED_TIMING

	auto timeInitDataCostsEnd = std::chrono::system_clock::now();
	diff = timeInitDataCostsEnd-timeInitDataCostsStart;

	double totalTimeGetDataCostsBottomLevel = diff.count();

#endif

	int offsetLevel = 0;

	//stores the number of bytes for the data costs and one set of message values in each of the two "checkerboards" at the current level
	//this is half the total number of bytes for the data/message info at the level, since there are two equal-sized checkerboards
	//initially at "bottom level" of width widthImages and height heightImages
	int numBytesDataAndMessageSetInCheckerboardAtLevel = (ceil(widthLevelActualIntegerSize/2.0f))*(heightLevelActualIntegerSize)*totalPossibleMovements*sizeof(float);

#ifdef RUN_DETAILED_TIMING

	auto timeInitDataCostsHigherLevelsStart = std::chrono::system_clock::now();

#endif

	//set the data costs at each level from the bottom level "up"
	for (int levelNum = 1; levelNum < algSettings.numLevels; levelNum++)
	{
		int prev_level_offset_level = offsetLevel;

		//width is half since each part of the checkboard contains half the values going across
		//retrieve offset where the data starts at the "current level"
		offsetLevel += ((int)(ceil(widthLevelActualIntegerSize/2.0f))) *(heightLevelActualIntegerSize)*totalPossibleMovements;

		widthLevel /= 2.0f;
		heightLevel /= 2.0f;

		int prevWidthLevelActualIntegerSize = widthLevelActualIntegerSize;
		int prevHeightLevelActualIntegerSize = heightLevelActualIntegerSize;

		widthLevelActualIntegerSize = (int)ceil(widthLevel);
		heightLevelActualIntegerSize = (int)ceil(heightLevel);
		int widthCheckerboard = (int)ceil(((float)widthLevelActualIntegerSize) / 2.0f);

		//printf("LevelNum: %d  Width: %d  Height: %d \n", levelNum, widthLevelActualIntegerSize, heightLevelActualIntegerSize);

		//each pixel "checkerboard" is half the width of the level and there are two of them; each "pixel/point" at the level belongs to one checkerboard and
		//the four-connected neighbors are in the other checkerboard
		grid.x = (unsigned int)ceil(((float)widthCheckerboard) / (float)threads.x);
		grid.y = (unsigned int)ceil((float)heightLevel / (float)threads.y);

		size_t offsetNum = 0;

		initializeCurrentLevelDataStereoNoTextures <<< grid, threads >>> (&dataCostDeviceCheckerboard1[prev_level_offset_level], &dataCostDeviceCheckerboard2[prev_level_offset_level], &dataCostDeviceCheckerboard1[offsetLevel], widthLevelActualIntegerSize, heightLevelActualIntegerSize, prevWidthLevelActualIntegerSize, prevHeightLevelActualIntegerSize, CHECKERBOARD_PART_1, ((int)offsetNum/sizeof(float)));

		( hipDeviceSynchronize() );

		initializeCurrentLevelDataStereoNoTextures <<< grid, threads >>> (&dataCostDeviceCheckerboard1[prev_level_offset_level], &dataCostDeviceCheckerboard2[prev_level_offset_level], &dataCostDeviceCheckerboard2[offsetLevel], widthLevelActualIntegerSize, heightLevelActualIntegerSize, prevWidthLevelActualIntegerSize, prevHeightLevelActualIntegerSize, CHECKERBOARD_PART_2, ((int)offsetNum/sizeof(float)));

		( hipDeviceSynchronize() );

		//update number of bytes of data and message cost if not at bottom level
		if (levelNum < (algSettings.numLevels - 1))
		{
			//each "checkerboard" where the computation alternates contains half the data
			numBytesDataAndMessageSetInCheckerboardAtLevel = (ceil(widthLevelActualIntegerSize/2.0f))*(heightLevelActualIntegerSize)*totalPossibleMovements*sizeof(float);
		}
	}

#ifdef RUN_DETAILED_TIMING

	auto timeInitDataCostsHigherLevelsEnd = std::chrono::system_clock::now();
	diff = timeInitDataCostsHigherLevelsEnd-timeInitDataCostsHigherLevelsStart;

	double totalTimeGetDataCostsHigherLevels = diff.count();

#endif

	( hipDeviceSynchronize() );

	//declare the space to pass the BP messages
	//need to have two "sets" of checkerboards because
	//the message values at the "higher" level in the image
	//pyramid need copied to a lower level without overwriting
	//values
	float* dataCostDeviceCurrentLevelCheckerboard1;
	float* dataCostDeviceCurrentLevelCheckerboard2;
	float* messageUDeviceSet0Checkerboard1;
	float* messageDDeviceSet0Checkerboard1;
	float* messageLDeviceSet0Checkerboard1;
	float* messageRDeviceSet0Checkerboard1;

	float* messageUDeviceSet0Checkerboard2;
	float* messageDDeviceSet0Checkerboard2;
	float* messageLDeviceSet0Checkerboard2;
	float* messageRDeviceSet0Checkerboard2;

	float* messageUDeviceSet1Checkerboard1;
	float* messageDDeviceSet1Checkerboard1;
	float* messageLDeviceSet1Checkerboard1;
	float* messageRDeviceSet1Checkerboard1;

	float* messageUDeviceSet1Checkerboard2;
	float* messageDDeviceSet1Checkerboard2;
	float* messageLDeviceSet1Checkerboard2;
	float* messageRDeviceSet1Checkerboard2;

#ifdef RUN_DETAILED_TIMING

	auto timeInitMessageValuesStart = std::chrono::system_clock::now();

#endif

	dataCostDeviceCurrentLevelCheckerboard1 = &dataCostDeviceCheckerboard1[offsetLevel];
	dataCostDeviceCurrentLevelCheckerboard2 = &dataCostDeviceCheckerboard2[offsetLevel];

#if defined(USE_SAME_ARRAY_FOR_ALL_LEVEL_MESSAGE_VALS) || defined(USE_SAME_ARRAY_FOR_ALL_ALLOC)

	messageUDeviceSet0Checkerboard1 = &messageUDeviceCheckerboard1[offsetLevel];
	messageDDeviceSet0Checkerboard1 = &messageDDeviceCheckerboard1[offsetLevel];
	messageLDeviceSet0Checkerboard1 = &messageLDeviceCheckerboard1[offsetLevel];
	messageRDeviceSet0Checkerboard1 = &messageRDeviceCheckerboard1[offsetLevel];

	messageUDeviceSet0Checkerboard2 = &messageUDeviceCheckerboard2[offsetLevel];
	messageDDeviceSet0Checkerboard2 = &messageDDeviceCheckerboard2[offsetLevel];
	messageLDeviceSet0Checkerboard2 = &messageLDeviceCheckerboard2[offsetLevel];
	messageRDeviceSet0Checkerboard2 = &messageRDeviceCheckerboard2[offsetLevel];

#else

	//allocate the space for the message values in the first checkboard set at the current level
	(hipMalloc((void**) &messageUDeviceSet0Checkerboard1, numBytesDataAndMessageSetInCheckerboardAtLevel));
	(hipMalloc((void**) &messageDDeviceSet0Checkerboard1, numBytesDataAndMessageSetInCheckerboardAtLevel));
	(hipMalloc((void**) &messageLDeviceSet0Checkerboard1, numBytesDataAndMessageSetInCheckerboardAtLevel));
	(hipMalloc((void**) &messageRDeviceSet0Checkerboard1, numBytesDataAndMessageSetInCheckerboardAtLevel));

	(hipMalloc((void**) &messageUDeviceSet0Checkerboard2, numBytesDataAndMessageSetInCheckerboardAtLevel));
	(hipMalloc((void**) &messageDDeviceSet0Checkerboard2, numBytesDataAndMessageSetInCheckerboardAtLevel));
	(hipMalloc((void**) &messageLDeviceSet0Checkerboard2, numBytesDataAndMessageSetInCheckerboardAtLevel));
	(hipMalloc((void**) &messageRDeviceSet0Checkerboard2, numBytesDataAndMessageSetInCheckerboardAtLevel));

#endif

	auto timeInitMessageValuesKernelTimeStart = std::chrono::system_clock::now();

	//retrieve the number of bytes needed to store the data cost/each set of messages in the checkerboard
	numBytesDataAndMessageSetInCheckerboardAtLevel = (ceil(widthLevelActualIntegerSize/2.0f))*(heightLevelActualIntegerSize)*totalPossibleMovements*sizeof(float);

	//initialize all the BP message values at every pixel for every disparity to 0
	initializeMessageValsToDefault(messageUDeviceSet0Checkerboard1, messageDDeviceSet0Checkerboard1, messageLDeviceSet0Checkerboard1, messageRDeviceSet0Checkerboard1,
											messageUDeviceSet0Checkerboard2, messageDDeviceSet0Checkerboard2, messageLDeviceSet0Checkerboard2, messageRDeviceSet0Checkerboard2,
											(int)(ceil(widthLevelActualIntegerSize/2.0f)), heightLevelActualIntegerSize, totalPossibleMovements);

	auto timeInitMessageValuesKernelTimeEnd = std::chrono::system_clock::now();
	diff = timeInitMessageValuesKernelTimeEnd-timeInitMessageValuesKernelTimeStart;

	double totalTimeInitMessageValuesKernelTime = diff.count();

	( hipDeviceSynchronize() );

#ifdef RUN_DETAILED_TIMING

	auto timeInitMessageValuesEnd = std::chrono::system_clock::now();
	diff = timeInitMessageValuesEnd-timeInitMessageValuesStart;

	double totalTimeInitMessageVals = diff.count();

#endif

	//alternate between checkerboard sets 0 and 1
	int currentCheckerboardSet = 0;

#ifdef RUN_DETAILED_TIMING

	double totalTimeBpIters = 0.0;
	double totalTimeCopyData = 0.0;

#endif


	//run BP at each level in the "pyramid" starting on top and continuing to the bottom
	//where the final movement values are computed...the message values are passed from
	//the upper level to the lower levels; this pyramid methods causes the BP message values
	//to converge more quickly
	for (int levelNum = algSettings.numLevels - 1; levelNum >= 0; levelNum--)
	{
		gpuErrchk( hipPeekAtLastError() );

		//offset needed because of alignment requirement for textures
		size_t offset = 0;
		( hipDeviceSynchronize() );

#ifdef RUN_DETAILED_TIMING

		auto timeBpIterStart = std::chrono::system_clock::now();

#endif

		//printf("LevelNumBP: %d  Width: %f  Height: %f \n", levelNum, widthLevel, heightLevel);
		int widthCheckerboard = (int)ceil(((float)widthLevelActualIntegerSize) / 2.0f);
		grid.x = (unsigned int) ceil(
				(float) (widthCheckerboard) / (float) threads.x); //only updating half at a time
		grid.y = (unsigned int) ceil((float) heightLevel / (float) threads.y);

		//need to alternate which checkerboard set to work on since copying from one to the other...need to avoid read-write conflict when copying in parallel
		if (currentCheckerboardSet == 0)
		{
			runBPAtCurrentLevel(algSettings.numIterations,
					widthLevelActualIntegerSize, heightLevelActualIntegerSize,
					offset, messageUDeviceSet0Checkerboard1,
					messageDDeviceSet0Checkerboard1,
					messageLDeviceSet0Checkerboard1,
					messageRDeviceSet0Checkerboard1,
					messageUDeviceSet0Checkerboard2,
					messageDDeviceSet0Checkerboard2,
					messageLDeviceSet0Checkerboard2,
					messageRDeviceSet0Checkerboard2, grid, threads,
					numBytesDataAndMessageSetInCheckerboardAtLevel,
					dataCostDeviceCurrentLevelCheckerboard1,
					dataCostDeviceCurrentLevelCheckerboard2);
		}
		else
		{
			runBPAtCurrentLevel(algSettings.numIterations,
					widthLevelActualIntegerSize, heightLevelActualIntegerSize,
					offset, messageUDeviceSet1Checkerboard1,
					messageDDeviceSet1Checkerboard1,
					messageLDeviceSet1Checkerboard1,
					messageRDeviceSet1Checkerboard1,
					messageUDeviceSet1Checkerboard2,
					messageDDeviceSet1Checkerboard2,
					messageLDeviceSet1Checkerboard2,
					messageRDeviceSet1Checkerboard2, grid, threads,
					numBytesDataAndMessageSetInCheckerboardAtLevel,
					dataCostDeviceCurrentLevelCheckerboard1,
					dataCostDeviceCurrentLevelCheckerboard2);
		}

		(hipDeviceSynchronize());

#ifdef RUN_DETAILED_TIMING

		auto timeBpIterEnd = std::chrono::system_clock::now();
		diff = timeBpIterEnd-timeBpIterStart;

		totalTimeBpIters += diff.count();
		
		auto timeCopyMessageValuesStart = std::chrono::system_clock::now();

#endif

		//if not at the "bottom level" copy the current message values at the current level to the corresponding slots next level 
		if (levelNum > 0)
		{	
			int prevWidthLevelActualIntegerSize = widthLevelActualIntegerSize;
			int prevHeightLevelActualIntegerSize = heightLevelActualIntegerSize;

			//the "next level" down has double the width and height of the current level
			widthLevel *= 2.0f;
			heightLevel *= 2.0f;

			widthLevelActualIntegerSize = (int)ceil(widthLevel);
			heightLevelActualIntegerSize = (int)ceil(heightLevel);
			int widthCheckerboard = (int)ceil(((float)widthLevelActualIntegerSize) / 2.0f);

			offsetLevel -= ((int)ceil(widthLevelActualIntegerSize/2.0f))*(heightLevelActualIntegerSize)*totalPossibleMovements;
			printf("OffsetLevel: %d\n", offsetLevel);

			//update the number of bytes needed to store each set
			numBytesDataAndMessageSetInCheckerboardAtLevel = ((int)(ceil(widthLevelActualIntegerSize/2.0f)))*(heightLevelActualIntegerSize)*totalPossibleMovements*sizeof(float);

			grid.x = (unsigned int)ceil((float)(widthCheckerboard / 2.0f) / (float)threads.x);
			grid.y = (unsigned int)ceil((float)(heightLevel / 2.0f) / (float)threads.y);

			dataCostDeviceCurrentLevelCheckerboard1 = &dataCostDeviceCheckerboard1[offsetLevel];
			dataCostDeviceCurrentLevelCheckerboard2 = &dataCostDeviceCheckerboard2[offsetLevel];

			//bind messages in the current checkerboard set to the texture to copy to the "other" checkerboard set at the next level 
			if (currentCheckerboardSet == 0)
			{

#if defined(USE_SAME_ARRAY_FOR_ALL_LEVEL_MESSAGE_VALS) || defined(USE_SAME_ARRAY_FOR_ALL_ALLOC)

				messageUDeviceSet1Checkerboard1 = &messageUDeviceCheckerboard1[offsetLevel];
				messageDDeviceSet1Checkerboard1 = &messageDDeviceCheckerboard1[offsetLevel];
				messageLDeviceSet1Checkerboard1 = &messageLDeviceCheckerboard1[offsetLevel];
				messageRDeviceSet1Checkerboard1 = &messageRDeviceCheckerboard1[offsetLevel];

				messageUDeviceSet1Checkerboard2 = &messageUDeviceCheckerboard2[offsetLevel];
				messageDDeviceSet1Checkerboard2 = &messageDDeviceCheckerboard2[offsetLevel];
				messageLDeviceSet1Checkerboard2 = &messageLDeviceCheckerboard2[offsetLevel];
				messageRDeviceSet1Checkerboard2 = &messageRDeviceCheckerboard2[offsetLevel];

#endif

				copyMessageValuesToNextLevelDown(
						prevWidthLevelActualIntegerSize,
						prevHeightLevelActualIntegerSize,
						widthLevelActualIntegerSize,
						heightLevelActualIntegerSize,
						messageUDeviceSet0Checkerboard1,
						messageDDeviceSet0Checkerboard1,
						messageLDeviceSet0Checkerboard1,
						messageRDeviceSet0Checkerboard1,
						messageUDeviceSet0Checkerboard2,
						messageDDeviceSet0Checkerboard2,
						messageLDeviceSet0Checkerboard2,
						messageRDeviceSet0Checkerboard2,
						messageUDeviceSet1Checkerboard1,
						messageDDeviceSet1Checkerboard1,
						messageLDeviceSet1Checkerboard1,
						messageRDeviceSet1Checkerboard1,
						messageUDeviceSet1Checkerboard2,
						messageDDeviceSet1Checkerboard2,
						messageLDeviceSet1Checkerboard2,
						messageRDeviceSet1Checkerboard2,
						numBytesDataAndMessageSetInCheckerboardAtLevel, grid,
						threads);

				currentCheckerboardSet = 1;
			}
			else
			{

#if defined(USE_SAME_ARRAY_FOR_ALL_LEVEL_MESSAGE_VALS) || defined(USE_SAME_ARRAY_FOR_ALL_ALLOC)

				messageUDeviceSet0Checkerboard1 = &messageUDeviceCheckerboard1[offsetLevel];
				messageDDeviceSet0Checkerboard1 = &messageDDeviceCheckerboard1[offsetLevel];
				messageLDeviceSet0Checkerboard1 = &messageLDeviceCheckerboard1[offsetLevel];
				messageRDeviceSet0Checkerboard1 = &messageRDeviceCheckerboard1[offsetLevel];

				messageUDeviceSet0Checkerboard2 = &messageUDeviceCheckerboard2[offsetLevel];
				messageDDeviceSet0Checkerboard2 = &messageDDeviceCheckerboard2[offsetLevel];
				messageLDeviceSet0Checkerboard2 = &messageLDeviceCheckerboard2[offsetLevel];
				messageRDeviceSet0Checkerboard2 = &messageRDeviceCheckerboard2[offsetLevel];

#endif

				copyMessageValuesToNextLevelDown(
						prevWidthLevelActualIntegerSize,
						prevHeightLevelActualIntegerSize,
						widthLevelActualIntegerSize,
						heightLevelActualIntegerSize,
						messageUDeviceSet1Checkerboard1,
						messageDDeviceSet1Checkerboard1,
						messageLDeviceSet1Checkerboard1,
						messageRDeviceSet1Checkerboard1,
						messageUDeviceSet1Checkerboard2,
						messageDDeviceSet1Checkerboard2,
						messageLDeviceSet1Checkerboard2,
						messageRDeviceSet1Checkerboard2,
						messageUDeviceSet0Checkerboard1,
						messageDDeviceSet0Checkerboard1,
						messageLDeviceSet0Checkerboard1,
						messageRDeviceSet0Checkerboard1,
						messageUDeviceSet0Checkerboard2,
						messageDDeviceSet0Checkerboard2,
						messageLDeviceSet0Checkerboard2,
						messageRDeviceSet0Checkerboard2,
						numBytesDataAndMessageSetInCheckerboardAtLevel, grid,
						threads);

				currentCheckerboardSet = 0;
			}
		}
		//otherwise in "bottom level"; use message values and data costs to retrieve final movement values
		( hipDeviceSynchronize() );

#ifdef RUN_DETAILED_TIMING

		auto timeCopyMessageValuesEnd = std::chrono::system_clock::now();
		diff = timeCopyMessageValuesEnd-timeCopyMessageValuesStart;

		totalTimeCopyData += diff.count();

#endif
	}
	gpuErrchk( hipPeekAtLastError() );

	//printf("Final  Width: %d  Height: %d \n", widthLevelActualIntegerSize, heightLevelActualIntegerSize);

#ifdef RUN_DETAILED_TIMING

	auto timeGetOutputDisparityStart = std::chrono::system_clock::now();
	gpuErrchk( hipPeekAtLastError() );

#endif

	grid.x = (unsigned int)ceil((float)widthLevel / (float)threads.x);
	grid.y = (unsigned int)ceil((float)heightLevel / (float)threads.y);

	if (currentCheckerboardSet == 0)
	{
		retrieveOutputDisparityCheckerboardStereoNoTextures <<< grid, threads >>> (dataCostDeviceCurrentLevelCheckerboard1, dataCostDeviceCurrentLevelCheckerboard2,
				messageUDeviceSet0Checkerboard1, messageDDeviceSet0Checkerboard1, messageLDeviceSet0Checkerboard1, messageRDeviceSet0Checkerboard1,
				messageUDeviceSet0Checkerboard2, messageDDeviceSet0Checkerboard2, messageLDeviceSet0Checkerboard2, messageRDeviceSet0Checkerboard2,
				resultingDisparityMapDevice, widthLevel, heightLevel);
	}
	else
	{
		retrieveOutputDisparityCheckerboardStereoNoTextures <<< grid, threads >>> (dataCostDeviceCurrentLevelCheckerboard1, dataCostDeviceCurrentLevelCheckerboard2,
				messageUDeviceSet1Checkerboard1, messageDDeviceSet1Checkerboard1, messageLDeviceSet1Checkerboard1, messageRDeviceSet1Checkerboard1,
				messageUDeviceSet1Checkerboard2, messageDDeviceSet1Checkerboard2, messageLDeviceSet1Checkerboard2, messageRDeviceSet1Checkerboard2,
				resultingDisparityMapDevice, widthLevel, heightLevel);
	}

	( hipDeviceSynchronize() );
	gpuErrchk( hipPeekAtLastError() );

#ifdef RUN_DETAILED_TIMING

	auto timeGetOutputDisparityEnd = std::chrono::system_clock::now();
	diff = timeGetOutputDisparityEnd-timeGetOutputDisparityStart;

	double totalTimeGetOutputDisparity = diff.count();

	auto timeFinalUnbindFreeStart = std::chrono::system_clock::now();
	double totalTimeFinalUnbind = 0.0;

#endif

#ifdef RUN_DETAILED_TIMING

	auto timeFinalFreeStart = std::chrono::system_clock::now();

#endif
	gpuErrchk( hipPeekAtLastError() );

#if defined(USE_SAME_ARRAY_FOR_ALL_LEVEL_MESSAGE_VALS) && !defined(USE_SAME_ARRAY_FOR_ALL_ALLOC)

	hipFree(messageUDeviceCheckerboard1);
	hipFree(messageDDeviceCheckerboard1);
	hipFree(messageLDeviceCheckerboard1);
	hipFree(messageRDeviceCheckerboard1);

	hipFree(messageUDeviceCheckerboard2);
	hipFree(messageDDeviceCheckerboard2);
	hipFree(messageLDeviceCheckerboard2);
	hipFree(messageRDeviceCheckerboard2);

#else

#ifndef USE_SAME_ARRAY_FOR_ALL_ALLOC

	//free the device storage for the message values used to retrieve the output movement values
	if (currentCheckerboardSet == 0)
	{
		//free device space allocated to message values
		hipFree(messageUDeviceSet0Checkerboard1);
		hipFree(messageDDeviceSet0Checkerboard1);
		hipFree(messageLDeviceSet0Checkerboard1);
		hipFree(messageRDeviceSet0Checkerboard1);

		hipFree(messageUDeviceSet0Checkerboard2);
		hipFree(messageDDeviceSet0Checkerboard2);
		hipFree(messageLDeviceSet0Checkerboard2);
		hipFree(messageRDeviceSet0Checkerboard2);
	}
	else
	{
		//free device space allocated to message values
		hipFree(messageUDeviceSet1Checkerboard1);
		hipFree(messageDDeviceSet1Checkerboard1);
		hipFree(messageLDeviceSet1Checkerboard1);
		hipFree(messageRDeviceSet1Checkerboard1);

		hipFree(messageUDeviceSet1Checkerboard2);
		hipFree(messageDDeviceSet1Checkerboard2);
		hipFree(messageLDeviceSet1Checkerboard2);
		hipFree(messageRDeviceSet1Checkerboard2);
	}

#endif
#endif
	gpuErrchk( hipPeekAtLastError() );

#ifdef USE_SAME_ARRAY_FOR_ALL_ALLOC
	printf("FREE ALL MEMORY\n");

	hipFree(dataCostDeviceCheckerboard1);

#else
	printf("ALLOC MULT MEM SEGMENTS\n");

	//now free the allocated data space
	hipFree(dataCostDeviceCheckerboard1);
	hipFree(dataCostDeviceCheckerboard2);

#endif
	gpuErrchk( hipPeekAtLastError() );
	( hipDeviceSynchronize() );

#ifdef RUN_DETAILED_TIMING

	auto timeFinalUnbindFreeEnd = std::chrono::system_clock::now();
	auto timeFinalFreeEnd = std::chrono::system_clock::now();

	diff = timeFinalUnbindFreeEnd-timeFinalUnbindFreeStart;
	double totalTimeFinalUnbindFree = diff.count();

	diff = timeFinalFreeEnd-timeFinalFreeStart;
	double totalTimeFinalFree = diff.count();

	double totalMemoryProcessingTime = totalTimeInitSettingsConstMem
			+ totalTimeInitSettingsMallocStart + totalTimeFinalUnbindFree
			+ (totalTimeInitMessageVals - totalTimeInitMessageValuesKernelTime)
			+ (totalTimeCopyData - timeCopyDataKernelTotalTime)
			+ (totalTimeBpIters - timeBpItersKernelTotalTime);
	double totalComputationProcessing = totalTimeGetDataCostsBottomLevel
			+ totalTimeGetDataCostsHigherLevels
			+ totalTimeInitMessageValuesKernelTime + timeCopyDataKernelTotalTime
			+ timeBpItersKernelTotalTime + totalTimeGetOutputDisparity;
	double totalTimed = totalTimeInitSettingsConstMem
			+ totalTimeInitSettingsMallocStart
			+ totalTimeGetDataCostsBottomLevel
			+ totalTimeGetDataCostsHigherLevels + totalTimeInitMessageVals
			+ totalTimeBpIters + totalTimeCopyData + totalTimeGetOutputDisparity
			+ totalTimeFinalUnbindFree;
	timings.totalTimeInitSettingsConstMem.push_back(
			totalTimeInitSettingsConstMem);
	timings.totalTimeInitSettingsMallocStart.push_back(
			totalTimeInitSettingsMallocStart);
	timings.totalTimeGetDataCostsBottomLevel.push_back(
			totalTimeGetDataCostsBottomLevel);
	timings.totalTimeGetDataCostsHigherLevels.push_back(
			totalTimeGetDataCostsHigherLevels);
	timings.totalTimeInitMessageVals.push_back(totalTimeInitMessageVals);
	timings.totalTimeInitMessageValuesKernelTime.push_back(totalTimeInitMessageValuesKernelTime);
	timings.totalTimeBpIters.push_back(totalTimeBpIters);
	timings.timeBpItersKernelTotalTime.push_back(timeBpItersKernelTotalTime);
	timings.totalTimeCopyData.push_back(totalTimeCopyData);
	timings.timeCopyDataKernelTotalTime.push_back(timeCopyDataKernelTotalTime);
	timings.totalTimeGetOutputDisparity.push_back(totalTimeGetOutputDisparity);
	timings.totalTimeFinalUnbindFree.push_back(totalTimeFinalUnbindFree);
	timings.totalTimeFinalUnbind.push_back(totalTimeFinalUnbind);
	timings.totalTimeFinalFree.push_back(totalTimeFinalFree);
	timings.totalTimed.push_back(totalTimed);
	timings.totalMemoryProcessingTime.push_back(totalMemoryProcessingTime);
	timings.totalComputationProcessing.push_back(totalComputationProcessing);
	timings.totNumTimings++;

#endif
}


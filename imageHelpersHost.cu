#include "hip/hip_runtime.h"
/*
Copyright (C) 2009 Scott Grauer-Gray, Chandra Kambhamettu, and Kannappan Palaniappan

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307 USA
*/

//Defines the functions used to load the input images and store the disparity map image for use in the CUDA BP implementation

#include "imageHelpersHostHeader.cuh"


//functions used to load input images/save resulting movment images

//function to retrieve the disparity values from a disparity map with a known scale factor
float* ImageHelperFunctions::retrieveDisparityValsFromStereoPGM(const char* filePathPgmImage, unsigned int widthImage, unsigned int heightImage, float scaleFactor)
{
	unsigned int* imageData = new unsigned int[widthImage*heightImage];

	float* disparityVals = new float[widthImage*heightImage];

	//go through every pixel and retrieve the Stereo value using the pixel value and the scale factor
	for (unsigned int pixelNum = 0; pixelNum < (widthImage*heightImage); pixelNum++)
	{
		disparityVals[pixelNum] = imageData[pixelNum] / scaleFactor;
	}

	delete [] imageData;

	return disparityVals;
}

unsigned int* ImageHelperFunctions::loadImageAsGrayScale(const char* filePathImage, unsigned int& widthImage, unsigned int& heightImage)
{
	char pgmExtension[] = "pgm";
	char ppmExtension[] = "ppm";
	char* filePathImageCopy = new char[strlen(filePathImage) + 1];
	strcpy(filePathImageCopy, filePathImage);

	//check if PGM or PPM image (types currently supported)
	char* token = strtok(filePathImageCopy, ".");
	char* lastToken = new char[strlen(token) + 1];;
	strcpy(lastToken, token);
	while( token != NULL )
	{
		delete [] lastToken;
		lastToken = new char[strlen(token) + 1];
		strcpy(lastToken, token);
	    token = strtok(NULL, ".");
	}

	//last token after "." is file extension
	if (strcmp(lastToken, pgmExtension) == 0)
	{
		delete [] filePathImageCopy;
		//printf("PGM IMAGE\n");
		return loadImageFromPGM(filePathImage, widthImage, heightImage);
	}
	else if (strcmp(lastToken, ppmExtension) == 0)
	{
		delete [] filePathImageCopy;
		//printf("PPM IMAGE\n");
		return loadImageFromPPM(filePathImage, widthImage, heightImage);
	}
	else
	{
		delete [] filePathImageCopy;
		printf("ERROR, IMAGE FILE %s NOT SUPPORTED\n", filePathImage);
		return NULL;
	}
}

//load the PGM image and return as an array of floats
unsigned int* ImageHelperFunctions::loadImageFromPGM(const char* filePathPgmImage, unsigned int& widthImage, unsigned int& heightImage)
{
	unsigned int* imageData;

	unsigned char *dataRead;

	pgmRead (filePathPgmImage, &widthImage, &heightImage,
	     dataRead);

	imageData = new unsigned int[widthImage*heightImage];

	for (int numPixel = 0; numPixel < (widthImage*heightImage); numPixel++)
	{
		imageData[numPixel] = (unsigned int)(dataRead[numPixel]);	
	}

	delete [] dataRead;
	return imageData;
}

//load the PPM image, convert to grayscale, and return as an array of floats
unsigned int* ImageHelperFunctions::loadImageFromPPM(const char* filePathPpmImage, unsigned int& widthImage, unsigned int& heightImage)
{
	unsigned int* imageData;
	unsigned char *dataRead;

	ppmReadReturnGrayScale(filePathPpmImage, &widthImage, &heightImage,
	     dataRead, USE_WEIGHTED_RGB_TO_GRAYSCALE_CONVERSION);

	imageData = new unsigned int[widthImage*heightImage];

	for (int numPixel = 0; numPixel < (widthImage*heightImage); numPixel++)
	{
		imageData[numPixel] = (unsigned int)(dataRead[numPixel]);
	}

	delete [] dataRead;
	return imageData;
}


/* INPUT: a filename (char*),row and column dimension variables (long), and
 *   a pointer to a 2D array of unsigned char's of size MAXROWS x MAXCOLS 
 *   (row major).
 * OUTPUT: an integer is returned indicating whether or not the
 *   file was read into memory (in row major order).  1 is returned if the 
 *   file is read correctly, 0 if it is not.  If there are 
 *   too few pixels, the function still returns 1, but returns an error 
 *   message.  Error messages are also returned if a file cannot be open, 
 *   or if the specifications of the file are invalid.
 * NOTE: The case where too many pixels are in a file is not detected.
 */
int ImageHelperFunctions::pgmRead (const char *fileName, unsigned int *cols, unsigned int *rows,
	     unsigned char*& image) {
      FILE *filePointer;    /* for file buffer */
      char line[MAXLENGTH]; /* for character input from file */
      int maximumValue = 0; /* max value from header */
      int binary;           /* flag to indicate if file is binary (P5)*/
      long numberRead = 0;  /* counter for number of pixels read */
      long i,j;             /* (i,j) for loops */
      int test,temp;        /* for detecting EOF(test) and temp storage */

      /* Open the file, return an error if necessary. */
      if ((filePointer = fopen(fileName,"r")) == NULL) {
	   printf ("ERROR: cannot open file\n\n");
	   fclose (filePointer);
	   return (0);
      }
    
      /* Initialize columnsidth, and height */
      *cols = *rows =0;

      /* Check the file signature ("Magic Numbers" P2 and P5); skip comments
       * and blank lines (CR with no spaces before it).*/
      fgets (line,MAXLENGTH,filePointer);
      while (line[0]=='#' || line[0]=='\n') fgets (line,MAXLENGTH,filePointer);
      if (line[0]=='P' && (line[1]=='2')) {
	   binary = 0;
	 /*   printf ("\nFile Format: P2\n"); */
      }
      else if (line[0]=='P' && (line[1]=='5')) {
	   binary = 1;
	  /*  printf ("\nFORMAT: P5\n"); */
      }
      else {
	   printf ("ERROR: incorrect file format\n\n");
	   fclose (filePointer);
	   return (0);
      }          

      /* Input the width, height and maximum value, skip comments and blank
       * lines. */
      fgets (line,MAXLENGTH,filePointer);
      while (line[0]=='#' || line[0]=='\n') fgets (line,MAXLENGTH,filePointer);
      sscanf (line,"%u %u",cols,rows);

      fgets (line,MAXLENGTH,filePointer);
      while (line[0]=='#' || line[0]=='\n') fgets(line,MAXLENGTH,filePointer);
      sscanf (line,"%d",&maximumValue);

      /* Check specifications and return an error if h,w, or
      *  maximum value is illegal.*/
      if ((*cols)<1 ||(*rows)<1 || maximumValue<0 || maximumValue>MAXVALUE){
	   printf ("ERROR: invalid file specifications (cols/rows/max value)\n\n");
	   fclose (filePointer);
	   return (0);
      }
      else if ((*cols) > MAXCOLS || (*rows) > MAXROWS) {
	   printf ("ERROR: increase MAXROWS/MAXCOLS in PGM.h");
	   fclose (filePointer);
	   return (0);
      } 

      image = new unsigned char[(*cols)*(*rows)];

      /* Read in the data for binary (P5) or ascii (P2) PGM formats   */
      if (binary) {
	   for (i = 0; i < (*rows); i++) {
	        numberRead += fread((void *)&(image[i*(*cols) + 0]),
		  sizeof(unsigned char), (*cols), filePointer); 
		if (feof(filePointer)) break;
	   }
      }
      else {
	   for (i= 0; i < (*rows); i++) {
	        for (j =0; j < (*cols); j++) { 
	             test = fscanf (filePointer,"%d",&temp);
		     if (test == EOF) break;
		     image[i*(*cols) + j] = (unsigned char)temp;

		     numberRead++;
		}
		if (test == EOF) break;
	   }
      } 
 
      /* Insure the number of pixels read is at least the
       *   number indicated by w*h.
       * If not, return an error message, but proceed */
      if (numberRead < ((*rows)*(*cols))) {
	   printf ("ERROR: fewer pixels than rows*cols indicates\n\n");
      }
     
      /* close the file and return 1 indicating success */
      fclose (filePointer);
      return (1);
}

/* INPUT: a filename (char*),row and column dimension variables (long), and
 *   a pointer to a 2D array of unsigned char's of size MAXROWS x MAXCOLS
 *   (row major).
 * OUTPUT: an integer is returned indicating whether or not the
 *   file was read into memory (in row major order).  1 is returned if the
 *   file is read correctly, 0 if it is not.  If there are
 *   too few pixels, the function still returns 1, but returns an error
 *   message.  Error messages are also returned if a file cannot be open,
 *   or if the specifications of the file are invalid.
 * NOTE: The case where too many pixels are in a file is not detected.
 */
int ImageHelperFunctions::ppmReadReturnGrayScale (const char *fileName, unsigned int *cols, unsigned int *rows,
	     unsigned char*& image, bool weightedRGBConversion) {
      FILE *filePointer;    /* for file buffer */
      char line[MAXLENGTH]; /* for character input from file */
      int maximumValue = 0; /* max value from header */
      int binary;           /* flag to indicate if file is binary (P5)*/
      long numberRead = 0;  /* counter for number of pixels read */
      long i,j;             /* (i,j) for loops */
      int test,temp;        /* for detecting EOF(test) and temp storage */

      /* Open the file, return an error if necessary. */
      if ((filePointer = fopen(fileName,"r")) == NULL) {
	   printf ("ERROR: cannot open file\n\n");
	   fclose (filePointer);
	   return (0);
      }

      /* Initialize columnsidth, and height */
      *cols = *rows =0;

      /* Check the file signature ("Magic Numbers" P2 and P5); skip comments
       * and blank lines (CR with no spaces before it).*/
      fgets (line,MAXLENGTH,filePointer);
      while (line[0]=='#' || line[0]=='\n') fgets (line,MAXLENGTH,filePointer);
      if (line[0]=='P' && (line[1]=='3')) {
	   binary = 0;
	 /*   printf ("\nFile Format: P2\n"); */
      }
      else if (line[0]=='P' && (line[1]=='6')) {
	   binary = 1;
	  /*  printf ("\nFORMAT: P5\n"); */
      }
      else {
	   printf ("ERROR: incorrect file format\n\n");
	   fclose (filePointer);
	   return (0);
      }

      /* Input the width, height and maximum value, skip comments and blank
       * lines. */
      fgets (line,MAXLENGTH,filePointer);
      while (line[0]=='#' || line[0]=='\n') fgets (line,MAXLENGTH,filePointer);
      sscanf (line,"%u %u",cols,rows);

      fgets (line,MAXLENGTH,filePointer);
      while (line[0]=='#' || line[0]=='\n') fgets(line,MAXLENGTH,filePointer);
      sscanf (line,"%d",&maximumValue);

      /* Check specifications and return an error if h,w, or
      *  maximum value is illegal.*/
      if ((*cols)<1 ||(*rows)<1 || maximumValue<0 || maximumValue>MAXVALUE){
	   printf ("ERROR: invalid file specifications (cols/rows/max value)\n\n");
	   fclose (filePointer);
	   return (0);
      }
      else if ((*cols) > MAXCOLS || (*rows) > MAXROWS) {
	   printf ("ERROR: increase MAXROWS/MAXCOLS in PGM.h");
	   fclose (filePointer);
	   return (0);
      }

      unsigned char* rgbImage = new unsigned char[3*(*cols)*(*rows)];
      image = new unsigned char[(*cols)*(*rows)];

      /* Read in the data for binary (P5) or ascii (P2) PGM formats   */
      if (binary) {
	   for (i = 0; i < (*rows); i++) {
	        numberRead += fread((void *)&(rgbImage[(3*i)*(*cols) + 0]),
		  sizeof(unsigned char), 3*(*cols), filePointer);
		if (feof(filePointer)) break;
	   }
      }
      else {
	   for (i= 0; i < (*rows); i++) {
	        for (j =0; j < (3*(*cols)); j++) {
	             test = fscanf (filePointer,"%d",&temp);
	             if (test == EOF) break;
	             rgbImage[i*(*cols) + j] = (unsigned char)temp;
	             numberRead++;
	        }
		if (test == EOF) break;
	   }
      }

      /* Insure the number of pixels read is at least the
       *   number indicated by w*h.
       * If not, return an error message, but proceed */
      if (numberRead < (3*(*rows)*(*cols))) {
	   printf ("ERROR: fewer pixels than rows*cols indicates\n\n");
      }

      //convert the RGB image to grayscale
      for (i = 0; i < (*rows)*(*cols); i++)
      {
    	  float rChannelWeight = 1.0f / 3.0f;
    	  float bChannelWeight = 1.0f / 3.0f;
    	  float gChannelWeight = 1.0f / 3.0f;
    	  if (weightedRGBConversion)
    	  {
    		  rChannelWeight = 0.299f;
    		  bChannelWeight = 0.587f;
    		  gChannelWeight = 0.114f;
    	  }
    	  image[i] = (unsigned char)floor(rChannelWeight*((float)rgbImage[i*3]) + gChannelWeight*((float)rgbImage[i*3 + 1]) + bChannelWeight*((float)rgbImage[i*3 + 2]) + 0.5f);
      }

      //free memory used for storing rgb image (since using grayscale image)
      delete [] rgbImage;

      /* close the file and return 1 indicating success */
      fclose (filePointer);
      return (1);
}

    
/* INPUT: a filename (char*), the dimensions of the pixmap (rows,cols of
 *   type long), and a pointer to a 2D array (MAXROWS x MAXCOLS) in row
 *   major order.
 * OUTPUT: an integer is returned indicating if the desired file was written
 *   (in P5 PGM format (binary)).  A 1 is returned if the write was completed
 *   and 0 if it was not.  An error message is returned if the file is not
 *   properly opened.  
 */ 
int ImageHelperFunctions::pgmWrite(const char* filename, unsigned int cols, unsigned int rows,
	     unsigned char* image,char* comment_string) {
      FILE* file;        /* pointer to the file buffer */
      //int maxval;        /* maximum value in the image array */
      long nwritten = 0; /* counter for the number of pixels written */
      long i;//,j;          /* for loop counters */

      /* open the file; write header and comments specified by the user. */
      if ((file = fopen(filename, "w")) == NULL)	{
           printf("ERROR: file open failed\n");
	   return(0);
      }
      fprintf(file,"P5\n");

      if (comment_string != NULL) fprintf(file,"# %s \n", comment_string);
    
      /* write the dimensions of the image */	
      fprintf(file,"%ld %ld \n", cols, rows);

      /* NOTE: MAXIMUM VALUE IS WHITE; COLOURS ARE SCALED FROM 0 - */
      /* MAXVALUE IN A .PGM FILE. */
      
      /* WRITE MAXIMUM VALUE TO FILE */
      fprintf(file, "%d\n", (int)255);

      /* Write data */

      for (i=0; i < rows; i++) {
          nwritten += fwrite((void*)&(image[i*cols]),sizeof(unsigned char),
	  		   cols, file);
      }	

      fclose(file);
      return(1);
}

//save the calculated disparity map from image 1 to image 2 as a grayscale image using the SCALE_MOVEMENT factor with
//0 representing "zero" intensity and the intensity linearly increasing from there using SCALE_MOVEMENT
void ImageHelperFunctions::saveDisparityImageToPGM(const char* filePathSaveImage, float scaleMovement, float*& calcDisparityBetweenImages, unsigned int widthImage, unsigned int heightImage)
{
	//declare and allocate the space for the movement image to save
	unsigned char* movementImageToSave = new unsigned char[widthImage*heightImage];

	//go though every value in the movementBetweenImages data and retrieve the intensity value to use in the resulting "movement image" where minMovementDirection
	//represents 0 intensity and the intensity increases linearly using scaleMovement from minMovementDirection
	for (unsigned int currentPixel = 0; currentPixel < (widthImage*heightImage); currentPixel++)
	{
		//add .5 and truncate to "round" the intensity to save to an integer
		movementImageToSave[currentPixel] = (unsigned char)((calcDisparityBetweenImages[currentPixel])*scaleMovement + .5f);
	}

	pgmWrite(filePathSaveImage, widthImage, heightImage,
	     movementImageToSave, "blah");
}

//save the output disparity map using the scale defined in scaleDisparityInOutput at each pixel to the file at disparityMapSaveImagePath
//also takes in the timer to time the implementation including the transfer time from the device to the host
void ImageHelperFunctions::saveResultingDisparityMap(const char* disparityMapSaveImagePath,
		float*& disparityMapFromImage1To2Device, float scaleDisparityInOutput,
		unsigned int widthImages, unsigned int heightImages,
		std::chrono::time_point<std::chrono::system_clock>& timeWithTransferStart,
		double& totalTimeIncludeTransfer) {
	//allocate the space on the host for and x and y movement between images
	float* disparityMapFromImage1To2Host = new float[widthImages * heightImages];

	//transfer the disparity map estimation on the device to the host for output
	(hipMemcpy(disparityMapFromImage1To2Host, disparityMapFromImage1To2Device, widthImages*heightImages*sizeof(float),
						  hipMemcpyDeviceToHost) );

	auto timeWithTransferEnd = std::chrono::system_clock::now();

	//printf("Running time including transfer time: %.10lf seconds\n", timeEnd-timeStart);
	std::chrono::duration<double> diff = timeWithTransferEnd-timeWithTransferStart;
	totalTimeIncludeTransfer = diff.count();
	//stop the timer and print the total time of the BP implementation including the device-host transfer time
	//printf("Time to retrieve movement on host (including transfer): %f (ms) \n", totalTimeIncludeTransfer);

	//save the resulting disparity map images to a file
	ImageHelperFunctions::saveDisparityImageToPGM(disparityMapSaveImagePath, scaleDisparityInOutput, disparityMapFromImage1To2Host, widthImages, heightImages);

	delete [] disparityMapFromImage1To2Host;
}




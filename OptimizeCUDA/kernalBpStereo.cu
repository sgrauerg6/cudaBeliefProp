#include "hip/hip_runtime.h"
/*
Copyright (C) 2009 Scott Grauer-Gray, Chandra Kambhamettu, and Kannappan Palaniappan

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307 USA
*/

//This file defines the methods to perform belief propagation for disparity map estimation from stereo images on CUDA


//#include "kernalBpStereoHeader.cuh"
#include "../ParameterFiles/bpStereoCudaParameters.h"
#define PROCESSING_ON_GPU
#include "../SharedFuncts/SharedBPProcessingFuncts.h"
#undef PROCESSING_ON_GPU

#if ((USE_SHARED_MEMORY == 1) && (DISP_INDEX_START_REG_LOCAL_MEM > 0))
#include "SharedMemoryKernels/KernalBpStereoUseSharedMemory.cu"
#elif ((USE_SHARED_MEMORY == 2) && (DISP_INDEX_START_REG_LOCAL_MEM > 0))
#include "SharedMemoryKernels/KernalBpStereoUseSharedMemoryActuallyDuplicateRegMem.cu"
#elif ((USE_SHARED_MEMORY == 3) && (DISP_INDEX_START_REG_LOCAL_MEM > 0))
#include "SharedMemoryKernels/KernelBpStereoUseDynamicSharedMemory.cu"
#elif ((USE_SHARED_MEMORY == 4) && (DISP_INDEX_START_REG_LOCAL_MEM > 0))
#include "SharedMemoryKernels/KernelBpStereoDataAndMessageInDynamicSharedMemory.cu"
#else

//set constexpr unsigned int values for number of disparity values for each image set used
constexpr unsigned int DISP_VALS_0{bp_params::NUM_POSSIBLE_DISPARITY_VALUES[0]};
constexpr unsigned int DISP_VALS_1{bp_params::NUM_POSSIBLE_DISPARITY_VALUES[1]};
constexpr unsigned int DISP_VALS_2{bp_params::NUM_POSSIBLE_DISPARITY_VALUES[2]};
constexpr unsigned int DISP_VALS_3{bp_params::NUM_POSSIBLE_DISPARITY_VALUES[3]};
constexpr unsigned int DISP_VALS_4{bp_params::NUM_POSSIBLE_DISPARITY_VALUES[4]};

#ifdef CUDA_HALF_SUPPORT
//template specialization for processing messages with half-precision; has safeguard to check if valToNormalize goes to infinity and set output
//for every disparity at point to be 0.0 if that's the case; this has only been observed when using more than 5 computation levels with half-precision
template<>
__device__ inline void msgStereo<half, half>(const unsigned int xVal, const unsigned int yVal,
		const levelProperties& currentLevelProperties,
		half* messageValsNeighbor1,
		half* messageValsNeighbor2,
		half* messageValsNeighbor3,
		half* dataCosts, half* dstMessageArray,
		const half disc_k_bp, const bool dataAligned,
		const unsigned int bpSettingsDispVals)
{
	// aggregate and find min
	half minimum = bp_consts::INF_BP;

	half* dst = new half[bpSettingsDispVals];

	for (unsigned int currentDisparity = 0;
			currentDisparity < bpSettingsDispVals;
			currentDisparity++) {
		dst[currentDisparity] = messageValsNeighbor1[currentDisparity]
				+ messageValsNeighbor2[currentDisparity]
				+ messageValsNeighbor3[currentDisparity]
				+ dataCosts[currentDisparity];
		if (dst[currentDisparity] < minimum)
			minimum = dst[currentDisparity];
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
	dtStereo<half>(dst, bpSettingsDispVals);

	// truncate
	minimum += disc_k_bp;

	// normalize
	half valToNormalize = 0;

	for (unsigned int currentDisparity = 0;
			currentDisparity < bpSettingsDispVals;
			currentDisparity++)
	{
		if (minimum < dst[currentDisparity])
		{
			dst[currentDisparity] = minimum;
		}

		valToNormalize += dst[currentDisparity];
	}

	//if valToNormalize is infinite or NaN (observed when using more than 5 computation levels with half-precision),
	//set destination vector to 0 for all disparities
	//note that may cause results to differ a little from ideal
	if (__hisnan(valToNormalize) || ((__hisinf(valToNormalize)) != 0)) {
		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				bpSettingsDispVals);

		for (unsigned int currentDisparity = 0;
				currentDisparity < bpSettingsDispVals;
				currentDisparity++) {
			dstMessageArray[destMessageArrayIndex] = (half) 0.0;
			if /*constexpr*/ (OPTIMIZED_INDEXING_SETTING)
			{
				destMessageArrayIndex +=
					currentLevelProperties.paddedWidthCheckerboardLevel_;
			}
			else
			{
				destMessageArrayIndex++;
			}
		}
	}
	else
	{
		valToNormalize /= bpSettingsDispVals;

		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				bpSettingsDispVals);

		for (unsigned int currentDisparity = 0;
				currentDisparity < bpSettingsDispVals;
				currentDisparity++)
		{
			dst[currentDisparity] -= valToNormalize;
			dstMessageArray[destMessageArrayIndex] = dst[currentDisparity];
			if /*constexpr*/ (OPTIMIZED_INDEXING_SETTING)
			{
				destMessageArrayIndex +=
						currentLevelProperties.paddedWidthCheckerboardLevel_;
			}
			else
			{
				destMessageArrayIndex++;
			}
		}
	}

	delete [] dst;
}

template<>
__device__ inline void msgStereo<half, half>(const unsigned int xVal, const unsigned int yVal, const levelProperties& currentLevelProperties,
		half* prevUMessageArray, half* prevDMessageArray,
		half* prevLMessageArray, half* prevRMessageArray,
		half* dataMessageArray, half* dstMessageArray,
		half disc_k_bp, const bool dataAligned, const unsigned int bpSettingsDispVals,
		half* dstProcessing, const messageComp& currMessageComp,
		const unsigned int checkerboardAdjustment,
		const unsigned int offsetData)
{
	// aggregate and find min
	half minimum{(half)bp_consts::INF_BP};
	unsigned int processingArrIndexDisp0 = retrieveIndexInDataAndMessage(xVal, yVal,
					currentLevelProperties.paddedWidthCheckerboardLevel_,
					currentLevelProperties.heightLevel_, 0,
					bpSettingsDispVals);
	unsigned int procArrIdx{processingArrIndexDisp0};

	for (unsigned int currentDisparity = 0; currentDisparity < bpSettingsDispVals; currentDisparity++)
	{
		if (OPTIMIZED_INDEXING_SETTING) {
			procArrIdx += currentLevelProperties.paddedWidthCheckerboardLevel_;
		}
		else {
			procArrIdx++;
		}

		const half prevUVal = convertValToDifferentDataTypeIfNeeded<half, half>(prevUMessageArray[retrieveIndexInDataAndMessage(xVal, (yVal+1),
				currentLevelProperties.paddedWidthCheckerboardLevel_, currentLevelProperties.heightLevel_,
				currentDisparity, bpSettingsDispVals)]);
		const half prevDVal = convertValToDifferentDataTypeIfNeeded<half, half>(prevDMessageArray[retrieveIndexInDataAndMessage(xVal, (yVal-1),
				currentLevelProperties.paddedWidthCheckerboardLevel_, currentLevelProperties.heightLevel_,
				currentDisparity, bpSettingsDispVals)]);
		const half prevLVal = convertValToDifferentDataTypeIfNeeded<half, half>(prevLMessageArray[retrieveIndexInDataAndMessage((xVal + checkerboardAdjustment), yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_, currentLevelProperties.heightLevel_,
				currentDisparity, bpSettingsDispVals)]);
		const half prevRVal = convertValToDifferentDataTypeIfNeeded<half, half>(prevRMessageArray[retrieveIndexInDataAndMessage(((xVal + checkerboardAdjustment) - 1), yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_, currentLevelProperties.heightLevel_,
				currentDisparity, bpSettingsDispVals)]);
		const half dataVal = convertValToDifferentDataTypeIfNeeded<half, half>(dataMessageArray[retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_, currentLevelProperties.heightLevel_,
				currentDisparity, bpSettingsDispVals, offsetData)]);

		if (currMessageComp == messageComp::U_MESSAGE) {
			dstProcessing[procArrIdx] = prevUVal + prevLVal + prevRVal + dataVal;
		}
		else if (currMessageComp == messageComp::D_MESSAGE) {
			dstProcessing[procArrIdx] = prevDVal + prevLVal + prevRVal + dataVal;
		}
		else if (currMessageComp == messageComp::L_MESSAGE) {
			dstProcessing[procArrIdx] = prevUVal + prevDVal + prevLVal + dataVal;
		}
		else if (currMessageComp == messageComp::R_MESSAGE) {
			dstProcessing[procArrIdx] = prevUVal + prevDVal + prevRVal + dataVal;
		}

		if (dstProcessing[procArrIdx] < minimum)
			minimum = dstProcessing[procArrIdx];
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
	dtStereo<half>(dstProcessing, bpSettingsDispVals, xVal, yVal, currentLevelProperties);

	// truncate
	minimum += disc_k_bp;

	// normalize
	half valToNormalize{(half)0.0};

	procArrIdx = processingArrIndexDisp0;
	for (unsigned int currentDisparity = 0; currentDisparity < bpSettingsDispVals; currentDisparity++) {
		if (OPTIMIZED_INDEXING_SETTING) {
			procArrIdx += currentLevelProperties.paddedWidthCheckerboardLevel_;
		}
		else {
			procArrIdx++;
		}
		if (minimum < dstProcessing[procArrIdx]) {
			dstProcessing[procArrIdx] = minimum;
		}

		valToNormalize += dstProcessing[procArrIdx];
	}

	//if valToNormalize is infinite or NaN (observed when using more than 5 computation levels with half-precision),
	//set destination vector to 0 for all disparities
	//note that may cause results to differ a little from ideal
	if (__hisnan(valToNormalize) || ((__hisinf(valToNormalize)) != 0)) {
		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				bpSettingsDispVals);

		for (unsigned int currentDisparity = 0;
				currentDisparity < bpSettingsDispVals;
				currentDisparity++) {
			dstMessageArray[destMessageArrayIndex] = (half)0.0;
			if /*constexpr*/ (OPTIMIZED_INDEXING_SETTING)
			{
				destMessageArrayIndex +=
					currentLevelProperties.paddedWidthCheckerboardLevel_;
			}
			else
			{
				destMessageArrayIndex++;
			}
		}
	}
	else {
		valToNormalize /= ((half)bpSettingsDispVals);

		//dst processing index and message array index are the same for each disparity value in this processing
		procArrIdx = processingArrIndexDisp0;

		for (unsigned int currentDisparity = 0; currentDisparity < bpSettingsDispVals; currentDisparity++) {
			dstProcessing[procArrIdx] -= valToNormalize;
			dstMessageArray[procArrIdx] = convertValToDifferentDataTypeIfNeeded<half, half>(dstProcessing[procArrIdx]);
#ifdef _WIN32
			//assuming that width includes padding
			if /*constexpr*/ (OPTIMIZED_INDEXING_SETTING)
#else
			if (OPTIMIZED_INDEXING_SETTING)
#endif
			{
				procArrIdx += currentLevelProperties.paddedWidthCheckerboardLevel_;
			}
			else {
				procArrIdx++;
			}
		}
	}
}

template<>
__device__ inline void msgStereo<half, half, DISP_VALS_0>(const unsigned int xVal, const unsigned int yVal,
		const levelProperties& currentLevelProperties,
		half messageValsNeighbor1[DISP_VALS_0],
		half messageValsNeighbor2[DISP_VALS_0],
		half messageValsNeighbor3[DISP_VALS_0],
		half dataCosts[DISP_VALS_0], half* dstMessageArray,
		const half disc_k_bp, const bool dataAligned)
{
	// aggregate and find min
	half minimum = bp_consts::INF_BP;

	half dst[DISP_VALS_0];

	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_VALS_0;
			currentDisparity++) {
		dst[currentDisparity] = messageValsNeighbor1[currentDisparity]
				+ messageValsNeighbor2[currentDisparity]
				+ messageValsNeighbor3[currentDisparity]
				+ dataCosts[currentDisparity];
		if (dst[currentDisparity] < minimum)
			minimum = dst[currentDisparity];
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
	dtStereo<half, DISP_VALS_0>(dst);

	// truncate
	minimum += disc_k_bp;

	// normalize
	half valToNormalize = 0;

	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_VALS_0;
			currentDisparity++)
	{
		if (minimum < dst[currentDisparity])
		{
			dst[currentDisparity] = minimum;
		}

		valToNormalize += dst[currentDisparity];
	}

	//if valToNormalize is infinite or NaN (observed when using more than 5 computation levels with half-precision),
	//set destination vector to 0 for all disparities
	//note that may cause results to differ a little from ideal
	if (__hisnan(valToNormalize) || ((__hisinf(valToNormalize)) != 0)) {
		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				DISP_VALS_0);

		for (unsigned int currentDisparity = 0;
				currentDisparity < DISP_VALS_0;
				currentDisparity++) {
			dstMessageArray[destMessageArrayIndex] = (half) 0.0;
			if /*constexpr*/ (OPTIMIZED_INDEXING_SETTING)
			{
				destMessageArrayIndex +=
					currentLevelProperties.paddedWidthCheckerboardLevel_;
			}
			else
			{
				destMessageArrayIndex++;
			}
		}
	}
	else
	{
		valToNormalize /= DISP_VALS_0;

		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				DISP_VALS_0);

		for (unsigned int currentDisparity = 0;
				currentDisparity < DISP_VALS_0;
				currentDisparity++)
		{
			dst[currentDisparity] -= valToNormalize;
			dstMessageArray[destMessageArrayIndex] = dst[currentDisparity];
			if /*constexpr*/ (OPTIMIZED_INDEXING_SETTING)
			{
				destMessageArrayIndex +=
						currentLevelProperties.paddedWidthCheckerboardLevel_;
			}
			else
			{
				destMessageArrayIndex++;
			}
		}
	}
}

template<>
__device__ inline void msgStereo<half, half, DISP_VALS_1>(const unsigned int xVal, const unsigned int yVal,
		const levelProperties& currentLevelProperties,
		half messageValsNeighbor1[DISP_VALS_1],
		half messageValsNeighbor2[DISP_VALS_1],
		half messageValsNeighbor3[DISP_VALS_1],
		half dataCosts[DISP_VALS_1], half* dstMessageArray,
		const half disc_k_bp, const bool dataAligned)
{
	// aggregate and find min
	half minimum = bp_consts::INF_BP;

	half dst[DISP_VALS_1];

	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_VALS_1;
			currentDisparity++) {
		dst[currentDisparity] = messageValsNeighbor1[currentDisparity]
				+ messageValsNeighbor2[currentDisparity]
				+ messageValsNeighbor3[currentDisparity]
				+ dataCosts[currentDisparity];
		if (dst[currentDisparity] < minimum)
			minimum = dst[currentDisparity];
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
	dtStereo<half, DISP_VALS_1>(dst);

	// truncate
	minimum += disc_k_bp;

	// normalize
	half valToNormalize = 0;

	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_VALS_1;
			currentDisparity++)
	{
		if (minimum < dst[currentDisparity])
		{
			dst[currentDisparity] = minimum;
		}

		valToNormalize += dst[currentDisparity];
	}

	//if valToNormalize is infinite or NaN (observed when using more than 5 computation levels with half-precision),
	//set destination vector to 0 for all disparities
	//note that may cause results to differ a little from ideal
	if (__hisnan(valToNormalize) || ((__hisinf(valToNormalize)) != 0)) {
		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				DISP_VALS_1);

		for (unsigned int currentDisparity = 0;
				currentDisparity < DISP_VALS_1;
				currentDisparity++) {
			dstMessageArray[destMessageArrayIndex] = (half) 0.0;
			if /*constexpr*/ (OPTIMIZED_INDEXING_SETTING)
			{
				destMessageArrayIndex +=
					currentLevelProperties.paddedWidthCheckerboardLevel_;
			}
			else
			{
				destMessageArrayIndex++;
			}
		}
	}
	else
	{
		valToNormalize /= DISP_VALS_1;

		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				DISP_VALS_1);

		for (unsigned int currentDisparity = 0;
				currentDisparity < DISP_VALS_1;
				currentDisparity++)
		{
			dst[currentDisparity] -= valToNormalize;
			dstMessageArray[destMessageArrayIndex] = dst[currentDisparity];
			if /*constexpr*/ (OPTIMIZED_INDEXING_SETTING)
			{
				destMessageArrayIndex +=
						currentLevelProperties.paddedWidthCheckerboardLevel_;
			}
			else
			{
				destMessageArrayIndex++;
			}
		}
	}
}

template<>
__device__ inline void msgStereo<half, half, DISP_VALS_2>(const unsigned int xVal, const unsigned int yVal,
		const levelProperties& currentLevelProperties,
		half messageValsNeighbor1[DISP_VALS_2],
		half messageValsNeighbor2[DISP_VALS_2],
		half messageValsNeighbor3[DISP_VALS_2],
		half dataCosts[DISP_VALS_2], half* dstMessageArray,
		const half disc_k_bp, const bool dataAligned)
{
	// aggregate and find min
	half minimum = bp_consts::INF_BP;

	half dst[DISP_VALS_2];

	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_VALS_2;
			currentDisparity++) {
		dst[currentDisparity] = messageValsNeighbor1[currentDisparity]
				+ messageValsNeighbor2[currentDisparity]
				+ messageValsNeighbor3[currentDisparity]
				+ dataCosts[currentDisparity];
		if (dst[currentDisparity] < minimum)
			minimum = dst[currentDisparity];
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
	dtStereo<half, DISP_VALS_2>(dst);

	// truncate
	minimum += disc_k_bp;

	// normalize
	half valToNormalize = 0;

	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_VALS_2;
			currentDisparity++)
	{
		if (minimum < dst[currentDisparity])
		{
			dst[currentDisparity] = minimum;
		}

		valToNormalize += dst[currentDisparity];
	}

	//if valToNormalize is infinite or NaN (observed when using more than 5 computation levels with half-precision),
	//set destination vector to 0 for all disparities
	//note that may cause results to differ a little from ideal
	if (__hisnan(valToNormalize) || ((__hisinf(valToNormalize)) != 0)) {
		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				DISP_VALS_2);

		for (unsigned int currentDisparity = 0;
				currentDisparity < DISP_VALS_2;
				currentDisparity++) {
			dstMessageArray[destMessageArrayIndex] = (half) 0.0;
			if /*constexpr*/ (OPTIMIZED_INDEXING_SETTING)
			{
				destMessageArrayIndex +=
					currentLevelProperties.paddedWidthCheckerboardLevel_;
			}
			else
			{
				destMessageArrayIndex++;
			}
		}
	}
	else
	{
		valToNormalize /= DISP_VALS_2;

		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				DISP_VALS_2);

		for (unsigned int currentDisparity = 0;
				currentDisparity < DISP_VALS_2;
				currentDisparity++)
		{
			dst[currentDisparity] -= valToNormalize;
			dstMessageArray[destMessageArrayIndex] = dst[currentDisparity];
			if /*constexpr*/ (OPTIMIZED_INDEXING_SETTING)
			{
				destMessageArrayIndex +=
						currentLevelProperties.paddedWidthCheckerboardLevel_;
			}
			else
			{
				destMessageArrayIndex++;
			}
		}
	}
}

template<>
__device__ inline void msgStereo<half, half, DISP_VALS_3>(const unsigned int xVal, const unsigned int yVal,
		const levelProperties& currentLevelProperties,
		half messageValsNeighbor1[DISP_VALS_3],
		half messageValsNeighbor2[DISP_VALS_3],
		half messageValsNeighbor3[DISP_VALS_3],
		half dataCosts[DISP_VALS_3], half* dstMessageArray,
		const half disc_k_bp, const bool dataAligned)
{
	// aggregate and find min
	half minimum = bp_consts::INF_BP;

	half dst[DISP_VALS_3];

	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_VALS_3;
			currentDisparity++) {
		dst[currentDisparity] = messageValsNeighbor1[currentDisparity]
				+ messageValsNeighbor2[currentDisparity]
				+ messageValsNeighbor3[currentDisparity]
				+ dataCosts[currentDisparity];
		if (dst[currentDisparity] < minimum)
			minimum = dst[currentDisparity];
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
	dtStereo<half, DISP_VALS_3>(dst);

	// truncate
	minimum += disc_k_bp;

	// normalize
	half valToNormalize = 0;

	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_VALS_3;
			currentDisparity++)
	{
		if (minimum < dst[currentDisparity])
		{
			dst[currentDisparity] = minimum;
		}

		valToNormalize += dst[currentDisparity];
	}

	//if valToNormalize is infinite or NaN (observed when using more than 5 computation levels with half-precision),
	//set destination vector to 0 for all disparities
	//note that may cause results to differ a little from ideal
	if (__hisnan(valToNormalize) || ((__hisinf(valToNormalize)) != 0)) {
		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				DISP_VALS_3);

		for (unsigned int currentDisparity = 0;
				currentDisparity < DISP_VALS_3;
				currentDisparity++) {
			dstMessageArray[destMessageArrayIndex] = (half) 0.0;
			if /*constexpr*/ (OPTIMIZED_INDEXING_SETTING)
			{
				destMessageArrayIndex +=
					currentLevelProperties.paddedWidthCheckerboardLevel_;
			}
			else
			{
				destMessageArrayIndex++;
			}
		}
	}
	else
	{
		valToNormalize /= DISP_VALS_3;

		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				DISP_VALS_3);

		for (unsigned int currentDisparity = 0;
				currentDisparity < DISP_VALS_3;
				currentDisparity++)
		{
			dst[currentDisparity] -= valToNormalize;
			dstMessageArray[destMessageArrayIndex] = dst[currentDisparity];
			if /*constexpr*/ (OPTIMIZED_INDEXING_SETTING)
			{
				destMessageArrayIndex +=
						currentLevelProperties.paddedWidthCheckerboardLevel_;
			}
			else
			{
				destMessageArrayIndex++;
			}
		}
	}
}

template<>
__device__ inline void msgStereo<half, half, DISP_VALS_4>(const unsigned int xVal, const unsigned int yVal,
		const levelProperties& currentLevelProperties,
		half messageValsNeighbor1[DISP_VALS_4],
		half messageValsNeighbor2[DISP_VALS_4],
		half messageValsNeighbor3[DISP_VALS_4],
		half dataCosts[DISP_VALS_4], half* dstMessageArray,
		const half disc_k_bp, const bool dataAligned)
{
	// aggregate and find min
	half minimum = bp_consts::INF_BP;

	half dst[DISP_VALS_4];

	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_VALS_4;
			currentDisparity++) {
		dst[currentDisparity] = messageValsNeighbor1[currentDisparity]
				+ messageValsNeighbor2[currentDisparity]
				+ messageValsNeighbor3[currentDisparity]
				+ dataCosts[currentDisparity];
		if (dst[currentDisparity] < minimum)
			minimum = dst[currentDisparity];
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
	dtStereo<half, DISP_VALS_4>(dst);

	// truncate
	minimum += disc_k_bp;

	// normalize
	half valToNormalize = 0;

	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_VALS_4;
			currentDisparity++)
	{
		if (minimum < dst[currentDisparity])
		{
			dst[currentDisparity] = minimum;
		}

		valToNormalize += dst[currentDisparity];
	}

	//if valToNormalize is infinite or NaN (observed when using more than 5 computation levels with half-precision),
	//set destination vector to 0 for all disparities
	//note that may cause results to differ a little from ideal
	if (__hisnan(valToNormalize) || ((__hisinf(valToNormalize)) != 0)) {
		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				DISP_VALS_4);

		for (unsigned int currentDisparity = 0;
				currentDisparity < DISP_VALS_4;
				currentDisparity++) {
			dstMessageArray[destMessageArrayIndex] = (half) 0.0;
			if /*constexpr*/ (OPTIMIZED_INDEXING_SETTING)
			{
				destMessageArrayIndex +=
					currentLevelProperties.paddedWidthCheckerboardLevel_;
			}
			else
			{
				destMessageArrayIndex++;
			}
		}
	}
	else
	{
		valToNormalize /= DISP_VALS_4;

		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				DISP_VALS_4);

		for (unsigned int currentDisparity = 0;
				currentDisparity < DISP_VALS_4;
				currentDisparity++)
		{
			dst[currentDisparity] -= valToNormalize;
			dstMessageArray[destMessageArrayIndex] = dst[currentDisparity];
			if /*constexpr*/ (OPTIMIZED_INDEXING_SETTING)
			{
				destMessageArrayIndex +=
						currentLevelProperties.paddedWidthCheckerboardLevel_;
			}
			else
			{
				destMessageArrayIndex++;
			}
		}
	}
}


#endif //#if ((USE_SHARED_MEMORY == 1) && (DISP_INDEX_START_REG_LOCAL_MEM > 0))

#endif //CUDA_HALF_SUPPORT

//initialize the "data cost" for each possible disparity between the two full-sized input images ("bottom" of the image pyramid)
//the image data is stored in the CUDA arrays image1PixelsTextureBPStereo and image2PixelsTextureBPStereo
template<typename T, unsigned int DISP_VALS>
__global__ void initializeBottomLevelDataStereo(
		const levelProperties currentLevelProperties,
		float* image1PixelsDevice, float* image2PixelsDevice,
		T* dataCostDeviceStereoCheckerboard0, T* dataCostDeviceStereoCheckerboard1,
		const float lambda_bp, float data_k_bp, const unsigned int bpSettingsDispVals)
{
	// Block index
    const unsigned int bx = blockIdx.x;
    const unsigned int by = blockIdx.y;

    // Thread index
    const unsigned int tx = threadIdx.x;
    const unsigned int ty = threadIdx.y;

    const unsigned int xVal = bx * bp_cuda_params::BLOCK_SIZE_WIDTH_BP + tx;
    const unsigned int yVal = by * bp_cuda_params::BLOCK_SIZE_HEIGHT_BP + ty;

    const unsigned int xInCheckerboard = xVal / 2;

	if (withinImageBounds(xInCheckerboard, yVal, currentLevelProperties.widthLevel_, currentLevelProperties.heightLevel_))
	{
		initializeBottomLevelDataStereoPixel<T, DISP_VALS>(xVal, yVal,
				currentLevelProperties, image1PixelsDevice,
				image2PixelsDevice, dataCostDeviceStereoCheckerboard0,
				dataCostDeviceStereoCheckerboard1, lambda_bp,
				data_k_bp, bpSettingsDispVals);
	}
}

//initialize the data costs at the "next" level up in the pyramid given that the data at the lower has been set
template<typename T, unsigned int DISP_VALS>
__global__ void initializeCurrentLevelDataStereo(
		const Checkerboard_Parts checkerboardPart,
		const levelProperties currentLevelProperties,
		const levelProperties prevLevelProperties, T* dataCostStereoCheckerboard0,
		T* dataCostStereoCheckerboard1, T* dataCostDeviceToWriteTo,
		const unsigned int offsetNum, const unsigned int bpSettingsDispVals)
{
	// Block index
	const unsigned int bx = blockIdx.x;
	const unsigned int by = blockIdx.y;

	// Thread index
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;

	const unsigned int xVal = bx * bp_cuda_params::BLOCK_SIZE_WIDTH_BP + tx;
	const unsigned int yVal = by * bp_cuda_params::BLOCK_SIZE_HEIGHT_BP + ty;

	if (withinImageBounds(xVal, yVal, currentLevelProperties.widthCheckerboardLevel_, currentLevelProperties.heightLevel_))
	{
		initializeCurrentLevelDataStereoPixel<T, T, DISP_VALS>(
				xVal, yVal, checkerboardPart,
				currentLevelProperties,
				prevLevelProperties, dataCostStereoCheckerboard0,
				dataCostStereoCheckerboard1, dataCostDeviceToWriteTo,
				offsetNum, bpSettingsDispVals);
	}
}


//initialize the message values at each pixel of the current level to the default value
template<typename T, unsigned int DISP_VALS>
__global__ void initializeMessageValsToDefaultKernel(
		const levelProperties currentLevelProperties,
		T* messageUDeviceCurrentCheckerboard0,
		T* messageDDeviceCurrentCheckerboard0,
		T* messageLDeviceCurrentCheckerboard0,
		T* messageRDeviceCurrentCheckerboard0,
		T* messageUDeviceCurrentCheckerboard1,
		T* messageDDeviceCurrentCheckerboard1,
		T* messageLDeviceCurrentCheckerboard1,
		T* messageRDeviceCurrentCheckerboard1,
		const unsigned int bpSettingsDispVals)
{
	// Block index
	const unsigned int bx = blockIdx.x;
	const unsigned int by = blockIdx.y;

	// Thread index
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;

	const unsigned int xValInCheckerboard = bx * bp_cuda_params::BLOCK_SIZE_WIDTH_BP + tx;
	const unsigned int yVal = by * bp_cuda_params::BLOCK_SIZE_HEIGHT_BP + ty;

	if (withinImageBounds(xValInCheckerboard, yVal, currentLevelProperties.widthCheckerboardLevel_, currentLevelProperties.heightLevel_))
	{
		//initialize message values in both checkerboards
		initializeMessageValsToDefaultKernelPixel<T, DISP_VALS>(xValInCheckerboard,  yVal, currentLevelProperties,
				messageUDeviceCurrentCheckerboard0, messageDDeviceCurrentCheckerboard0,
				messageLDeviceCurrentCheckerboard0, messageRDeviceCurrentCheckerboard0,
				messageUDeviceCurrentCheckerboard1, messageDDeviceCurrentCheckerboard1,
				messageLDeviceCurrentCheckerboard1, messageRDeviceCurrentCheckerboard1,
				bpSettingsDispVals);
	}
}


//kernal function to run the current iteration of belief propagation in parallel using the checkerboard update method where half the pixels in the "checkerboard"
//scheme retrieve messages from each 4-connected neighbor and then update their message based on the retrieved messages and the data cost
template<typename T, unsigned int DISP_VALS>
__global__ void runBPIterationUsingCheckerboardUpdates(
		const Checkerboard_Parts checkerboardToUpdate, const levelProperties currentLevelProperties,
		T* dataCostStereoCheckerboard0, T* dataCostStereoCheckerboard1,
		T* messageUDeviceCurrentCheckerboard0, T* messageDDeviceCurrentCheckerboard0,
		T* messageLDeviceCurrentCheckerboard0, T* messageRDeviceCurrentCheckerboard0,
		T* messageUDeviceCurrentCheckerboard1, T* messageDDeviceCurrentCheckerboard1,
		T* messageLDeviceCurrentCheckerboard1, T* messageRDeviceCurrentCheckerboard1,
		const float disc_k_bp, const bool dataAligned, const unsigned int bpSettingsDispVals)
{
	// Block index
	const unsigned int bx = blockIdx.x;
	const unsigned int by = blockIdx.y;

	// Thread index
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;

	const unsigned int xVal = bx * bp_cuda_params::BLOCK_SIZE_WIDTH_BP + tx;
	const unsigned int yVal = by * bp_cuda_params::BLOCK_SIZE_HEIGHT_BP + ty;

	if (withinImageBounds(xVal, yVal, currentLevelProperties.widthLevel_/2, currentLevelProperties.heightLevel_))
	{
		runBPIterationUsingCheckerboardUpdatesDeviceNoTexBoundAndLocalMemPixel<T, T, DISP_VALS>(
				xVal, yVal, checkerboardToUpdate, currentLevelProperties,
				dataCostStereoCheckerboard0, dataCostStereoCheckerboard1,
				messageUDeviceCurrentCheckerboard0, messageDDeviceCurrentCheckerboard0,
				messageLDeviceCurrentCheckerboard0, messageRDeviceCurrentCheckerboard0,
				messageUDeviceCurrentCheckerboard1, messageDDeviceCurrentCheckerboard1,
				messageLDeviceCurrentCheckerboard1, messageRDeviceCurrentCheckerboard1,
				disc_k_bp, 0, dataAligned, bpSettingsDispVals);
	}
}


template<typename T, unsigned int DISP_VALS>
__global__ void runBPIterationUsingCheckerboardUpdates(
		const Checkerboard_Parts checkerboardToUpdate, const levelProperties currentLevelProperties,
		T* dataCostStereoCheckerboard0, T* dataCostStereoCheckerboard1,
		T* messageUDeviceCurrentCheckerboard0, T* messageDDeviceCurrentCheckerboard0,
		T* messageLDeviceCurrentCheckerboard0, T* messageRDeviceCurrentCheckerboard0,
		T* messageUDeviceCurrentCheckerboard1, T* messageDDeviceCurrentCheckerboard1,
		T* messageLDeviceCurrentCheckerboard1, T* messageRDeviceCurrentCheckerboard1,
		const float disc_k_bp, const bool dataAligned, const unsigned int bpSettingsDispVals,
		void* dstProcessing)
{
	// Block index
	const unsigned int bx = blockIdx.x;
	const unsigned int by = blockIdx.y;

	// Thread index
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;

	const unsigned int xVal = bx * bp_cuda_params::BLOCK_SIZE_WIDTH_BP + tx;
	const unsigned int yVal = by * bp_cuda_params::BLOCK_SIZE_HEIGHT_BP + ty;

	if (withinImageBounds(xVal, yVal, currentLevelProperties.widthLevel_/2, currentLevelProperties.heightLevel_))
	{
		runBPIterationUsingCheckerboardUpdatesDeviceNoTexBoundAndLocalMemPixel<T, T, DISP_VALS>(
				xVal, yVal, checkerboardToUpdate, currentLevelProperties,
				dataCostStereoCheckerboard0, dataCostStereoCheckerboard1,
				messageUDeviceCurrentCheckerboard0, messageDDeviceCurrentCheckerboard0,
				messageLDeviceCurrentCheckerboard0, messageRDeviceCurrentCheckerboard0,
				messageUDeviceCurrentCheckerboard1, messageDDeviceCurrentCheckerboard1,
				messageLDeviceCurrentCheckerboard1, messageRDeviceCurrentCheckerboard1,
				disc_k_bp, 0, dataAligned, bpSettingsDispVals, dstProcessing);
	}
}


//kernal to copy the computed BP message values at the current level to the corresponding locations at the "next" level down
//the kernal works from the point of view of the pixel at the prev level that is being copied to four different places
template<typename T, unsigned int DISP_VALS>
__global__ void copyPrevLevelToNextLevelBPCheckerboardStereo(
		const Checkerboard_Parts checkerboardPart,
		const levelProperties currentLevelProperties,
		const levelProperties nextLevelProperties,
		T* messageUPrevStereoCheckerboard0, T* messageDPrevStereoCheckerboard0,
		T* messageLPrevStereoCheckerboard0, T* messageRPrevStereoCheckerboard0,
		T* messageUPrevStereoCheckerboard1, T* messageDPrevStereoCheckerboard1,
		T* messageLPrevStereoCheckerboard1, T* messageRPrevStereoCheckerboard1,
		T* messageUDeviceCurrentCheckerboard0, T* messageDDeviceCurrentCheckerboard0,
		T* messageLDeviceCurrentCheckerboard0, T* messageRDeviceCurrentCheckerboard0,
		T* messageUDeviceCurrentCheckerboard1, T* messageDDeviceCurrentCheckerboard1,
		T* messageLDeviceCurrentCheckerboard1, T* messageRDeviceCurrentCheckerboard1,
		const unsigned int bpSettingsDispVals)
{
	// Block index
	const unsigned int bx = blockIdx.x;
	const unsigned int by = blockIdx.y;

	// Thread index
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;

	const unsigned int xVal = bx * bp_cuda_params::BLOCK_SIZE_WIDTH_BP + tx;
	const unsigned int yVal = by * bp_cuda_params::BLOCK_SIZE_HEIGHT_BP + ty;

	if (withinImageBounds(xVal, yVal, currentLevelProperties.widthCheckerboardLevel_, currentLevelProperties.heightLevel_))
	{
		copyPrevLevelToNextLevelBPCheckerboardStereoPixel<T, DISP_VALS>(xVal, yVal,
				checkerboardPart, currentLevelProperties, nextLevelProperties,
				messageUPrevStereoCheckerboard0, messageDPrevStereoCheckerboard0,
				messageLPrevStereoCheckerboard0, messageRPrevStereoCheckerboard0,
				messageUPrevStereoCheckerboard1, messageDPrevStereoCheckerboard1,
				messageLPrevStereoCheckerboard1, messageRPrevStereoCheckerboard1,
				messageUDeviceCurrentCheckerboard0, messageDDeviceCurrentCheckerboard0,
				messageLDeviceCurrentCheckerboard0, messageRDeviceCurrentCheckerboard0,
				messageUDeviceCurrentCheckerboard1, messageDDeviceCurrentCheckerboard1,
				messageLDeviceCurrentCheckerboard1, messageRDeviceCurrentCheckerboard1,
				bpSettingsDispVals);
	}
}


//retrieve the best disparity estimate from image 1 to image 2 for each pixel in parallel
template<typename T, unsigned int DISP_VALS>
__global__ void retrieveOutputDisparityCheckerboardStereoOptimized(
		const levelProperties currentLevelProperties, T* dataCostStereoCheckerboard0,
		T* dataCostStereoCheckerboard1, T* messageUPrevStereoCheckerboard0,
		T* messageDPrevStereoCheckerboard0, T* messageLPrevStereoCheckerboard0,
		T* messageRPrevStereoCheckerboard0, T* messageUPrevStereoCheckerboard1,
		T* messageDPrevStereoCheckerboard1, T* messageLPrevStereoCheckerboard1,
		T* messageRPrevStereoCheckerboard1,
		float* disparityBetweenImagesDevice, const unsigned int bpSettingsDispVals)
{
	// Block index
	const unsigned int bx = blockIdx.x;
	const unsigned int by = blockIdx.y;

	// Thread index
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;

	const unsigned int xVal = bx * bp_cuda_params::BLOCK_SIZE_WIDTH_BP + tx;
	const unsigned int yVal = by * bp_cuda_params::BLOCK_SIZE_HEIGHT_BP + ty;

	if (withinImageBounds(xVal, yVal, currentLevelProperties.widthCheckerboardLevel_, currentLevelProperties.heightLevel_))
	{
		retrieveOutputDisparityCheckerboardStereoOptimizedPixel<T, T, DISP_VALS>(
				xVal, yVal, currentLevelProperties,
				dataCostStereoCheckerboard0, dataCostStereoCheckerboard1,
				messageUPrevStereoCheckerboard0, messageDPrevStereoCheckerboard0,
				messageLPrevStereoCheckerboard0, messageRPrevStereoCheckerboard0,
				messageUPrevStereoCheckerboard1, messageDPrevStereoCheckerboard1,
				messageLPrevStereoCheckerboard1, messageRPrevStereoCheckerboard1,
				disparityBetweenImagesDevice, bpSettingsDispVals);
	}
}

template<typename T, unsigned int DISP_VALS>
__global__ void printDataAndMessageValsAtPointKernel(
		const unsigned int xVal, const unsigned int yVal,
		T* dataCostStereoCheckerboard0, T* dataCostStereoCheckerboard1,
		T* messageUDeviceCurrentCheckerboard0, T* messageDDeviceCurrentCheckerboard0,
		T* messageLDeviceCurrentCheckerboard0, T* messageRDeviceCurrentCheckerboard0,
		T* messageUDeviceCurrentCheckerboard1, T* messageDDeviceCurrentCheckerboard1,
		T* messageLDeviceCurrentCheckerboard1, T* messageRDeviceCurrentCheckerboard1,
		const unsigned int widthLevelCheckerboardPart, const unsigned int heightLevel)
{
	if (((xVal + yVal) % 2) == 0) {
		printf("xVal: %d\n", xVal);
		printf("yVal: %d\n", yVal);
		for (unsigned int currentDisparity = 0; currentDisparity < DISP_VALS; currentDisparity++) {
			printf("DISP: %d\n", currentDisparity);
			printf("messageUPrevStereoCheckerboard: %f \n",
					(float) messageUDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageDPrevStereoCheckerboard: %f \n",
					(float) messageDDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageLPrevStereoCheckerboard: %f \n",
					(float) messageLDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageRPrevStereoCheckerboard: %f \n",
					(float) messageRDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("dataCostStereoCheckerboard: %f \n",
					(float) dataCostStereoCheckerboard0[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
		}
	} else {
		printf("xVal: %d\n", xVal);
		printf("yVal: %d\n", yVal);
		for (unsigned int currentDisparity = 0; currentDisparity < DISP_VALS; currentDisparity++) {
			printf("DISP: %d\n", currentDisparity);
			printf("messageUPrevStereoCheckerboard: %f \n",
					(float) messageUDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageDPrevStereoCheckerboard: %f \n",
					(float) messageDDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageLPrevStereoCheckerboard: %f \n",
					(float) messageLDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageRPrevStereoCheckerboard: %f \n",
					(float) messageRDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("dataCostStereoCheckerboard: %f \n",
					(float) dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
		}
	}
}


template<typename T, unsigned int DISP_VALS>
__device__ void printDataAndMessageValsAtPointDevice(
		const unsigned int xVal, const unsigned int yVal,
		T* dataCostStereoCheckerboard0, T* dataCostStereoCheckerboard1,
		T* messageUDeviceCurrentCheckerboard0, T* messageDDeviceCurrentCheckerboard0,
		T* messageLDeviceCurrentCheckerboard0, T* messageRDeviceCurrentCheckerboard0,
		T* messageUDeviceCurrentCheckerboard1, T* messageDDeviceCurrentCheckerboard1,
		T* messageLDeviceCurrentCheckerboard1, T* messageRDeviceCurrentCheckerboard1,
		const unsigned int widthLevelCheckerboardPart, const unsigned int heightLevel)
{
	if (((xVal + yVal) % 2) == 0) {
		printf("xVal: %d\n", xVal);
		printf("yVal: %d\n", yVal);
		for (unsigned int currentDisparity = 0; currentDisparity < DISP_VALS; currentDisparity++) {
			printf("DISP: %d\n", currentDisparity);
			printf("messageUPrevStereoCheckerboard: %f \n",
					(float) messageUDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageDPrevStereoCheckerboard: %f \n",
					(float) messageDDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageLPrevStereoCheckerboard: %f \n",
					(float) messageLDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageRPrevStereoCheckerboard: %f \n",
					(float) messageRDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("dataCostStereoCheckerboard: %f \n",
					(float) dataCostStereoCheckerboard0[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
		}
	} else {
		printf("xVal: %d\n", xVal);
		printf("yVal: %d\n", yVal);
		for (unsigned int currentDisparity = 0; currentDisparity < DISP_VALS; currentDisparity++) {
			printf("DISP: %d\n", currentDisparity);
			printf("messageUPrevStereoCheckerboard: %f \n",
					(float) messageUDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageDPrevStereoCheckerboard: %f \n",
					(float) messageDDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageLPrevStereoCheckerboard: %f \n",
					(float) messageLDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageRPrevStereoCheckerboard: %f \n",
					(float) messageRDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("dataCostStereoCheckerboard: %f \n",
					(float) dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
		}
	}
}


template<typename T, unsigned int DISP_VALS>
__global__ void printDataAndMessageValsToPointKernel(
		const unsigned int xVal, const unsigned int yVal,
		T* dataCostStereoCheckerboard0, T* dataCostStereoCheckerboard1,
		T* messageUDeviceCurrentCheckerboard0, T* messageDDeviceCurrentCheckerboard0,
		T* messageLDeviceCurrentCheckerboard0, T* messageRDeviceCurrentCheckerboard0,
		T* messageUDeviceCurrentCheckerboard1, T* messageDDeviceCurrentCheckerboard1,
		T* messageLDeviceCurrentCheckerboard1, T* messageRDeviceCurrentCheckerboard1,
		const unsigned int widthLevelCheckerboardPart, const unsigned int heightLevel)
{
	const unsigned int checkerboardAdjustment = (((xVal + yVal) % 2) == 0) ? ((yVal)%2) : ((yVal+1)%2);
	if (((xVal + yVal) % 2) == 0) {
			printf("xVal: %d\n", xVal);
			printf("yVal: %d\n", yVal);
			for (unsigned int currentDisparity = 0; currentDisparity < DISP_VALS; currentDisparity++) {
				printf("DISP: %d\n", currentDisparity);
				printf("messageUPrevStereoCheckerboard: %f \n",
						(float) messageUDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
								xVal / 2, yVal + 1, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, DISP_VALS)]);
				printf("messageDPrevStereoCheckerboard: %f \n",
						(float) messageDDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
								xVal / 2, yVal - 1, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, DISP_VALS)]);
				printf("messageLPrevStereoCheckerboard: %f \n",
						(float) messageLDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
								xVal / 2 + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, DISP_VALS)]);
				printf("messageRPrevStereoCheckerboard: %f \n",
						(float) messageRDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
								(xVal / 2 - 1) + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, DISP_VALS)]);
				printf("dataCostStereoCheckerboard: %f \n",
						(float) dataCostStereoCheckerboard0[retrieveIndexInDataAndMessage(
								xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, DISP_VALS)]);
			}
		} else {
			printf("xVal: %d\n", xVal);
			printf("yVal: %d\n", yVal);
			for (unsigned int currentDisparity = 0; currentDisparity < DISP_VALS; currentDisparity++) {
				printf("DISP: %d\n", currentDisparity);
				printf("messageUPrevStereoCheckerboard: %f \n",
						(float) messageUDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
								xVal / 2, yVal + 1, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, DISP_VALS)]);
				printf("messageDPrevStereoCheckerboard: %f \n",
						(float) messageDDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
								xVal / 2, yVal - 1, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, DISP_VALS)]);
				printf("messageLPrevStereoCheckerboard: %f \n",
						(float) messageLDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
								xVal / 2 + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, DISP_VALS)]);
				printf("messageRPrevStereoCheckerboard: %f \n",
						(float) messageRDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
								(xVal / 2 - 1) + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, DISP_VALS)]);
				printf("dataCostStereoCheckerboard: %f \n",
						(float) dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(
								xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, DISP_VALS)]);
			}
		}
}


template<typename T, unsigned int DISP_VALS>
__device__ void printDataAndMessageValsToPointDevice(
		const unsigned int xVal, const unsigned int yVal,
		T* dataCostStereoCheckerboard0, T* dataCostStereoCheckerboard1,
		T* messageUDeviceCurrentCheckerboard0, T* messageDDeviceCurrentCheckerboard0,
		T* messageLDeviceCurrentCheckerboard0, T* messageRDeviceCurrentCheckerboard0,
		T* messageUDeviceCurrentCheckerboard1, T* messageDDeviceCurrentCheckerboard1,
		T* messageLDeviceCurrentCheckerboard1, T* messageRDeviceCurrentCheckerboard1,
		const unsigned int widthLevelCheckerboardPart, const unsigned int heightLevel)
{
	const unsigned int checkerboardAdjustment = (((xVal + yVal) % 2) == 0) ? ((yVal)%2) : ((yVal+1)%2);

	if (((xVal + yVal) % 2) == 0) {
		printf("xVal: %d\n", xVal);
		printf("yVal: %d\n", yVal);
		for (unsigned int currentDisparity = 0; currentDisparity < DISP_VALS; currentDisparity++) {
			printf("DISP: %d\n", currentDisparity);
			printf("messageUPrevStereoCheckerboard: %f \n",
					(float) messageUDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal + 1, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageDPrevStereoCheckerboard: %f \n",
					(float) messageDDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal - 1, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageLPrevStereoCheckerboard: %f \n",
					(float) messageLDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2 + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageRPrevStereoCheckerboard: %f \n",
					(float) messageRDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							(xVal / 2 - 1) + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("dataCostStereoCheckerboard: %f \n",
					(float) dataCostStereoCheckerboard0[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
		}
	} else {
		printf("xVal: %d\n", xVal);
		printf("yVal: %d\n", yVal);
		for (unsigned int currentDisparity = 0; currentDisparity < DISP_VALS; currentDisparity++) {
			printf("DISP: %d\n", currentDisparity);
			printf("messageUPrevStereoCheckerboard: %f \n",
					(float) messageUDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
							xVal / 2, yVal + 1, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageDPrevStereoCheckerboard: %f \n",
					(float) messageDDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
							xVal / 2, yVal - 1, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageLPrevStereoCheckerboard: %f \n",
					(float) messageLDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
							xVal / 2 + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("messageRPrevStereoCheckerboard: %f \n",
					(float) messageRDeviceCurrentCheckerboard0[retrieveIndexInDataAndMessage(
							(xVal / 2 - 1) + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
			printf("dataCostStereoCheckerboard: %f \n",
					(float) dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, DISP_VALS)]);
		}
	}
}


/*template<>
__device__ half2 getZeroVal<half2>()
{
	return __floats2half2_rn (0.0, 0.0);
}


__device__ half2 getMinBothPartsHalf2(half2 val1, half2 val2)
{
	half2 val1Less = __hlt2(val1, val2);
	half2 val2LessOrEqual = __hle2(val2, val1);
	return __hadd2(__hmul2(val1Less, val1), __hmul2(val2LessOrEqual, val2));
}

template<>
__device__ void dtStereo<half2>(half2 f[bp_params::NUM_POSSIBLE_DISPARITY_VALUES])
{
	half2 prev;
	for (int currentDisparity = 1; currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
	{
		prev = __hadd2(f[currentDisparity-1], __float2half2_rn(1.0f));
		f[currentDisparity] = getMinBothPartsHalf2(prev, f[currentDisparity]);
	}

	for (int currentDisparity = bp_params::NUM_POSSIBLE_DISPARITY_VALUES-2; currentDisparity >= 0; currentDisparity--)
	{
		prev = __hadd2(f[currentDisparity+1], __float2half2_rn(1.0f));
		f[currentDisparity] = getMinBothPartsHalf2(prev, f[currentDisparity]);
	}
}*/


/*template<>
__device__ void msgStereo<half2>(half2 messageValsNeighbor1[bp_params::NUM_POSSIBLE_DISPARITY_VALUES], half2 messageValsNeighbor2[bp_params::NUM_POSSIBLE_DISPARITY_VALUES],
		half2 messageValsNeighbor3[bp_params::NUM_POSSIBLE_DISPARITY_VALUES], half2 dataCosts[bp_params::NUM_POSSIBLE_DISPARITY_VALUES],
		half2 dst[bp_params::NUM_POSSIBLE_DISPARITY_VALUES], half2 disc_k_bp)
{
	// aggregate and find min
	half2 minimum = __float2half2_rn(INF_BP);

	for (int currentDisparity = 0; currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
	{
		dst[currentDisparity] = __hadd2(messageValsNeighbor1[currentDisparity], messageValsNeighbor2[currentDisparity]);
		dst[currentDisparity] = __hadd2(dst[currentDisparity], messageValsNeighbor3[currentDisparity]);
		dst[currentDisparity] = __hadd2(dst[currentDisparity], dataCosts[currentDisparity]);

		minimum = getMinBothPartsHalf2(dst[currentDisparity], minimum);
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
	dtStereo<half2>(dst);

	// truncate
	minimum = __hadd2(minimum, disc_k_bp);

	// normalize
	half2 valToNormalize = __float2half2_rn(0.0f);

	for (int currentDisparity = 0; currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
	{
		dst[currentDisparity] = getMinBothPartsHalf2(minimum, dst[currentDisparity]);
		valToNormalize = __hadd2(valToNormalize, dst[currentDisparity]);
	}

	//if either valToNormalize in half2 is infinite or NaN, set destination vector to 0 for all disparities
	//note that may cause results to differ a little from ideal
	if (((__hisnan(__low2half(valToNormalize)))
			|| ((__hisinf(__low2half(valToNormalize)) != 0)))
			|| ((__hisnan(__high2half(valToNormalize)))
					|| ((__hisinf(__high2half(valToNormalize)) != 0))))
	{
		for (int currentDisparity = 0;
				currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = __floats2half2_rn(0.0f, 0.0f);
		}
	}
	else
	{
		valToNormalize = __h2div(valToNormalize,
				__float2half2_rn((float) bp_params::NUM_POSSIBLE_DISPARITY_VALUES));

		for (int currentDisparity = 0;
				currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = __hsub2(dst[currentDisparity],
					valToNormalize);
		}
	}
	//check if both values in half2 are inf or nan
	/*if (((__hisnan(__low2half(valToNormalize)))
			|| ((__hisinf(__low2half(valToNormalize)) != 0)))
			&& ((__hisnan(__high2half(valToNormalize)))
					|| ((__hisinf(__high2half(valToNormalize)) != 0))))
	{
		for (int currentDisparity = 0;
				currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = __floats2half2_rn(0.0f, 0.0f);
		}
	}
	else if (((__hisnan(__low2half(valToNormalize)))
			|| ((__hisinf(__low2half(valToNormalize)) != 0))))
	{
		//lower half of half2 is inf or nan
		valToNormalize = __h2div(valToNormalize,
				__float2half2_rn((float) bp_params::NUM_POSSIBLE_DISPARITY_VALUES));

		for (int currentDisparity = 0;
				currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = __hsub2(dst[currentDisparity],
					valToNormalize);
		}

		for (int currentDisparity = 0;
				currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = __halves2half2((half)0.0f,
					__high2half(dst[currentDisparity]));
		}
	}
	else if ((__hisnan(__high2half(valToNormalize)))
			|| ((__hisinf(__high2half(valToNormalize)) != 0)))
	{
		//higher half of half2 is inf or nan
		valToNormalize = __h2div(valToNormalize,
				__float2half2_rn((float) bp_params::NUM_POSSIBLE_DISPARITY_VALUES));

		for (int currentDisparity = 0;
				currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = __hsub2(dst[currentDisparity],
					valToNormalize);
		}

		for (int currentDisparity = 0;
				currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = __halves2half2(
					__low2half(dst[currentDisparity]), (half)0.0f);
		}
	}
}*/


//device portion of the kernal function to run the current iteration of belief propagation in parallel using the checkerboard update method where half the pixels in the
//"checkerboard" scheme retrieve messages from each 4-connected neighbor and then update their message based on the retrieved messages and the data cost
//this function uses local memory to store the message and data values at each disparity in the intermediate step of current message computation
//this function uses linear memory bound to textures to access the current data and message values
/*template<>
__device__ void runBPIterationUsingCheckerboardUpdatesDeviceNoTexBoundAndLocalMem<half2>(int xVal, int yVal,
		int checkerboardToUpdate, levelProperties& currentLevelProperties, half2* dataCostStereoCheckerboard1, half2* dataCostStereoCheckerboard2,
		half2* messageUDeviceCurrentCheckerboard1,
		half2* messageDDeviceCurrentCheckerboard1,
		half2* messageLDeviceCurrentCheckerboard1,
		half2* messageRDeviceCurrentCheckerboard1,
		half2* messageUDeviceCurrentCheckerboard2,
		half2* messageDDeviceCurrentCheckerboard2,
		half2* messageLDeviceCurrentCheckerboard2,
		half2* messageRDeviceCurrentCheckerboard2,
		float disc_k_bp, int offsetData)
{
}

	int indexWriteTo;
	int checkerboardAdjustment;

	//checkerboardAdjustment used for indexing into current checkerboard to update
	if (checkerboardToUpdate == CHECKERBOARD_PART_0)
	{
		checkerboardAdjustment = ((yVal)%2);
	}
	else //checkerboardToUpdate == CHECKERBOARD_PART_1
	{
		checkerboardAdjustment = ((yVal+1)%2);
	}

	//may want to look into (xVal < (widthLevelCheckerboardPart - 1) since it may affect the edges
	//make sure that the current point is not an edge/corner that doesn't have four neighbors that can pass values to it
	//if ((xVal >= (1 - checkerboardAdjustment)) && (xVal < (widthLevelCheckerboardPart - 1)) && (yVal > 0) && (yVal < (heightLevel - 1)))
	if ((xVal >= (1/*switch to 0 if trying to match half results exactly*//* - checkerboardAdjustment)) && (xVal < (widthLevelCheckerboardPart - checkerboardAdjustment)) && (yVal > 0) && (yVal < (heightLevel - 1)))
	{
		half2 prevUMessage[bp_params::NUM_POSSIBLE_DISPARITY_VALUES];
		half2 prevDMessage[bp_params::NUM_POSSIBLE_DISPARITY_VALUES];
		half2 prevLMessage[bp_params::NUM_POSSIBLE_DISPARITY_VALUES];
		half2 prevRMessage[bp_params::NUM_POSSIBLE_DISPARITY_VALUES];

		half2 dataMessage[bp_params::NUM_POSSIBLE_DISPARITY_VALUES];

		if (checkerboardToUpdate == CHECKERBOARD_PART_0)
		{
			half* messageLDeviceCurrentCheckerboard2Half = (half*)messageLDeviceCurrentCheckerboard2;
			half* messageRDeviceCurrentCheckerboard2Half = (half*)messageRDeviceCurrentCheckerboard2;

			for (int currentDisparity = 0;
					currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES;
					currentDisparity++)
			{
				dataMessage[currentDisparity] =
						dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(
								xVal, yVal, widthLevelCheckerboardPart,
								heightLevel, currentDisparity,
								bp_params::NUM_POSSIBLE_DISPARITY_VALUES, offsetData)];
				prevUMessage[currentDisparity] =
						messageUDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
								xVal, (yVal + 1), widthLevelCheckerboardPart,
								heightLevel, currentDisparity,
								bp_params::NUM_POSSIBLE_DISPARITY_VALUES)];
				prevDMessage[currentDisparity] =
						messageDDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
								xVal, (yVal - 1), widthLevelCheckerboardPart,
								heightLevel, currentDisparity,
								bp_params::NUM_POSSIBLE_DISPARITY_VALUES)];
				prevLMessage[currentDisparity] =
						__halves2half2(
								messageLDeviceCurrentCheckerboard2Half[retrieveIndexInDataAndMessage(
										((xVal * 2) + checkerboardAdjustment),
										yVal, widthLevelCheckerboardPart * 2,
										heightLevel, currentDisparity,
										bp_params::NUM_POSSIBLE_DISPARITY_VALUES)],
								messageLDeviceCurrentCheckerboard2Half[retrieveIndexInDataAndMessage(
										((xVal * 2 + 1) + checkerboardAdjustment),
										yVal, widthLevelCheckerboardPart * 2,
										heightLevel, currentDisparity,
										bp_params::NUM_POSSIBLE_DISPARITY_VALUES)]);

				//if ((((xVal * 2) - 1) + checkerboardAdjustment) >= 0)
				{
					prevRMessage[currentDisparity] =
							__halves2half2(
									messageRDeviceCurrentCheckerboard2Half[retrieveIndexInDataAndMessage(
											(((xVal * 2) - 1)
													+ checkerboardAdjustment),
											yVal,
											widthLevelCheckerboardPart * 2,
											heightLevel, currentDisparity,
											bp_params::NUM_POSSIBLE_DISPARITY_VALUES)],
									messageRDeviceCurrentCheckerboard2Half[retrieveIndexInDataAndMessage(
											(((xVal * 2 + 1) - 1)
													+ checkerboardAdjustment),
											yVal,
											widthLevelCheckerboardPart * 2,
											heightLevel, currentDisparity,
											bp_params::NUM_POSSIBLE_DISPARITY_VALUES)]);
				}
				/*else
				{
					prevRMessage[currentDisparity] =
							__halves2half2((half)0.0f,
									messageRDeviceCurrentCheckerboard2Half[retrieveIndexInDataAndMessage(
											(((xVal * 2 + 1) - 1)
													+ checkerboardAdjustment),
											yVal,
											widthLevelCheckerboardPart * 2,
											heightLevel, currentDisparity,
											bp_params::NUM_POSSIBLE_DISPARITY_VALUES)]);
				}*//*
			}
		}
		else //checkerboardToUpdate == CHECKERBOARD_PART_1
		{
			half* messageLDeviceCurrentCheckerboard1Half = (half*)messageLDeviceCurrentCheckerboard1;
			half* messageRDeviceCurrentCheckerboard1Half = (half*)messageRDeviceCurrentCheckerboard1;

			for (int currentDisparity = 0;
					currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES;
					currentDisparity++)
			{
				dataMessage[currentDisparity] =
						dataCostStereoCheckerboard2[retrieveIndexInDataAndMessage(
								xVal, yVal, widthLevelCheckerboardPart,
								heightLevel, currentDisparity,
								bp_params::NUM_POSSIBLE_DISPARITY_VALUES, offsetData)];
				prevUMessage[currentDisparity] =
						messageUDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
								xVal, (yVal + 1), widthLevelCheckerboardPart,
								heightLevel, currentDisparity,
								bp_params::NUM_POSSIBLE_DISPARITY_VALUES)];
				prevDMessage[currentDisparity] =
						messageDDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
								xVal, (yVal - 1), widthLevelCheckerboardPart,
								heightLevel, currentDisparity,
								bp_params::NUM_POSSIBLE_DISPARITY_VALUES)];
				prevLMessage[currentDisparity] =
						__halves2half2(
								messageLDeviceCurrentCheckerboard1Half[retrieveIndexInDataAndMessage(
										((xVal * 2)
												+ checkerboardAdjustment),
										yVal, widthLevelCheckerboardPart * 2,
										heightLevel, currentDisparity,
										bp_params::NUM_POSSIBLE_DISPARITY_VALUES)],
								messageLDeviceCurrentCheckerboard1Half[retrieveIndexInDataAndMessage(
										((xVal * 2 + 1)
												+ checkerboardAdjustment),
										yVal, widthLevelCheckerboardPart * 2,
										heightLevel, currentDisparity,
										bp_params::NUM_POSSIBLE_DISPARITY_VALUES)]);

				//if ((((xVal * 2) - 1) + checkerboardAdjustment) >= 0)
				{
					prevRMessage[currentDisparity] =
							__halves2half2(
									messageRDeviceCurrentCheckerboard1Half[retrieveIndexInDataAndMessage(
											(((xVal * 2) - 1)
													+ checkerboardAdjustment),
											yVal,
											widthLevelCheckerboardPart * 2,
											heightLevel, currentDisparity,
											bp_params::NUM_POSSIBLE_DISPARITY_VALUES)],
									messageRDeviceCurrentCheckerboard1Half[retrieveIndexInDataAndMessage(
											(((xVal * 2 + 1) - 1)
													+ checkerboardAdjustment),
											yVal,
											widthLevelCheckerboardPart * 2,
											heightLevel, currentDisparity,
											bp_params::NUM_POSSIBLE_DISPARITY_VALUES)]);
				}
				/*else
				{
					prevRMessage[currentDisparity] =
							__halves2half2((half) 0.0,
									messageRDeviceCurrentCheckerboard1Half[retrieveIndexInDataAndMessage(
											(((xVal * 2 + 1) - 1)
													+ checkerboardAdjustment),
											yVal,
											widthLevelCheckerboardPart * 2,
											heightLevel, currentDisparity,
											bp_params::NUM_POSSIBLE_DISPARITY_VALUES)]);
				}*//*
			}
		}

		half2 currentUMessage[bp_params::NUM_POSSIBLE_DISPARITY_VALUES];
		half2 currentDMessage[bp_params::NUM_POSSIBLE_DISPARITY_VALUES];
		half2 currentLMessage[bp_params::NUM_POSSIBLE_DISPARITY_VALUES];
		half2 currentRMessage[bp_params::NUM_POSSIBLE_DISPARITY_VALUES];

		//uses the previous message values and data cost to calculate the current message values and store the results
		runBPIterationInOutDataInLocalMem<half2>(prevUMessage, prevDMessage, prevLMessage, prevRMessage, dataMessage,
							currentUMessage, currentDMessage, currentLMessage, currentRMessage, __float2half2_rn(disc_k_bp));

		//write the calculated message values to global memory
		for (int currentDisparity = 0; currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
		{
			indexWriteTo = retrieveIndexInDataAndMessage(xVal, yVal, widthLevelCheckerboardPart, heightLevel, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES);
			if (checkerboardToUpdate == CHECKERBOARD_PART_0)
			{
				messageUDeviceCurrentCheckerboard1[indexWriteTo] = currentUMessage[currentDisparity];
				messageDDeviceCurrentCheckerboard1[indexWriteTo] = currentDMessage[currentDisparity];
				messageLDeviceCurrentCheckerboard1[indexWriteTo] = currentLMessage[currentDisparity];
				messageRDeviceCurrentCheckerboard1[indexWriteTo] = currentRMessage[currentDisparity];
			}
			else //checkerboardToUpdate == CHECKERBOARD_PART_1
			{
				messageUDeviceCurrentCheckerboard2[indexWriteTo] = currentUMessage[currentDisparity];
				messageDDeviceCurrentCheckerboard2[indexWriteTo] = currentDMessage[currentDisparity];
				messageLDeviceCurrentCheckerboard2[indexWriteTo] = currentLMessage[currentDisparity];
				messageRDeviceCurrentCheckerboard2[indexWriteTo] = currentRMessage[currentDisparity];
			}
		}
	}
}
*/

//retrieve the best disparity estimate from image 1 to image 2 for each pixel in parallel
/*template<>
__global__ void retrieveOutputDisparityCheckerboardStereoOptimized<half2>(levelProperties currentLevelProperties, half2* dataCostStereoCheckerboard1, half2* dataCostStereoCheckerboard2, half2* messageUPrevStereoCheckerboard1, half2* messageDPrevStereoCheckerboard1, half2* messageLPrevStereoCheckerboard1, half2* messageRPrevStereoCheckerboard1, half2* messageUPrevStereoCheckerboard2, half2* messageDPrevStereoCheckerboard2, half2* messageLPrevStereoCheckerboard2, half2* messageRPrevStereoCheckerboard2, float* disparityBetweenImagesDevice)
{

}*/

//retrieve the best disparity estimate from image 1 to image 2 for each pixel in parallel
/*template<typename T>
__global__ void retrieveOutputDisparityCheckerboardStereo(T* dataCostStereoCheckerboard1, T* dataCostStereoCheckerboard2, T* messageUPrevStereoCheckerboard1, T* messageDPrevStereoCheckerboard1, T* messageLPrevStereoCheckerboard1, T* messageRPrevStereoCheckerboard1, T* messageUPrevStereoCheckerboard2, T* messageDPrevStereoCheckerboard2, T* messageLPrevStereoCheckerboard2, T* messageRPrevStereoCheckerboard2, float* disparityBetweenImagesDevice, int widthLevel, int heightLevel)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_BP + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_BP + ty;

	if (withinImageBounds(xVal, yVal, widthLevel, heightLevel))
	{
		int widthCheckerboard = getCheckerboardWidth<T>(widthLevel);
		int xValInCheckerboardPart = xVal/2;

		if (((yVal+xVal) % 2) == 0) //if true, then pixel is from part 1 of the checkerboard; otherwise, it's from part 2
		{
			int	checkerboardPartAdjustment = (yVal%2);

			if ((xVal >= 1) && (xVal < (widthLevel - 1)) && (yVal >= 1) && (yVal < (heightLevel - 1)))
			{
				// keep track of "best" disparity for current pixel
				int bestDisparity = 0;
				T best_val = INF_BP;
				for (int currentDisparity = 0; currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
				{
					T val = messageUPrevStereoCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal + 1), widthCheckerboard, heightLevel, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES)] +
						 messageDPrevStereoCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal - 1), widthCheckerboard, heightLevel, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES)] +
						 messageLPrevStereoCheckerboard2[retrieveIndexInDataAndMessage((xValInCheckerboardPart + checkerboardPartAdjustment), yVal, widthCheckerboard, heightLevel, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES)] +
						 messageRPrevStereoCheckerboard2[retrieveIndexInDataAndMessage((xValInCheckerboardPart - 1 + checkerboardPartAdjustment), yVal, widthCheckerboard, heightLevel, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES)] +
						 dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboardPart, yVal, widthCheckerboard, heightLevel, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES)];

					if (val < (best_val)) {
						best_val = val;
						bestDisparity = currentDisparity;
					}
				}
				disparityBetweenImagesDevice[yVal*widthLevel + xVal] = bestDisparity;
			}
			else
			{
				disparityBetweenImagesDevice[yVal*widthLevel + xVal] = 0;
			}
		}
		else //pixel from part 2 of checkerboard
		{
			int	checkerboardPartAdjustment = ((yVal + 1) % 2);

			if ((xVal >= 1) && (xVal < (widthLevel - 1)) && (yVal >= 1) && (yVal < (heightLevel - 1)))
			{


				// keep track of "best" disparity for current pixel
				int bestDisparity = 0;
				T best_val = INF_BP;
				for (int currentDisparity = 0; currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
				{
					T val = messageUPrevStereoCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal + 1), widthCheckerboard, heightLevel, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES)] +
						messageDPrevStereoCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal - 1), widthCheckerboard, heightLevel, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES)] +
						messageLPrevStereoCheckerboard1[retrieveIndexInDataAndMessage((xValInCheckerboardPart + checkerboardPartAdjustment), yVal, widthCheckerboard, heightLevel, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES)] +
						messageRPrevStereoCheckerboard1[retrieveIndexInDataAndMessage((xValInCheckerboardPart - 1 + checkerboardPartAdjustment), yVal, widthCheckerboard, heightLevel, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES)] +
						dataCostStereoCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboardPart, yVal, widthCheckerboard, heightLevel, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES)];

					if (val < (best_val))
					{
						best_val = val;
						bestDisparity = currentDisparity;
					}
				}
				disparityBetweenImagesDevice[yVal*widthLevel + xVal] = bestDisparity;
			}
			else
			{
				disparityBetweenImagesDevice[yVal*widthLevel + xVal] = 0;
			}
		}
	}
}

//retrieve the best disparity estimate from image 1 to image 2 for each pixel in parallel
template<>
__global__ void retrieveOutputDisparityCheckerboardStereo<half2>(half2* dataCostStereoCheckerboard1, half2* dataCostStereoCheckerboard2, half2* messageUPrevStereoCheckerboard1, half2* messageDPrevStereoCheckerboard1, half2* messageLPrevStereoCheckerboard1, half2* messageRPrevStereoCheckerboard1, half2* messageUPrevStereoCheckerboard2, half2* messageDPrevStereoCheckerboard2, half2* messageLPrevStereoCheckerboard2, half2* messageRPrevStereoCheckerboard2, float* disparityBetweenImagesDevice, int widthLevel, int heightLevel)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_BP + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_BP + ty;

	if (withinImageBounds(xVal*2, yVal, widthLevel, heightLevel))
	{
		int widthCheckerboard = getCheckerboardWidth<half2>(widthLevel);
		int xValInCheckerboardPart = xVal/2;

		if (((yVal+xVal) % 2) == 0) //if true, then pixel is from part 1 of the checkerboard; otherwise, it's from part 2
		{
			int	checkerboardPartAdjustment = (yVal%2);

			half* messageLPrevStereoCheckerboard2Half = (half*)messageLPrevStereoCheckerboard2;
			half* messageRPrevStereoCheckerboard2Half = (half*)messageRPrevStereoCheckerboard2;

			if ((xVal >= 1) && (xVal < (widthLevel - 1)) && (yVal >= 1) && (yVal < (heightLevel - 1)))
			{
				// keep track of "best" disparity for current pixel
				int bestDisparity1 = 0;
				int bestDisparity2 = 0;
				float best_val1 = INF_BP;
				float best_val2 = INF_BP;
				for (int currentDisparity = 0; currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
				{
					half2 val = __hadd2(messageUPrevStereoCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal + 1), widthCheckerboard, heightLevel, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES)],
											 messageDPrevStereoCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal - 1), widthCheckerboard, heightLevel, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES)]);
					val =
							__hadd2(val,
									__halves2half2(
											messageLPrevStereoCheckerboard2Half[retrieveIndexInDataAndMessage(
													((xValInCheckerboardPart * 2)
															+ checkerboardPartAdjustment),
													yVal, widthCheckerboard * 2,
													heightLevel,
													currentDisparity,
													bp_params::NUM_POSSIBLE_DISPARITY_VALUES)],
									messageLPrevStereoCheckerboard2Half[retrieveIndexInDataAndMessage(
											((xValInCheckerboardPart * 2 + 1)
													+ checkerboardPartAdjustment),
											yVal, widthCheckerboard * 2,
											heightLevel, currentDisparity,
											bp_params::NUM_POSSIBLE_DISPARITY_VALUES)]));
					val =
							__hadd2(val,
									__halves2half2(
											messageRPrevStereoCheckerboard2Half[retrieveIndexInDataAndMessage(
													((xValInCheckerboardPart * 2)
															- 1
															+ checkerboardPartAdjustment),
													yVal, widthCheckerboard * 2,
													heightLevel,
													currentDisparity,
													bp_params::NUM_POSSIBLE_DISPARITY_VALUES)],
									messageRPrevStereoCheckerboard2Half[retrieveIndexInDataAndMessage(
											((xValInCheckerboardPart * 2 + 1)
													- 1
													+ checkerboardPartAdjustment),
											yVal, widthCheckerboard * 2,
											heightLevel, currentDisparity,
											bp_params::NUM_POSSIBLE_DISPARITY_VALUES)]));
					val = __hadd2(val, dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboardPart, yVal, widthCheckerboard, heightLevel, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES)]);

					float valLow = __low2float ( val);
					float valHigh = __high2float ( val);
					if (valLow < best_val1)
					{
						best_val1 = valLow;
						bestDisparity1 = currentDisparity;
					}
					if (valHigh < best_val2)
					{
						best_val2 = valHigh;
						bestDisparity2 = currentDisparity;
					}
				}
				disparityBetweenImagesDevice[yVal*widthLevel + (xVal*2 - checkerboardPartAdjustment)] = bestDisparity1;
				if (((xVal*2) + 2) < widthLevel)
				{
					disparityBetweenImagesDevice[yVal*widthLevel + (xVal*2 - checkerboardPartAdjustment) + 2] = bestDisparity2;
				}
			}
			else
			{
				disparityBetweenImagesDevice[yVal * widthLevel + (xVal * 2 - checkerboardPartAdjustment)] =
						0;
				if (((xVal * 2) + 2) < widthLevel)
				{
					disparityBetweenImagesDevice[yVal * widthLevel + (xVal * 2 - checkerboardPartAdjustment)
							+ 2] = 0;
				}
			}
		}
		else //pixel from part 2 of checkerboard
		{
			int	checkerboardPartAdjustment = ((yVal + 1) % 2);
			half* messageLPrevStereoCheckerboard1Half = (half*)messageLPrevStereoCheckerboard1;
			half* messageRPrevStereoCheckerboard1Half = (half*)messageRPrevStereoCheckerboard1;

			if ((xVal >= 1) && (xVal < (widthLevel - 1)) && (yVal >= 1) && (yVal < (heightLevel - 1)))
			{
				// keep track of "best" disparity for current pixel
				int bestDisparity1 = 0;
				int bestDisparity2 = 0;
				float best_val1 = INF_BP;
				float best_val2 = INF_BP;
				for (int currentDisparity = 0; currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
				{
					half2 val =
							__hadd2(
									messageUPrevStereoCheckerboard1[retrieveIndexInDataAndMessage(
											xValInCheckerboardPart, (yVal + 1),
											widthCheckerboard, heightLevel,
											currentDisparity,
											bp_params::NUM_POSSIBLE_DISPARITY_VALUES)],
											messageDPrevStereoCheckerboard1[retrieveIndexInDataAndMessage(
													xValInCheckerboardPart,
													(yVal - 1),
													widthCheckerboard,
													heightLevel,
													currentDisparity,
													bp_params::NUM_POSSIBLE_DISPARITY_VALUES)]);
					val =
							__hadd2(val,
									__halves2half2(
											messageLPrevStereoCheckerboard1Half[retrieveIndexInDataAndMessage(
													((xValInCheckerboardPart * 2)
															+ checkerboardPartAdjustment),
													yVal, widthCheckerboard * 2,
													heightLevel,
													currentDisparity,
													bp_params::NUM_POSSIBLE_DISPARITY_VALUES)],
									messageLPrevStereoCheckerboard1Half[retrieveIndexInDataAndMessage(
											((xValInCheckerboardPart * 2 + 1)
													+ checkerboardPartAdjustment),
											yVal, widthCheckerboard * 2,
											heightLevel, currentDisparity,
											bp_params::NUM_POSSIBLE_DISPARITY_VALUES)]));
					val =
							__hadd2(val,
									__halves2half2(
											messageRPrevStereoCheckerboard1Half[retrieveIndexInDataAndMessage(
													((xValInCheckerboardPart * 2)
															- 1
															+ checkerboardPartAdjustment),
													yVal, widthCheckerboard * 2,
													heightLevel,
													currentDisparity,
													bp_params::NUM_POSSIBLE_DISPARITY_VALUES)],
									messageRPrevStereoCheckerboard1Half[retrieveIndexInDataAndMessage(
											((xValInCheckerboardPart * 2 + 1)
													- 1
													+ checkerboardPartAdjustment),
											yVal, widthCheckerboard * 2,
											heightLevel, currentDisparity,
											bp_params::NUM_POSSIBLE_DISPARITY_VALUES)]));

					val =
							__hadd2(val,
									dataCostStereoCheckerboard2[retrieveIndexInDataAndMessage(
											xValInCheckerboardPart, yVal,
											widthCheckerboard, heightLevel,
											currentDisparity,
											bp_params::NUM_POSSIBLE_DISPARITY_VALUES)]);

					float val1 = __low2float(val);
					float val2 = __high2float(val);
					if (val1 < best_val1) {
						best_val1 = val1;
						bestDisparity1 = currentDisparity;
					}
					if (val2 < best_val2) {
						best_val2 = val2;
						bestDisparity2 = currentDisparity;
					}
				}

				disparityBetweenImagesDevice[yVal * widthLevel + (xVal * 2 - checkerboardPartAdjustment)] =
						bestDisparity1;
				if (((xVal * 2) + 2) < widthLevel) {
					disparityBetweenImagesDevice[yVal * widthLevel + (xVal * 2 - checkerboardPartAdjustment)
							+ 2] = bestDisparity2;
				}
			}
			else
			{
				disparityBetweenImagesDevice[yVal * widthLevel + (xVal * 2 - checkerboardPartAdjustment)] =
						0;
				if (((xVal * 2) + 2) < widthLevel) {
					disparityBetweenImagesDevice[yVal * widthLevel + (xVal * 2 - checkerboardPartAdjustment)
							+ 2] = 0;
				}
			}
		}
	}
}
*/

/*template<>
__global__ void initializeBottomLevelDataStereo<half2>(levelProperties currentLevelProperties, float* image1PixelsDevice, float* image2PixelsDevice, half2* dataCostDeviceStereoCheckerboard1, half2* dataCostDeviceStereoCheckerboard2, float lambda_bp, float data_k_bp)
{
	// Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_BP + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_BP + ty;

	int indexVal;
	int imageCheckerboardWidth = getCheckerboardWidth<half2>(widthImages);
	int xInCheckerboard = xVal / 2;

	if (withinImageBounds(xInCheckerboard, yVal, imageCheckerboardWidth, heightImages))
	{
		int imageXPixelIndexStart = 0;
		int checkerboardNum = 1;

		//check which checkerboard data values for and make necessary adjustment to start
		if (((yVal) % 2) == 0) {
			if (((xVal) % 2) == 0) {
				checkerboardNum = 1;
			} else {
				checkerboardNum = 2;
			}
		} else {
			if (((xVal) % 2) == 0) {
				checkerboardNum = 2;
			} else {
				checkerboardNum = 1;
			}
		}

		imageXPixelIndexStart = xVal*2;
		if ((((yVal) % 2) == 0) && (checkerboardNum == 2)) {
			imageXPixelIndexStart -= 1;
		}
		if ((((yVal) % 2) == 1) && (checkerboardNum == 1)) {
			imageXPixelIndexStart -= 1;
		}

		//make sure that it is possible to check every disparity value
		if ((((imageXPixelIndexStart + 2) - (bp_params::NUM_POSSIBLE_DISPARITY_VALUES-1)) >= 0))
		{
			for (int currentDisparity = 0; currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
			{
				float currentPixelImage1_low = 0.0;
				float currentPixelImage2_low = 0.0;

				if ((((imageXPixelIndexStart) - (bp_params::NUM_POSSIBLE_DISPARITY_VALUES-1)) >= 0))
				{
					if (withinImageBounds(imageXPixelIndexStart, yVal, widthImages,
							heightImages)) {
						currentPixelImage1_low = image1PixelsDevice[yVal
								* widthImages + imageXPixelIndexStart];
						currentPixelImage2_low = image2PixelsDevice[yVal
								* widthImages + (imageXPixelIndexStart - currentDisparity)];
					}
				}

				float currentPixelImage1_high = 0.0;
				float currentPixelImage2_high = 0.0;

				if (withinImageBounds(imageXPixelIndexStart + 2, yVal, widthImages,
						heightImages))
				{
					currentPixelImage1_high = image1PixelsDevice[yVal * widthImages
							+ (imageXPixelIndexStart + 2)];
					currentPixelImage2_high = image2PixelsDevice[yVal * widthImages
							+ ((imageXPixelIndexStart + 2) - currentDisparity)];
				}

				indexVal = retrieveIndexInDataAndMessage(xInCheckerboard, yVal, imageCheckerboardWidth, heightImages, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES);

				half lowVal = (half)(lambda_bp * min(abs(currentPixelImage1_low - currentPixelImage2_low), data_k_bp));
				half highVal = (half)(lambda_bp * min(abs(currentPixelImage1_high - currentPixelImage2_high), data_k_bp));

				//data cost is equal to dataWeight value for weighting times the absolute difference in corresponding pixel intensity values capped at dataCostCap
				if (checkerboardNum == 1)
				{
					dataCostDeviceStereoCheckerboard1[indexVal] = __halves2half2(lowVal, highVal);
				}
				else
				{
					dataCostDeviceStereoCheckerboard2[indexVal] = __halves2half2(lowVal, highVal);
				}
			}
		}
		else
		{
			for (int currentDisparity = 0; currentDisparity < bp_params::NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
			{
				indexVal = retrieveIndexInDataAndMessage(xInCheckerboard, yVal, imageCheckerboardWidth, heightImages, currentDisparity, bp_params::NUM_POSSIBLE_DISPARITY_VALUES);

				//data cost is equal to dataWeight value for weighting times the absolute difference in corresponding pixel intensity values capped at dataCostCap
				if (((xVal + yVal) % 2) == 0)
				{
					dataCostDeviceStereoCheckerboard1[indexVal] = getZeroVal<half2>();
				}
				else
				{
					dataCostDeviceStereoCheckerboard2[indexVal] = getZeroVal<half2>();
				}
			}
		}
	}
}*/

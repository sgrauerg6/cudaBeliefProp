#include "hip/hip_runtime.h"
/*
Copyright (C) 2009 Scott Grauer-Gray, Chandra Kambhamettu, and Kannappan Palaniappan

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307 USA
*/

//This file defines the methods to perform belief propagation for disparity map estimation from stereo images on CUDA


#include "kernalBpStereoHeader.cuh"

//checks if the current point is within the image bounds
__device__ bool withinImageBounds(int xVal, int yVal, int width, int height)
{
	return ((xVal >= 0) && (xVal < width) && (yVal >= 0) && (yVal < height));
}


//retrieve the current 1-D index value of the given point at the given disparity in the data cost and message data
__device__ int retrieveIndexInDataAndMessage(int xVal, int yVal, int width, int height, int currentDisparity, int totalNumDispVals, int offsetData)
{
	return RETRIEVE_INDEX_IN_DATA_OR_MESSAGE_ARRAY_EQUATION + offsetData;
}

template<typename T>
__device__ __host__ int getCheckerboardWidth(int imageWidth)
{
	return (int)ceil(((float)imageWidth) / 2.0);
}

//checkerboard width is half size when using half2 data type
template <>
__device__ __host__ int getCheckerboardWidth<half2>(int imageWidth)
{
	return (int)ceil(((ceil(((float)imageWidth) / 2.0)) / 2.0));
}

template <>
__device__ __host__ int getCheckerboardWidth<half>(int imageWidth)
{
	return (getCheckerboardWidth<half2>(imageWidth)) * 2;
}

template<typename T>
__device__ T getZeroVal()
{
	return (T)0.0;
}

template<>
__device__ half2 getZeroVal<half2>()
{
	return __floats2half2_rn (0.0, 0.0);
}

//function retrieve the minimum value at each 1-d disparity value in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
template<typename T>
__device__ void dtStereo(T f[NUM_POSSIBLE_DISPARITY_VALUES])
{
	T prev;
	for (int currentDisparity = 1; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
	{
		prev = f[currentDisparity-1] + (T)1.0;
		if (prev < f[currentDisparity])
			f[currentDisparity] = prev;
	}

	for (int currentDisparity = NUM_POSSIBLE_DISPARITY_VALUES-2; currentDisparity >= 0; currentDisparity--)
	{
		prev = f[currentDisparity+1] + (T)1.0;
		if (prev < f[currentDisparity])
			f[currentDisparity] = prev;
	}
}

__device__ half2 getMinBothPartsHalf2(half2 val1, half2 val2)
{
	half2 val1Less = __hlt2(val1, val2);
	half2 val2LessOrEqual = __hle2(val2, val1);
	return __hadd2(__hmul2(val1Less, val1), __hmul2(val2LessOrEqual, val2));
}

template<>
__device__ void dtStereo<half2>(half2 f[NUM_POSSIBLE_DISPARITY_VALUES])
{
	half2 prev;
	for (int currentDisparity = 1; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
	{
		prev = __hadd2(f[currentDisparity-1], __float2half2_rn(1.0f));
		f[currentDisparity] = getMinBothPartsHalf2(prev, f[currentDisparity]);
	}

	for (int currentDisparity = NUM_POSSIBLE_DISPARITY_VALUES-2; currentDisparity >= 0; currentDisparity--)
	{
		prev = __hadd2(f[currentDisparity+1], __float2half2_rn(1.0f));
		f[currentDisparity] = getMinBothPartsHalf2(prev, f[currentDisparity]);
	}
}


// compute current message
template<typename T>
__device__ void msgStereo(T messageValsNeighbor1[NUM_POSSIBLE_DISPARITY_VALUES], T messageValsNeighbor2[NUM_POSSIBLE_DISPARITY_VALUES],
	T messageValsNeighbor3[NUM_POSSIBLE_DISPARITY_VALUES], T dataCosts[NUM_POSSIBLE_DISPARITY_VALUES],
	T dst[NUM_POSSIBLE_DISPARITY_VALUES], T disc_k_bp)
{
	// aggregate and find min
	T minimum = INF_BP;

	for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
	{
		dst[currentDisparity] = messageValsNeighbor1[currentDisparity] + messageValsNeighbor2[currentDisparity] + messageValsNeighbor3[currentDisparity] + dataCosts[currentDisparity];
		if (dst[currentDisparity] < minimum)
			minimum = dst[currentDisparity];
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
	dtStereo<T>(dst);

	// truncate 
	minimum += disc_k_bp;

	// normalize
	T valToNormalize = 0;

	for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
	{
		if (minimum < dst[currentDisparity])
		{
			dst[currentDisparity] = minimum;
		}

		valToNormalize += dst[currentDisparity];
	}
	
	valToNormalize /= NUM_POSSIBLE_DISPARITY_VALUES;

	for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++) 
		dst[currentDisparity] -= valToNormalize;
}


//template specialization for processing messages with half-precision; has safeguard to check if valToNormalize goes to infinity and set output
//for every disparity at point to be 0.0 if that's the case; this has only been observed when using more than 5 computation levels with half-precision
template<>
__device__ void msgStereo<half>(half messageValsNeighbor1[NUM_POSSIBLE_DISPARITY_VALUES], half messageValsNeighbor2[NUM_POSSIBLE_DISPARITY_VALUES],
		half messageValsNeighbor3[NUM_POSSIBLE_DISPARITY_VALUES], half dataCosts[NUM_POSSIBLE_DISPARITY_VALUES],
		half dst[NUM_POSSIBLE_DISPARITY_VALUES], half disc_k_bp)
{
	// aggregate and find min
	half minimum = INF_BP;

	for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
	{
		dst[currentDisparity] = messageValsNeighbor1[currentDisparity] + messageValsNeighbor2[currentDisparity] + messageValsNeighbor3[currentDisparity] + dataCosts[currentDisparity];
		if (dst[currentDisparity] < minimum)
			minimum = dst[currentDisparity];
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
	dtStereo<half>(dst);

	// truncate
	minimum += disc_k_bp;

	// normalize
	half valToNormalize = 0;

	for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
	{
		if (minimum < dst[currentDisparity])
		{
			dst[currentDisparity] = minimum;
		}

		valToNormalize += dst[currentDisparity];
	}

	//if valToNormalize is infinite or NaN (observed when using more than 5 computation levels with half-precision),
	//set destination vector to 0 for all disparities
	//note that may cause results to differ a little from ideal
	if (__hisnan(valToNormalize) || ((__hisinf(valToNormalize)) != 0))
	{
		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = (half)0.0;
		}
	}
	else
	{
		valToNormalize /= NUM_POSSIBLE_DISPARITY_VALUES;

		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] -= valToNormalize;
		}
	}
}


template<>
__device__ void msgStereo<half2>(half2 messageValsNeighbor1[NUM_POSSIBLE_DISPARITY_VALUES], half2 messageValsNeighbor2[NUM_POSSIBLE_DISPARITY_VALUES],
		half2 messageValsNeighbor3[NUM_POSSIBLE_DISPARITY_VALUES], half2 dataCosts[NUM_POSSIBLE_DISPARITY_VALUES],
		half2 dst[NUM_POSSIBLE_DISPARITY_VALUES], half2 disc_k_bp)
{
	// aggregate and find min
	half2 minimum = __float2half2_rn(INF_BP);

	for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
	{
		dst[currentDisparity] = __hadd2(messageValsNeighbor1[currentDisparity], messageValsNeighbor2[currentDisparity]);
		dst[currentDisparity] = __hadd2(dst[currentDisparity], messageValsNeighbor3[currentDisparity]);
		dst[currentDisparity] = __hadd2(dst[currentDisparity], dataCosts[currentDisparity]);

		minimum = getMinBothPartsHalf2(dst[currentDisparity], minimum);
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
	dtStereo<half2>(dst);

	// truncate
	minimum = __hadd2(minimum, disc_k_bp);

	// normalize
	half2 valToNormalize = __float2half2_rn(0.0f);

	for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
	{
		dst[currentDisparity] = getMinBothPartsHalf2(minimum, dst[currentDisparity]);
		valToNormalize = __hadd2(valToNormalize, dst[currentDisparity]);
	}

	//if either valToNormalize in half2 is infinite or NaN, set destination vector to 0 for all disparities
	//note that may cause results to differ a little from ideal
	if (((__hisnan(__low2half(valToNormalize)))
			|| ((__hisinf(__low2half(valToNormalize)) != 0)))
			|| ((__hisnan(__high2half(valToNormalize)))
					|| ((__hisinf(__high2half(valToNormalize)) != 0))))
	{
		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = __floats2half2_rn(0.0f, 0.0f);
		}
	}
	else
	{
		valToNormalize = __h2div(valToNormalize,
				__float2half2_rn((float) NUM_POSSIBLE_DISPARITY_VALUES));

		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = __hsub2(dst[currentDisparity],
					valToNormalize);
		}
	}
	//check if both values in half2 are inf or nan
	/*if (((__hisnan(__low2half(valToNormalize)))
			|| ((__hisinf(__low2half(valToNormalize)) != 0)))
			&& ((__hisnan(__high2half(valToNormalize)))
					|| ((__hisinf(__high2half(valToNormalize)) != 0))))
	{
		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = __floats2half2_rn(0.0f, 0.0f);
		}
	}
	else if (((__hisnan(__low2half(valToNormalize)))
			|| ((__hisinf(__low2half(valToNormalize)) != 0))))
	{
		//lower half of half2 is inf or nan
		valToNormalize = __h2div(valToNormalize,
				__float2half2_rn((float) NUM_POSSIBLE_DISPARITY_VALUES));

		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = __hsub2(dst[currentDisparity],
					valToNormalize);
		}

		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = __halves2half2((half)0.0f,
					__high2half(dst[currentDisparity]));
		}
	}
	else if ((__hisnan(__high2half(valToNormalize)))
			|| ((__hisinf(__high2half(valToNormalize)) != 0)))
	{
		//higher half of half2 is inf or nan
		valToNormalize = __h2div(valToNormalize,
				__float2half2_rn((float) NUM_POSSIBLE_DISPARITY_VALUES));

		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = __hsub2(dst[currentDisparity],
					valToNormalize);
		}

		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dst[currentDisparity] = __halves2half2(
					__low2half(dst[currentDisparity]), (half)0.0f);
		}
	}*/
}


//initialize the "data cost" for each possible disparity between the two full-sized input images ("bottom" of the image pyramid)
//the image data is stored in the CUDA arrays image1PixelsTextureBPStereo and image2PixelsTextureBPStereo
template<typename T>
__global__ void initializeBottomLevelDataStereo(levelProperties currentLevelProperties, float* image1PixelsDevice, float* image2PixelsDevice, T* dataCostDeviceStereoCheckerboard1, T* dataCostDeviceStereoCheckerboard2, float lambda_bp, float data_k_bp)
{
	// Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_BP + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_BP + ty;

	int indexVal;
	//int imageCheckerboardWidth = getCheckerboardWidth<T>(widthImages);
	int xInCheckerboard = xVal / 2;

	if (withinImageBounds(xInCheckerboard, yVal, currentLevelProperties.widthLevel, currentLevelProperties.heightLevel))
	{
		//make sure that it is possible to check every disparity value
		if ((xVal - (NUM_POSSIBLE_DISPARITY_VALUES-1)) >= 0)
		{
			for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
			{
				float currentPixelImage1 = 0.0f;
				float currentPixelImage2 = 0.0f;

				if (withinImageBounds(xVal, yVal, currentLevelProperties.widthLevel, currentLevelProperties.heightLevel))
				{
					currentPixelImage1 = image1PixelsDevice[yVal * currentLevelProperties.widthLevel
							+ xVal];
					currentPixelImage2 = image2PixelsDevice[yVal * currentLevelProperties.widthLevel
							+ (xVal - currentDisparity)];
				}

				indexVal = retrieveIndexInDataAndMessage(xInCheckerboard, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES);

				//data cost is equal to dataWeight value for weighting times the absolute difference in corresponding pixel intensity values capped at dataCostCap
				if (((xVal + yVal) % 2) == 0)
				{
					dataCostDeviceStereoCheckerboard1[indexVal] = (T)(lambda_bp * min(((T)abs(currentPixelImage1 - currentPixelImage2)), data_k_bp));
				}
				else
				{
					dataCostDeviceStereoCheckerboard2[indexVal] = (T)(lambda_bp * min(((T)abs(currentPixelImage1 - currentPixelImage2)), data_k_bp));
				}
			}
		}
		else
		{
			for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
			{
				indexVal = retrieveIndexInDataAndMessage(xInCheckerboard, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES);

				//data cost is equal to dataWeight value for weighting times the absolute difference in corresponding pixel intensity values capped at dataCostCap
				if (((xVal + yVal) % 2) == 0)
				{
					dataCostDeviceStereoCheckerboard1[indexVal] = getZeroVal<T>();
				}
				else
				{
					dataCostDeviceStereoCheckerboard2[indexVal] = getZeroVal<T>();
				}
			}
		}
	}
}


template<typename T>
__global__ void printDataAndMessageValsAtPointKernel(int xVal, int yVal, T* dataCostStereoCheckerboard1, T* dataCostStereoCheckerboard2,
		T* messageUDeviceCurrentCheckerboard1,
		T* messageDDeviceCurrentCheckerboard1,
		T* messageLDeviceCurrentCheckerboard1,
		T* messageRDeviceCurrentCheckerboard1,
		T* messageUDeviceCurrentCheckerboard2,
		T* messageDDeviceCurrentCheckerboard2,
		T* messageLDeviceCurrentCheckerboard2,
		T* messageRDeviceCurrentCheckerboard2, int widthLevelCheckerboardPart,
		int heightLevel)
{
	if (((xVal + yVal) % 2) == 0) {
		printf("xVal: %d\n", xVal);
		printf("yVal: %d\n", yVal);
		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++) {
			printf("DISP: %d\n", currentDisparity);
			printf("messageUPrevStereoCheckerboard: %f \n",
					(float) messageUDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageDPrevStereoCheckerboard: %f \n",
					(float) messageDDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageLPrevStereoCheckerboard: %f \n",
					(float) messageLDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageRPrevStereoCheckerboard: %f \n",
					(float) messageRDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("dataCostStereoCheckerboard: %f \n",
					(float) dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
		}
	} else {
		printf("xVal: %d\n", xVal);
		printf("yVal: %d\n", yVal);
		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++) {
			printf("DISP: %d\n", currentDisparity);
			printf("messageUPrevStereoCheckerboard: %f \n",
					(float) messageUDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageDPrevStereoCheckerboard: %f \n",
					(float) messageDDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageLPrevStereoCheckerboard: %f \n",
					(float) messageLDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageRPrevStereoCheckerboard: %f \n",
					(float) messageRDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("dataCostStereoCheckerboard: %f \n",
					(float) dataCostStereoCheckerboard2[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
		}
	}
}


template<typename T>
__device__ void printDataAndMessageValsAtPointDevice(int xVal, int yVal, T* dataCostStereoCheckerboard1, T* dataCostStereoCheckerboard2,
		T* messageUDeviceCurrentCheckerboard1,
		T* messageDDeviceCurrentCheckerboard1,
		T* messageLDeviceCurrentCheckerboard1,
		T* messageRDeviceCurrentCheckerboard1,
		T* messageUDeviceCurrentCheckerboard2,
		T* messageDDeviceCurrentCheckerboard2,
		T* messageLDeviceCurrentCheckerboard2,
		T* messageRDeviceCurrentCheckerboard2, int widthLevelCheckerboardPart,
		int heightLevel)
{
	if (((xVal + yVal) % 2) == 0) {
		printf("xVal: %d\n", xVal);
		printf("yVal: %d\n", yVal);
		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++) {
			printf("DISP: %d\n", currentDisparity);
			printf("messageUPrevStereoCheckerboard: %f \n",
					(float) messageUDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageDPrevStereoCheckerboard: %f \n",
					(float) messageDDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageLPrevStereoCheckerboard: %f \n",
					(float) messageLDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageRPrevStereoCheckerboard: %f \n",
					(float) messageRDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("dataCostStereoCheckerboard: %f \n",
					(float) dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
		}
	} else {
		printf("xVal: %d\n", xVal);
		printf("yVal: %d\n", yVal);
		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++) {
			printf("DISP: %d\n", currentDisparity);
			printf("messageUPrevStereoCheckerboard: %f \n",
					(float) messageUDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageDPrevStereoCheckerboard: %f \n",
					(float) messageDDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageLPrevStereoCheckerboard: %f \n",
					(float) messageLDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageRPrevStereoCheckerboard: %f \n",
					(float) messageRDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("dataCostStereoCheckerboard: %f \n",
					(float) dataCostStereoCheckerboard2[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
		}
	}
}


template<typename T>
__global__ void printDataAndMessageValsToPointKernel(int xVal, int yVal, T* dataCostStereoCheckerboard1, T* dataCostStereoCheckerboard2,
		T* messageUDeviceCurrentCheckerboard1,
		T* messageDDeviceCurrentCheckerboard1,
		T* messageLDeviceCurrentCheckerboard1,
		T* messageRDeviceCurrentCheckerboard1,
		T* messageUDeviceCurrentCheckerboard2,
		T* messageDDeviceCurrentCheckerboard2,
		T* messageLDeviceCurrentCheckerboard2,
		T* messageRDeviceCurrentCheckerboard2, int widthLevelCheckerboardPart,
		int heightLevel)
{
	int checkerboardAdjustment;
	if (((xVal + yVal) % 2) == 0)
		{
			checkerboardAdjustment = ((yVal)%2);
		}
		else //checkerboardToUpdate == CHECKERBOARD_PART_2
		{
			checkerboardAdjustment = ((yVal+1)%2);
		}
	if (((xVal + yVal) % 2) == 0) {
			printf("xVal: %d\n", xVal);
			printf("yVal: %d\n", yVal);
			for (int currentDisparity = 0;
					currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
					currentDisparity++) {
				printf("DISP: %d\n", currentDisparity);
				printf("messageUPrevStereoCheckerboard: %f \n",
						(float) messageUDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
								xVal / 2, yVal + 1, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
				printf("messageDPrevStereoCheckerboard: %f \n",
						(float) messageDDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
								xVal / 2, yVal - 1, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
				printf("messageLPrevStereoCheckerboard: %f \n",
						(float) messageLDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
								xVal / 2 + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
				printf("messageRPrevStereoCheckerboard: %f \n",
						(float) messageRDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
								(xVal / 2 - 1) + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
				printf("dataCostStereoCheckerboard: %f \n",
						(float) dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(
								xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			}
		} else {
			printf("xVal: %d\n", xVal);
			printf("yVal: %d\n", yVal);
			for (int currentDisparity = 0;
					currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
					currentDisparity++) {
				printf("DISP: %d\n", currentDisparity);
				printf("messageUPrevStereoCheckerboard: %f \n",
						(float) messageUDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
								xVal / 2, yVal + 1, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
				printf("messageDPrevStereoCheckerboard: %f \n",
						(float) messageDDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
								xVal / 2, yVal - 1, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
				printf("messageLPrevStereoCheckerboard: %f \n",
						(float) messageLDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
								xVal / 2 + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
				printf("messageRPrevStereoCheckerboard: %f \n",
						(float) messageRDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
								(xVal / 2 - 1) + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
				printf("dataCostStereoCheckerboard: %f \n",
						(float) dataCostStereoCheckerboard2[retrieveIndexInDataAndMessage(
								xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
								currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			}
		}
}


template<typename T>
__device__ void printDataAndMessageValsToPointDevice(int xVal, int yVal, T* dataCostStereoCheckerboard1, T* dataCostStereoCheckerboard2,
		T* messageUDeviceCurrentCheckerboard1,
		T* messageDDeviceCurrentCheckerboard1,
		T* messageLDeviceCurrentCheckerboard1,
		T* messageRDeviceCurrentCheckerboard1,
		T* messageUDeviceCurrentCheckerboard2,
		T* messageDDeviceCurrentCheckerboard2,
		T* messageLDeviceCurrentCheckerboard2,
		T* messageRDeviceCurrentCheckerboard2, int widthLevelCheckerboardPart,
		int heightLevel)
{
	int checkerboardAdjustment;
	if (((xVal + yVal) % 2) == 0)
		{
			checkerboardAdjustment = ((yVal)%2);
		}
		else //checkerboardToUpdate == CHECKERBOARD_PART_2
		{
			checkerboardAdjustment = ((yVal+1)%2);
		}

	if (((xVal + yVal) % 2) == 0) {
		printf("xVal: %d\n", xVal);
		printf("yVal: %d\n", yVal);
		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++) {
			printf("DISP: %d\n", currentDisparity);
			printf("messageUPrevStereoCheckerboard: %f \n",
					(float) messageUDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
							xVal / 2, yVal + 1, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageDPrevStereoCheckerboard: %f \n",
					(float) messageDDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
							xVal / 2, yVal - 1, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageLPrevStereoCheckerboard: %f \n",
					(float) messageLDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
							xVal / 2 + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageRPrevStereoCheckerboard: %f \n",
					(float) messageRDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
							(xVal / 2 - 1) + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("dataCostStereoCheckerboard: %f \n",
					(float) dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
		}
	} else {
		printf("xVal: %d\n", xVal);
		printf("yVal: %d\n", yVal);
		for (int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++) {
			printf("DISP: %d\n", currentDisparity);
			printf("messageUPrevStereoCheckerboard: %f \n",
					(float) messageUDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal + 1, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageDPrevStereoCheckerboard: %f \n",
					(float) messageDDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2, yVal - 1, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageLPrevStereoCheckerboard: %f \n",
					(float) messageLDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							xVal / 2 + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("messageRPrevStereoCheckerboard: %f \n",
					(float) messageRDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
							(xVal / 2 - 1) + checkerboardAdjustment, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
			printf("dataCostStereoCheckerboard: %f \n",
					(float) dataCostStereoCheckerboard2[retrieveIndexInDataAndMessage(
							xVal / 2, yVal, widthLevelCheckerboardPart, heightLevel,
							currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
		}
	}
}


template<>
__global__ void initializeBottomLevelDataStereo<half2>(levelProperties currentLevelProperties, float* image1PixelsDevice, float* image2PixelsDevice, half2* dataCostDeviceStereoCheckerboard1, half2* dataCostDeviceStereoCheckerboard2, float lambda_bp, float data_k_bp)
{/*
	// Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_BP + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_BP + ty;

	int indexVal;
	int imageCheckerboardWidth = getCheckerboardWidth<half2>(widthImages);
	int xInCheckerboard = xVal / 2;

	if (withinImageBounds(xInCheckerboard, yVal, imageCheckerboardWidth, heightImages))
	{
		int imageXPixelIndexStart = 0;
		int checkerboardNum = 1;

		//check which checkerboard data values for and make necessary adjustment to start
		if (((yVal) % 2) == 0) {
			if (((xVal) % 2) == 0) {
				checkerboardNum = 1;
			} else {
				checkerboardNum = 2;
			}
		} else {
			if (((xVal) % 2) == 0) {
				checkerboardNum = 2;
			} else {
				checkerboardNum = 1;
			}
		}

		imageXPixelIndexStart = xVal*2;
		if ((((yVal) % 2) == 0) && (checkerboardNum == 2)) {
			imageXPixelIndexStart -= 1;
		}
		if ((((yVal) % 2) == 1) && (checkerboardNum == 1)) {
			imageXPixelIndexStart -= 1;
		}

		//make sure that it is possible to check every disparity value
		if ((((imageXPixelIndexStart + 2) - (NUM_POSSIBLE_DISPARITY_VALUES-1)) >= 0))
		{
			for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
			{
				float currentPixelImage1_low = 0.0;
				float currentPixelImage2_low = 0.0;

				if ((((imageXPixelIndexStart) - (NUM_POSSIBLE_DISPARITY_VALUES-1)) >= 0))
				{
					if (withinImageBounds(imageXPixelIndexStart, yVal, widthImages,
							heightImages)) {
						currentPixelImage1_low = image1PixelsDevice[yVal
								* widthImages + imageXPixelIndexStart];
						currentPixelImage2_low = image2PixelsDevice[yVal
								* widthImages + (imageXPixelIndexStart - currentDisparity)];
					}
				}

				float currentPixelImage1_high = 0.0;
				float currentPixelImage2_high = 0.0;

				if (withinImageBounds(imageXPixelIndexStart + 2, yVal, widthImages,
						heightImages))
				{
					currentPixelImage1_high = image1PixelsDevice[yVal * widthImages
							+ (imageXPixelIndexStart + 2)];
					currentPixelImage2_high = image2PixelsDevice[yVal * widthImages
							+ ((imageXPixelIndexStart + 2) - currentDisparity)];
				}

				indexVal = retrieveIndexInDataAndMessage(xInCheckerboard, yVal, imageCheckerboardWidth, heightImages, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES);

				half lowVal = (half)(lambda_bp * min(abs(currentPixelImage1_low - currentPixelImage2_low), data_k_bp));
				half highVal = (half)(lambda_bp * min(abs(currentPixelImage1_high - currentPixelImage2_high), data_k_bp));

				//data cost is equal to dataWeight value for weighting times the absolute difference in corresponding pixel intensity values capped at dataCostCap
				if (checkerboardNum == 1)
				{
					dataCostDeviceStereoCheckerboard1[indexVal] = __halves2half2(lowVal, highVal);
				}
				else
				{
					dataCostDeviceStereoCheckerboard2[indexVal] = __halves2half2(lowVal, highVal);
				}
			}
		}
		else
		{
			for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
			{
				indexVal = retrieveIndexInDataAndMessage(xInCheckerboard, yVal, imageCheckerboardWidth, heightImages, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES);

				//data cost is equal to dataWeight value for weighting times the absolute difference in corresponding pixel intensity values capped at dataCostCap
				if (((xVal + yVal) % 2) == 0)
				{
					dataCostDeviceStereoCheckerboard1[indexVal] = getZeroVal<half2>();
				}
				else
				{
					dataCostDeviceStereoCheckerboard2[indexVal] = getZeroVal<half2>();
				}
			}
		}
	}*/
}


//initialize the data costs at the "next" level up in the pyramid given that the data at the lower has been set
template<typename T>
__global__ void initializeCurrentLevelDataStereoNoTextures(int checkerboardPart, levelProperties currentLevelProperties, levelProperties& prevLevelProperties, T* dataCostStereoCheckerboard1, T* dataCostStereoCheckerboard2, T* dataCostDeviceToWriteTo, int offsetNum)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_BP + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_BP + ty;
	//int widthCheckerboardCurrentLevel = getCheckerboardWidth<T>(widthCurrentLevel);
	//int widthCheckerboardPrevLevel = getCheckerboardWidth<T>(widthPrevLevel);

	if (withinImageBounds(xVal, yVal, currentLevelProperties.widthCheckerboardLevel, currentLevelProperties.heightLevel))
	{
		//add 1 or 0 to the x-value depending on checkerboard part and row adding to; CHECKERBOARD_PART_1 with slot at (0, 0) has adjustment of 0 in row 0,
		//while CHECKERBOARD_PART_2 with slot at (0, 1) has adjustment of 1 in row 0
		int checkerboardPartAdjustment = 0;

		if (checkerboardPart == CHECKERBOARD_PART_1)
		{
			checkerboardPartAdjustment = (yVal%2);
		}
		else if (checkerboardPart == CHECKERBOARD_PART_2)
		{
			checkerboardPartAdjustment = ((yVal+1)%2);
		}

		//the corresponding x-values at the "lower" level depends on which checkerboard the pixel is in
		int xValPrev = xVal*2 + checkerboardPartAdjustment;

		if (withinImageBounds(xValPrev, (yVal * 2 + 1), prevLevelProperties.widthCheckerboardLevel, prevLevelProperties.heightLevel))
		{
			for (int currentDisparity = 0;
					currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
					currentDisparity++)
			{
				dataCostDeviceToWriteTo[retrieveIndexInDataAndMessage(xVal, yVal,
						currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel,
						currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] =
						(dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(
								xValPrev, (yVal * 2), prevLevelProperties.paddedWidthCheckerboardLevel,
								prevLevelProperties.heightLevel, currentDisparity,
								NUM_POSSIBLE_DISPARITY_VALUES, offsetNum)]
								+ dataCostStereoCheckerboard2[retrieveIndexInDataAndMessage(
										xValPrev, (yVal * 2),
										prevLevelProperties.paddedWidthCheckerboardLevel, prevLevelProperties.heightLevel,
										currentDisparity,
										NUM_POSSIBLE_DISPARITY_VALUES, offsetNum)]
								+ dataCostStereoCheckerboard2[retrieveIndexInDataAndMessage(
										xValPrev, (yVal * 2 + 1),
										prevLevelProperties.paddedWidthCheckerboardLevel, prevLevelProperties.heightLevel,
										currentDisparity,
										NUM_POSSIBLE_DISPARITY_VALUES, offsetNum)]
								+ dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(
										xValPrev, (yVal * 2 + 1),
										prevLevelProperties.paddedWidthCheckerboardLevel, prevLevelProperties.heightLevel,
										currentDisparity,
										NUM_POSSIBLE_DISPARITY_VALUES, offsetNum)]);
			}
		}
	}
}


//initialize the message values at each pixel of the current level to the default value
template<typename T>
__global__ void initializeMessageValsToDefaultKernel(levelProperties currentLevelProperties, T* messageUDeviceCurrentCheckerboard1, T* messageDDeviceCurrentCheckerboard1, T* messageLDeviceCurrentCheckerboard1,
												T* messageRDeviceCurrentCheckerboard1, T* messageUDeviceCurrentCheckerboard2, T* messageDDeviceCurrentCheckerboard2,
												T* messageLDeviceCurrentCheckerboard2, T* messageRDeviceCurrentCheckerboard2)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xValInCheckerboard = bx * BLOCK_SIZE_WIDTH_BP + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_BP + ty;

	if (withinImageBounds(xValInCheckerboard, yVal, currentLevelProperties.widthCheckerboardLevel, currentLevelProperties.heightLevel))
	{
		//initialize message values in both checkerboards

		//set the message value at each pixel for each disparity to 0
		for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
		{
			messageUDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboard, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] = getZeroVal<T>();
			messageDDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboard, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] = getZeroVal<T>();
			messageLDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboard, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] = getZeroVal<T>();
			messageRDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboard, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] = getZeroVal<T>();
		}

		//retrieve the previous message value at each movement at each pixel
		for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
		{
			messageUDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboard, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] = getZeroVal<T>();
			messageDDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboard, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] = getZeroVal<T>();
			messageLDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboard, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] = getZeroVal<T>();
			messageRDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboard, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] = getZeroVal<T>();
		}	
	}
}


//device portion of the kernel function to run the current iteration of belief propagation where the input messages and data costs come in as array in local memory
//and the output message values are stored in local memory
template<typename T>
__device__ void runBPIterationInOutDataInLocalMem(T prevUMessage[NUM_POSSIBLE_DISPARITY_VALUES], T prevDMessage[NUM_POSSIBLE_DISPARITY_VALUES], T prevLMessage[NUM_POSSIBLE_DISPARITY_VALUES], T prevRMessage[NUM_POSSIBLE_DISPARITY_VALUES], T dataMessage[NUM_POSSIBLE_DISPARITY_VALUES],
								T currentUMessage[NUM_POSSIBLE_DISPARITY_VALUES], T currentDMessage[NUM_POSSIBLE_DISPARITY_VALUES], T currentLMessage[NUM_POSSIBLE_DISPARITY_VALUES], T currentRMessage[NUM_POSSIBLE_DISPARITY_VALUES], T disc_k_bp)
 {
	msgStereo<T>(prevUMessage, prevLMessage, prevRMessage, dataMessage,
			currentUMessage, disc_k_bp);

	msgStereo<T>(prevDMessage, prevLMessage, prevRMessage, dataMessage,
			currentDMessage, disc_k_bp);

	msgStereo<T>(prevUMessage, prevDMessage, prevRMessage, dataMessage,
			currentRMessage, disc_k_bp);

	msgStereo<T>(prevUMessage, prevDMessage, prevLMessage, dataMessage,
			currentLMessage, disc_k_bp);
}


//device portion of the kernal function to run the current iteration of belief propagation in parallel using the checkerboard update method where half the pixels in the
//"checkerboard" scheme retrieve messages from each 4-connected neighbor and then update their message based on the retrieved messages and the data cost
//this function uses local memory to store the message and data values at each disparity in the intermediate step of current message computation
//this function uses linear memory bound to textures to access the current data and message values
template<typename T>
__device__ void runBPIterationUsingCheckerboardUpdatesDeviceNoTexBoundAndLocalMem(int xVal, int yVal,
		int checkerboardToUpdate, levelProperties currentLevelProperties, T* dataCostStereoCheckerboard1, T* dataCostStereoCheckerboard2,
		T* messageUDeviceCurrentCheckerboard1,
		T* messageDDeviceCurrentCheckerboard1,
		T* messageLDeviceCurrentCheckerboard1,
		T* messageRDeviceCurrentCheckerboard1,
		T* messageUDeviceCurrentCheckerboard2,
		T* messageDDeviceCurrentCheckerboard2,
		T* messageLDeviceCurrentCheckerboard2,
		T* messageRDeviceCurrentCheckerboard2,
		float disc_k_bp, int offsetData)
{
	int indexWriteTo;
	int checkerboardAdjustment;

	//checkerboardAdjustment used for indexing into current checkerboard to update
	if (checkerboardToUpdate == CHECKERBOARD_PART_1)
	{
		checkerboardAdjustment = ((yVal)%2);
	}
	else //checkerboardToUpdate == CHECKERBOARD_PART_2
	{
		checkerboardAdjustment = ((yVal+1)%2);
	}

	//may want to look into (xVal < (widthLevelCheckerboardPart - 1) since it may affect the edges
	//make sure that the current point is not an edge/corner that doesn't have four neighbors that can pass values to it
	//if ((xVal >= (1 - checkerboardAdjustment)) && (xVal < (widthLevelCheckerboardPart - 1)) && (yVal > 0) && (yVal < (heightLevel - 1)))
	if ((xVal >= (1 - checkerboardAdjustment)) && (xVal < (currentLevelProperties.widthCheckerboardLevel - checkerboardAdjustment)) && (yVal > 0) && (yVal < (currentLevelProperties.heightLevel - 1)))
	{
		T prevUMessage[NUM_POSSIBLE_DISPARITY_VALUES];
		T prevDMessage[NUM_POSSIBLE_DISPARITY_VALUES];
		T prevLMessage[NUM_POSSIBLE_DISPARITY_VALUES];
		T prevRMessage[NUM_POSSIBLE_DISPARITY_VALUES];

		T dataMessage[NUM_POSSIBLE_DISPARITY_VALUES];

		for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
		{
			if (checkerboardToUpdate == CHECKERBOARD_PART_1)
			{
				dataMessage[currentDisparity] = dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(xVal, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES, offsetData)];
				prevUMessage[currentDisparity] = messageUDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(xVal, (yVal+1), currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)];
				prevDMessage[currentDisparity] = messageDDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(xVal, (yVal-1), currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)];
				prevLMessage[currentDisparity] = messageLDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage((xVal + checkerboardAdjustment), (yVal), currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)];
				prevRMessage[currentDisparity] = messageRDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(((xVal - 1) + checkerboardAdjustment), (yVal), currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)];
			}
			else //checkerboardToUpdate == CHECKERBOARD_PART_2
			{
				dataMessage[currentDisparity] = dataCostStereoCheckerboard2[retrieveIndexInDataAndMessage(xVal, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES, offsetData)];
				prevUMessage[currentDisparity] = messageUDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(xVal, (yVal+1), currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)];
				prevDMessage[currentDisparity] = messageDDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(xVal, (yVal-1), currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)];
				prevLMessage[currentDisparity] = messageLDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage((xVal + checkerboardAdjustment), (yVal), currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)];
				prevRMessage[currentDisparity] = messageRDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(((xVal - 1) + checkerboardAdjustment), (yVal), currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)];
			}
		}

		T currentUMessage[NUM_POSSIBLE_DISPARITY_VALUES];
		T currentDMessage[NUM_POSSIBLE_DISPARITY_VALUES];
		T currentLMessage[NUM_POSSIBLE_DISPARITY_VALUES];
		T currentRMessage[NUM_POSSIBLE_DISPARITY_VALUES];

		//uses the previous message values and data cost to calculate the current message values and store the results
		runBPIterationInOutDataInLocalMem<T>(prevUMessage, prevDMessage, prevLMessage, prevRMessage, dataMessage,
							currentUMessage, currentDMessage, currentLMessage, currentRMessage, (T)disc_k_bp);

		//write the calculated message values to global memory
		for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
		{
			indexWriteTo = retrieveIndexInDataAndMessage(xVal, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES);
			if (checkerboardToUpdate == CHECKERBOARD_PART_1)
			{
				messageUDeviceCurrentCheckerboard1[indexWriteTo] = currentUMessage[currentDisparity];
				messageDDeviceCurrentCheckerboard1[indexWriteTo] = currentDMessage[currentDisparity];
				messageLDeviceCurrentCheckerboard1[indexWriteTo] = currentLMessage[currentDisparity];
				messageRDeviceCurrentCheckerboard1[indexWriteTo] = currentRMessage[currentDisparity];
			}
			else //checkerboardToUpdate == CHECKERBOARD_PART_2
			{
				messageUDeviceCurrentCheckerboard2[indexWriteTo] = currentUMessage[currentDisparity];
				messageDDeviceCurrentCheckerboard2[indexWriteTo] = currentDMessage[currentDisparity];
				messageLDeviceCurrentCheckerboard2[indexWriteTo] = currentLMessage[currentDisparity];
				messageRDeviceCurrentCheckerboard2[indexWriteTo] = currentRMessage[currentDisparity];
			}
		}
	}
}


//device portion of the kernal function to run the current iteration of belief propagation in parallel using the checkerboard update method where half the pixels in the
//"checkerboard" scheme retrieve messages from each 4-connected neighbor and then update their message based on the retrieved messages and the data cost
//this function uses local memory to store the message and data values at each disparity in the intermediate step of current message computation
//this function uses linear memory bound to textures to access the current data and message values
template<>
__device__ void runBPIterationUsingCheckerboardUpdatesDeviceNoTexBoundAndLocalMem<half2>(int xVal, int yVal,
		int checkerboardToUpdate, levelProperties currentLevelProperties, half2* dataCostStereoCheckerboard1, half2* dataCostStereoCheckerboard2,
		half2* messageUDeviceCurrentCheckerboard1,
		half2* messageDDeviceCurrentCheckerboard1,
		half2* messageLDeviceCurrentCheckerboard1,
		half2* messageRDeviceCurrentCheckerboard1,
		half2* messageUDeviceCurrentCheckerboard2,
		half2* messageDDeviceCurrentCheckerboard2,
		half2* messageLDeviceCurrentCheckerboard2,
		half2* messageRDeviceCurrentCheckerboard2,
		float disc_k_bp, int offsetData)
{
}
/*
	int indexWriteTo;
	int checkerboardAdjustment;

	//checkerboardAdjustment used for indexing into current checkerboard to update
	if (checkerboardToUpdate == CHECKERBOARD_PART_1)
	{
		checkerboardAdjustment = ((yVal)%2);
	}
	else //checkerboardToUpdate == CHECKERBOARD_PART_2
	{
		checkerboardAdjustment = ((yVal+1)%2);
	}

	//may want to look into (xVal < (widthLevelCheckerboardPart - 1) since it may affect the edges
	//make sure that the current point is not an edge/corner that doesn't have four neighbors that can pass values to it
	//if ((xVal >= (1 - checkerboardAdjustment)) && (xVal < (widthLevelCheckerboardPart - 1)) && (yVal > 0) && (yVal < (heightLevel - 1)))
	if ((xVal >= (1/*switch to 0 if trying to match half results exactly*//* - checkerboardAdjustment)) && (xVal < (widthLevelCheckerboardPart - checkerboardAdjustment)) && (yVal > 0) && (yVal < (heightLevel - 1)))
	{
		half2 prevUMessage[NUM_POSSIBLE_DISPARITY_VALUES];
		half2 prevDMessage[NUM_POSSIBLE_DISPARITY_VALUES];
		half2 prevLMessage[NUM_POSSIBLE_DISPARITY_VALUES];
		half2 prevRMessage[NUM_POSSIBLE_DISPARITY_VALUES];

		half2 dataMessage[NUM_POSSIBLE_DISPARITY_VALUES];

		if (checkerboardToUpdate == CHECKERBOARD_PART_1)
		{
			half* messageLDeviceCurrentCheckerboard2Half = (half*)messageLDeviceCurrentCheckerboard2;
			half* messageRDeviceCurrentCheckerboard2Half = (half*)messageRDeviceCurrentCheckerboard2;

			for (int currentDisparity = 0;
					currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
					currentDisparity++)
			{
				dataMessage[currentDisparity] =
						dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(
								xVal, yVal, widthLevelCheckerboardPart,
								heightLevel, currentDisparity,
								NUM_POSSIBLE_DISPARITY_VALUES, offsetData)];
				prevUMessage[currentDisparity] =
						messageUDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
								xVal, (yVal + 1), widthLevelCheckerboardPart,
								heightLevel, currentDisparity,
								NUM_POSSIBLE_DISPARITY_VALUES)];
				prevDMessage[currentDisparity] =
						messageDDeviceCurrentCheckerboard2[retrieveIndexInDataAndMessage(
								xVal, (yVal - 1), widthLevelCheckerboardPart,
								heightLevel, currentDisparity,
								NUM_POSSIBLE_DISPARITY_VALUES)];
				prevLMessage[currentDisparity] =
						__halves2half2(
								messageLDeviceCurrentCheckerboard2Half[retrieveIndexInDataAndMessage(
										((xVal * 2) + checkerboardAdjustment),
										yVal, widthLevelCheckerboardPart * 2,
										heightLevel, currentDisparity,
										NUM_POSSIBLE_DISPARITY_VALUES)],
								messageLDeviceCurrentCheckerboard2Half[retrieveIndexInDataAndMessage(
										((xVal * 2 + 1) + checkerboardAdjustment),
										yVal, widthLevelCheckerboardPart * 2,
										heightLevel, currentDisparity,
										NUM_POSSIBLE_DISPARITY_VALUES)]);

				//if ((((xVal * 2) - 1) + checkerboardAdjustment) >= 0)
				{
					prevRMessage[currentDisparity] =
							__halves2half2(
									messageRDeviceCurrentCheckerboard2Half[retrieveIndexInDataAndMessage(
											(((xVal * 2) - 1)
													+ checkerboardAdjustment),
											yVal,
											widthLevelCheckerboardPart * 2,
											heightLevel, currentDisparity,
											NUM_POSSIBLE_DISPARITY_VALUES)],
									messageRDeviceCurrentCheckerboard2Half[retrieveIndexInDataAndMessage(
											(((xVal * 2 + 1) - 1)
													+ checkerboardAdjustment),
											yVal,
											widthLevelCheckerboardPart * 2,
											heightLevel, currentDisparity,
											NUM_POSSIBLE_DISPARITY_VALUES)]);
				}
				/*else
				{
					prevRMessage[currentDisparity] =
							__halves2half2((half)0.0f,
									messageRDeviceCurrentCheckerboard2Half[retrieveIndexInDataAndMessage(
											(((xVal * 2 + 1) - 1)
													+ checkerboardAdjustment),
											yVal,
											widthLevelCheckerboardPart * 2,
											heightLevel, currentDisparity,
											NUM_POSSIBLE_DISPARITY_VALUES)]);
				}*//*
			}
		}
		else //checkerboardToUpdate == CHECKERBOARD_PART_2
		{
			half* messageLDeviceCurrentCheckerboard1Half = (half*)messageLDeviceCurrentCheckerboard1;
			half* messageRDeviceCurrentCheckerboard1Half = (half*)messageRDeviceCurrentCheckerboard1;

			for (int currentDisparity = 0;
					currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
					currentDisparity++)
			{
				dataMessage[currentDisparity] =
						dataCostStereoCheckerboard2[retrieveIndexInDataAndMessage(
								xVal, yVal, widthLevelCheckerboardPart,
								heightLevel, currentDisparity,
								NUM_POSSIBLE_DISPARITY_VALUES, offsetData)];
				prevUMessage[currentDisparity] =
						messageUDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
								xVal, (yVal + 1), widthLevelCheckerboardPart,
								heightLevel, currentDisparity,
								NUM_POSSIBLE_DISPARITY_VALUES)];
				prevDMessage[currentDisparity] =
						messageDDeviceCurrentCheckerboard1[retrieveIndexInDataAndMessage(
								xVal, (yVal - 1), widthLevelCheckerboardPart,
								heightLevel, currentDisparity,
								NUM_POSSIBLE_DISPARITY_VALUES)];
				prevLMessage[currentDisparity] =
						__halves2half2(
								messageLDeviceCurrentCheckerboard1Half[retrieveIndexInDataAndMessage(
										((xVal * 2)
												+ checkerboardAdjustment),
										yVal, widthLevelCheckerboardPart * 2,
										heightLevel, currentDisparity,
										NUM_POSSIBLE_DISPARITY_VALUES)],
								messageLDeviceCurrentCheckerboard1Half[retrieveIndexInDataAndMessage(
										((xVal * 2 + 1)
												+ checkerboardAdjustment),
										yVal, widthLevelCheckerboardPart * 2,
										heightLevel, currentDisparity,
										NUM_POSSIBLE_DISPARITY_VALUES)]);

				//if ((((xVal * 2) - 1) + checkerboardAdjustment) >= 0)
				{
					prevRMessage[currentDisparity] =
							__halves2half2(
									messageRDeviceCurrentCheckerboard1Half[retrieveIndexInDataAndMessage(
											(((xVal * 2) - 1)
													+ checkerboardAdjustment),
											yVal,
											widthLevelCheckerboardPart * 2,
											heightLevel, currentDisparity,
											NUM_POSSIBLE_DISPARITY_VALUES)],
									messageRDeviceCurrentCheckerboard1Half[retrieveIndexInDataAndMessage(
											(((xVal * 2 + 1) - 1)
													+ checkerboardAdjustment),
											yVal,
											widthLevelCheckerboardPart * 2,
											heightLevel, currentDisparity,
											NUM_POSSIBLE_DISPARITY_VALUES)]);
				}
				/*else
				{
					prevRMessage[currentDisparity] =
							__halves2half2((half) 0.0,
									messageRDeviceCurrentCheckerboard1Half[retrieveIndexInDataAndMessage(
											(((xVal * 2 + 1) - 1)
													+ checkerboardAdjustment),
											yVal,
											widthLevelCheckerboardPart * 2,
											heightLevel, currentDisparity,
											NUM_POSSIBLE_DISPARITY_VALUES)]);
				}*//*
			}
		}

		half2 currentUMessage[NUM_POSSIBLE_DISPARITY_VALUES];
		half2 currentDMessage[NUM_POSSIBLE_DISPARITY_VALUES];
		half2 currentLMessage[NUM_POSSIBLE_DISPARITY_VALUES];
		half2 currentRMessage[NUM_POSSIBLE_DISPARITY_VALUES];

		//uses the previous message values and data cost to calculate the current message values and store the results
		runBPIterationInOutDataInLocalMem<half2>(prevUMessage, prevDMessage, prevLMessage, prevRMessage, dataMessage,
							currentUMessage, currentDMessage, currentLMessage, currentRMessage, __float2half2_rn(disc_k_bp));

		//write the calculated message values to global memory
		for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
		{
			indexWriteTo = retrieveIndexInDataAndMessage(xVal, yVal, widthLevelCheckerboardPart, heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES);
			if (checkerboardToUpdate == CHECKERBOARD_PART_1)
			{
				messageUDeviceCurrentCheckerboard1[indexWriteTo] = currentUMessage[currentDisparity];
				messageDDeviceCurrentCheckerboard1[indexWriteTo] = currentDMessage[currentDisparity];
				messageLDeviceCurrentCheckerboard1[indexWriteTo] = currentLMessage[currentDisparity];
				messageRDeviceCurrentCheckerboard1[indexWriteTo] = currentRMessage[currentDisparity];
			}
			else //checkerboardToUpdate == CHECKERBOARD_PART_2
			{
				messageUDeviceCurrentCheckerboard2[indexWriteTo] = currentUMessage[currentDisparity];
				messageDDeviceCurrentCheckerboard2[indexWriteTo] = currentDMessage[currentDisparity];
				messageLDeviceCurrentCheckerboard2[indexWriteTo] = currentLMessage[currentDisparity];
				messageRDeviceCurrentCheckerboard2[indexWriteTo] = currentRMessage[currentDisparity];
			}
		}
	}
}
*/

//kernal function to run the current iteration of belief propagation in parallel using the checkerboard update method where half the pixels in the "checkerboard"
//scheme retrieve messages from each 4-connected neighbor and then update their message based on the retrieved messages and the data cost
template<typename T>
__global__ void runBPIterationUsingCheckerboardUpdatesNoTextures(int checkerboardPartUpdate, levelProperties& currentLevelProperties, T* dataCostStereoCheckerboard1, T* dataCostStereoCheckerboard2,
								T* messageUDeviceCurrentCheckerboard1, T* messageDDeviceCurrentCheckerboard1, T* messageLDeviceCurrentCheckerboard1, T* messageRDeviceCurrentCheckerboard1,
								T* messageUDeviceCurrentCheckerboard2, T* messageDDeviceCurrentCheckerboard2, T* messageLDeviceCurrentCheckerboard2,
								T* messageRDeviceCurrentCheckerboard2, float disc_k_bp)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_BP + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_BP + ty;

	if (withinImageBounds(xVal, yVal, currentLevelProperties.widthLevel/2, currentLevelProperties.heightLevel))
	{
		runBPIterationUsingCheckerboardUpdatesDeviceNoTexBoundAndLocalMem<T>(
				xVal, yVal, checkerboardPartUpdate, currentLevelProperties,
				dataCostStereoCheckerboard1, dataCostStereoCheckerboard2,
				messageUDeviceCurrentCheckerboard1,
				messageDDeviceCurrentCheckerboard1,
				messageLDeviceCurrentCheckerboard1,
				messageRDeviceCurrentCheckerboard1,
				messageUDeviceCurrentCheckerboard2,
				messageDDeviceCurrentCheckerboard2,
				messageLDeviceCurrentCheckerboard2,
				messageRDeviceCurrentCheckerboard2,
				disc_k_bp, 0);
	}
}


//kernal to copy the computed BP message values at the current level to the corresponding locations at the "next" level down
//the kernal works from the point of view of the pixel at the prev level that is being copied to four different places
template<typename T>
__global__ void copyPrevLevelToNextLevelBPCheckerboardStereoNoTextures(
		int checkerboardPart,
		levelProperties currentLevelProperties,
		levelProperties nextLevelProperties,
		T* messageUPrevStereoCheckerboard1, T* messageDPrevStereoCheckerboard1,
		T* messageLPrevStereoCheckerboard1, T* messageRPrevStereoCheckerboard1,
		T* messageUPrevStereoCheckerboard2, T* messageDPrevStereoCheckerboard2,
		T* messageLPrevStereoCheckerboard2, T* messageRPrevStereoCheckerboard2,
		T* messageUDeviceCurrentCheckerboard1,
		T* messageDDeviceCurrentCheckerboard1,
		T* messageLDeviceCurrentCheckerboard1,
		T* messageRDeviceCurrentCheckerboard1,
		T* messageUDeviceCurrentCheckerboard2,
		T* messageDDeviceCurrentCheckerboard2,
		T* messageLDeviceCurrentCheckerboard2,
		T* messageRDeviceCurrentCheckerboard2)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_BP + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_BP + ty;

	if (withinImageBounds(xVal, yVal, currentLevelProperties.widthCheckerboardLevel, currentLevelProperties.heightLevel))
	{
		int indexCopyTo;
		int indexCopyFrom;

		int checkerboardPartAdjustment;

		T prevValU;
		T prevValD;
		T prevValL;
		T prevValR;

		if (checkerboardPart == CHECKERBOARD_PART_1)
		{
			checkerboardPartAdjustment = (yVal%2);
		}
		else if (checkerboardPart == CHECKERBOARD_PART_2)
		{
			checkerboardPartAdjustment = ((yVal+1)%2);
		}

		for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
		{
			indexCopyFrom = retrieveIndexInDataAndMessage(xVal, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES);

			if (checkerboardPart == CHECKERBOARD_PART_1)
			{
				prevValU = messageUPrevStereoCheckerboard1[indexCopyFrom];
				prevValD = messageDPrevStereoCheckerboard1[indexCopyFrom];
				prevValL = messageLPrevStereoCheckerboard1[indexCopyFrom];
				prevValR = messageRPrevStereoCheckerboard1[indexCopyFrom];
			}
			else if (checkerboardPart == CHECKERBOARD_PART_2)
			{
				prevValU = messageUPrevStereoCheckerboard2[indexCopyFrom];
				prevValD = messageDPrevStereoCheckerboard2[indexCopyFrom];
				prevValL = messageLPrevStereoCheckerboard2[indexCopyFrom];
				prevValR = messageRPrevStereoCheckerboard2[indexCopyFrom];
			}

			if (withinImageBounds(xVal*2 + checkerboardPartAdjustment, yVal*2, nextLevelProperties.widthCheckerboardLevel, nextLevelProperties.heightLevel))
			{
				indexCopyTo = retrieveIndexInDataAndMessage(
						(xVal * 2 + checkerboardPartAdjustment), (yVal * 2),
						nextLevelProperties.paddedWidthCheckerboardLevel, nextLevelProperties.heightLevel,
						currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES);

				messageUDeviceCurrentCheckerboard1[indexCopyTo] = prevValU;
				messageDDeviceCurrentCheckerboard1[indexCopyTo] = prevValD;
				messageLDeviceCurrentCheckerboard1[indexCopyTo] = prevValL;
				messageRDeviceCurrentCheckerboard1[indexCopyTo] = prevValR;

				messageUDeviceCurrentCheckerboard2[indexCopyTo] = prevValU;
				messageDDeviceCurrentCheckerboard2[indexCopyTo] = prevValD;
				messageLDeviceCurrentCheckerboard2[indexCopyTo] = prevValL;
				messageRDeviceCurrentCheckerboard2[indexCopyTo] = prevValR;
			}

			if (withinImageBounds(xVal*2 + checkerboardPartAdjustment, yVal*2 + 1, nextLevelProperties.widthCheckerboardLevel,
					nextLevelProperties.heightLevel))
			{
				indexCopyTo = retrieveIndexInDataAndMessage(
						(xVal * 2 + checkerboardPartAdjustment), (yVal * 2 + 1),
						nextLevelProperties.paddedWidthCheckerboardLevel, nextLevelProperties.heightLevel,
						currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES);

				messageUDeviceCurrentCheckerboard1[indexCopyTo] = prevValU;
				messageDDeviceCurrentCheckerboard1[indexCopyTo] = prevValD;
				messageLDeviceCurrentCheckerboard1[indexCopyTo] = prevValL;
				messageRDeviceCurrentCheckerboard1[indexCopyTo] = prevValR;

				messageUDeviceCurrentCheckerboard2[indexCopyTo] = prevValU;
				messageDDeviceCurrentCheckerboard2[indexCopyTo] = prevValD;
				messageLDeviceCurrentCheckerboard2[indexCopyTo] = prevValL;
				messageRDeviceCurrentCheckerboard2[indexCopyTo] = prevValR;
			}
		}
	}
}


//retrieve the best disparity estimate from image 1 to image 2 for each pixel in parallel
template<typename T>
__global__ void retrieveOutputDisparityCheckerboardStereoOptimized(levelProperties currentLevelProperties, T* dataCostStereoCheckerboard1, T* dataCostStereoCheckerboard2, T* messageUPrevStereoCheckerboard1, T* messageDPrevStereoCheckerboard1, T* messageLPrevStereoCheckerboard1, T* messageRPrevStereoCheckerboard1, T* messageUPrevStereoCheckerboard2, T* messageDPrevStereoCheckerboard2, T* messageLPrevStereoCheckerboard2, T* messageRPrevStereoCheckerboard2, float* disparityBetweenImagesDevice)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_BP + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_BP + ty;

	if (withinImageBounds(xVal, yVal, currentLevelProperties.widthCheckerboardLevel, currentLevelProperties.heightLevel))
	{
		int xValInCheckerboardPart = xVal;

		//if (((yVal+xVal) % 2) == 0) //if true, then pixel is from part 1 of the checkerboard; otherwise, it's from part 2
		//first processing from first part of checkerboard
		{
			int	checkerboardPartAdjustment = (yVal%2);

			if (withinImageBounds(xValInCheckerboardPart*2 + checkerboardPartAdjustment, yVal, currentLevelProperties.widthLevel, currentLevelProperties.heightLevel))
			{
				if ((xValInCheckerboardPart >= (1 - checkerboardPartAdjustment)) && (xValInCheckerboardPart < (currentLevelProperties.widthCheckerboardLevel - checkerboardPartAdjustment)) && (yVal > 0) && (yVal < (currentLevelProperties.heightLevel - 1)))
				{
					// keep track of "best" disparity for current pixel
					int bestDisparity = 0;
					T best_val = INF_BP;
					for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
					{
						T val = messageUPrevStereoCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal + 1), currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
							 messageDPrevStereoCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal - 1), currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
							 messageLPrevStereoCheckerboard2[retrieveIndexInDataAndMessage((xValInCheckerboardPart + checkerboardPartAdjustment), yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
							 messageRPrevStereoCheckerboard2[retrieveIndexInDataAndMessage((xValInCheckerboardPart - 1 + checkerboardPartAdjustment), yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
							 dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboardPart, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)];

						if (val < (best_val)) {
							best_val = val;
							bestDisparity = currentDisparity;
						}
					}

					disparityBetweenImagesDevice[yVal*currentLevelProperties.widthLevel + (xValInCheckerboardPart*2 + checkerboardPartAdjustment)] = bestDisparity;
				}
				else
				{
					disparityBetweenImagesDevice[yVal*currentLevelProperties.widthLevel + (xValInCheckerboardPart*2 + checkerboardPartAdjustment)] = 0;
				}
			}
		}
		//process from part 2 of checkerboard
		{
			int	checkerboardPartAdjustment = ((yVal + 1) % 2);

			if (withinImageBounds(xValInCheckerboardPart*2 + checkerboardPartAdjustment, yVal, currentLevelProperties.widthLevel, currentLevelProperties.heightLevel))
			{
				if ((xValInCheckerboardPart >= (1 - checkerboardPartAdjustment)) && (xValInCheckerboardPart < (currentLevelProperties.widthCheckerboardLevel - checkerboardPartAdjustment)) && (yVal > 0) && (yVal < (currentLevelProperties.heightLevel - 1)))
				{
					// keep track of "best" disparity for current pixel
					int bestDisparity = 0;
					T best_val = INF_BP;
					for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
					{
						T val = messageUPrevStereoCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal + 1), currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
							messageDPrevStereoCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal - 1), currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
							messageLPrevStereoCheckerboard1[retrieveIndexInDataAndMessage((xValInCheckerboardPart + checkerboardPartAdjustment), yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
							messageRPrevStereoCheckerboard1[retrieveIndexInDataAndMessage((xValInCheckerboardPart - 1 + checkerboardPartAdjustment), yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
							dataCostStereoCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboardPart, yVal, currentLevelProperties.paddedWidthCheckerboardLevel, currentLevelProperties.heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)];

						if (val < (best_val))
						{
							best_val = val;
							bestDisparity = currentDisparity;
						}
					}
					disparityBetweenImagesDevice[yVal*currentLevelProperties.widthLevel + (xValInCheckerboardPart*2 + checkerboardPartAdjustment)] = bestDisparity;
				}
				else
				{
					disparityBetweenImagesDevice[yVal*currentLevelProperties.widthLevel + (xValInCheckerboardPart*2 + checkerboardPartAdjustment)] = 0;
				}
			}
		}
	}
}

//retrieve the best disparity estimate from image 1 to image 2 for each pixel in parallel
template<>
__global__ void retrieveOutputDisparityCheckerboardStereoOptimized<half2>(levelProperties currentLevelProperties, half2* dataCostStereoCheckerboard1, half2* dataCostStereoCheckerboard2, half2* messageUPrevStereoCheckerboard1, half2* messageDPrevStereoCheckerboard1, half2* messageLPrevStereoCheckerboard1, half2* messageRPrevStereoCheckerboard1, half2* messageUPrevStereoCheckerboard2, half2* messageDPrevStereoCheckerboard2, half2* messageLPrevStereoCheckerboard2, half2* messageRPrevStereoCheckerboard2, float* disparityBetweenImagesDevice)
{

}

//retrieve the best disparity estimate from image 1 to image 2 for each pixel in parallel
/*template<typename T>
__global__ void retrieveOutputDisparityCheckerboardStereoNoTextures(T* dataCostStereoCheckerboard1, T* dataCostStereoCheckerboard2, T* messageUPrevStereoCheckerboard1, T* messageDPrevStereoCheckerboard1, T* messageLPrevStereoCheckerboard1, T* messageRPrevStereoCheckerboard1, T* messageUPrevStereoCheckerboard2, T* messageDPrevStereoCheckerboard2, T* messageLPrevStereoCheckerboard2, T* messageRPrevStereoCheckerboard2, float* disparityBetweenImagesDevice, int widthLevel, int heightLevel)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_BP + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_BP + ty;

	if (withinImageBounds(xVal, yVal, widthLevel, heightLevel))
	{
		int widthCheckerboard = getCheckerboardWidth<T>(widthLevel);
		int xValInCheckerboardPart = xVal/2;

		if (((yVal+xVal) % 2) == 0) //if true, then pixel is from part 1 of the checkerboard; otherwise, it's from part 2
		{
			int	checkerboardPartAdjustment = (yVal%2);

			if ((xVal >= 1) && (xVal < (widthLevel - 1)) && (yVal >= 1) && (yVal < (heightLevel - 1)))
			{
				// keep track of "best" disparity for current pixel
				int bestDisparity = 0;
				T best_val = INF_BP;
				for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
				{
					T val = messageUPrevStereoCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal + 1), widthCheckerboard, heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
						 messageDPrevStereoCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal - 1), widthCheckerboard, heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
						 messageLPrevStereoCheckerboard2[retrieveIndexInDataAndMessage((xValInCheckerboardPart + checkerboardPartAdjustment), yVal, widthCheckerboard, heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
						 messageRPrevStereoCheckerboard2[retrieveIndexInDataAndMessage((xValInCheckerboardPart - 1 + checkerboardPartAdjustment), yVal, widthCheckerboard, heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
						 dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboardPart, yVal, widthCheckerboard, heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)];

					if (val < (best_val)) {
						best_val = val;
						bestDisparity = currentDisparity;
					}
				}
				disparityBetweenImagesDevice[yVal*widthLevel + xVal] = bestDisparity;
			}
			else
			{
				disparityBetweenImagesDevice[yVal*widthLevel + xVal] = 0;
			}
		}
		else //pixel from part 2 of checkerboard
		{
			int	checkerboardPartAdjustment = ((yVal + 1) % 2);

			if ((xVal >= 1) && (xVal < (widthLevel - 1)) && (yVal >= 1) && (yVal < (heightLevel - 1)))
			{


				// keep track of "best" disparity for current pixel
				int bestDisparity = 0;
				T best_val = INF_BP;
				for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
				{
					T val = messageUPrevStereoCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal + 1), widthCheckerboard, heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
						messageDPrevStereoCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal - 1), widthCheckerboard, heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
						messageLPrevStereoCheckerboard1[retrieveIndexInDataAndMessage((xValInCheckerboardPart + checkerboardPartAdjustment), yVal, widthCheckerboard, heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
						messageRPrevStereoCheckerboard1[retrieveIndexInDataAndMessage((xValInCheckerboardPart - 1 + checkerboardPartAdjustment), yVal, widthCheckerboard, heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)] +
						dataCostStereoCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboardPart, yVal, widthCheckerboard, heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)];

					if (val < (best_val))
					{
						best_val = val;
						bestDisparity = currentDisparity;
					}
				}
				disparityBetweenImagesDevice[yVal*widthLevel + xVal] = bestDisparity;
			}
			else
			{
				disparityBetweenImagesDevice[yVal*widthLevel + xVal] = 0;
			}
		}
	}
}

//retrieve the best disparity estimate from image 1 to image 2 for each pixel in parallel
template<>
__global__ void retrieveOutputDisparityCheckerboardStereoNoTextures<half2>(half2* dataCostStereoCheckerboard1, half2* dataCostStereoCheckerboard2, half2* messageUPrevStereoCheckerboard1, half2* messageDPrevStereoCheckerboard1, half2* messageLPrevStereoCheckerboard1, half2* messageRPrevStereoCheckerboard1, half2* messageUPrevStereoCheckerboard2, half2* messageDPrevStereoCheckerboard2, half2* messageLPrevStereoCheckerboard2, half2* messageRPrevStereoCheckerboard2, float* disparityBetweenImagesDevice, int widthLevel, int heightLevel)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_BP + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_BP + ty;

	if (withinImageBounds(xVal*2, yVal, widthLevel, heightLevel))
	{
		int widthCheckerboard = getCheckerboardWidth<half2>(widthLevel);
		int xValInCheckerboardPart = xVal/2;

		if (((yVal+xVal) % 2) == 0) //if true, then pixel is from part 1 of the checkerboard; otherwise, it's from part 2
		{
			int	checkerboardPartAdjustment = (yVal%2);

			half* messageLPrevStereoCheckerboard2Half = (half*)messageLPrevStereoCheckerboard2;
			half* messageRPrevStereoCheckerboard2Half = (half*)messageRPrevStereoCheckerboard2;

			if ((xVal >= 1) && (xVal < (widthLevel - 1)) && (yVal >= 1) && (yVal < (heightLevel - 1)))
			{
				// keep track of "best" disparity for current pixel
				int bestDisparity1 = 0;
				int bestDisparity2 = 0;
				float best_val1 = INF_BP;
				float best_val2 = INF_BP;
				for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
				{
					half2 val = __hadd2(messageUPrevStereoCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal + 1), widthCheckerboard, heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)],
											 messageDPrevStereoCheckerboard2[retrieveIndexInDataAndMessage(xValInCheckerboardPart, (yVal - 1), widthCheckerboard, heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);
					val =
							__hadd2(val,
									__halves2half2(
											messageLPrevStereoCheckerboard2Half[retrieveIndexInDataAndMessage(
													((xValInCheckerboardPart * 2)
															+ checkerboardPartAdjustment),
													yVal, widthCheckerboard * 2,
													heightLevel,
													currentDisparity,
													NUM_POSSIBLE_DISPARITY_VALUES)],
									messageLPrevStereoCheckerboard2Half[retrieveIndexInDataAndMessage(
											((xValInCheckerboardPart * 2 + 1)
													+ checkerboardPartAdjustment),
											yVal, widthCheckerboard * 2,
											heightLevel, currentDisparity,
											NUM_POSSIBLE_DISPARITY_VALUES)]));
					val =
							__hadd2(val,
									__halves2half2(
											messageRPrevStereoCheckerboard2Half[retrieveIndexInDataAndMessage(
													((xValInCheckerboardPart * 2)
															- 1
															+ checkerboardPartAdjustment),
													yVal, widthCheckerboard * 2,
													heightLevel,
													currentDisparity,
													NUM_POSSIBLE_DISPARITY_VALUES)],
									messageRPrevStereoCheckerboard2Half[retrieveIndexInDataAndMessage(
											((xValInCheckerboardPart * 2 + 1)
													- 1
													+ checkerboardPartAdjustment),
											yVal, widthCheckerboard * 2,
											heightLevel, currentDisparity,
											NUM_POSSIBLE_DISPARITY_VALUES)]));
					val = __hadd2(val, dataCostStereoCheckerboard1[retrieveIndexInDataAndMessage(xValInCheckerboardPart, yVal, widthCheckerboard, heightLevel, currentDisparity, NUM_POSSIBLE_DISPARITY_VALUES)]);

					float valLow = __low2float ( val);
					float valHigh = __high2float ( val);
					if (valLow < best_val1)
					{
						best_val1 = valLow;
						bestDisparity1 = currentDisparity;
					}
					if (valHigh < best_val2)
					{
						best_val2 = valHigh;
						bestDisparity2 = currentDisparity;
					}
				}
				disparityBetweenImagesDevice[yVal*widthLevel + (xVal*2 - checkerboardPartAdjustment)] = bestDisparity1;
				if (((xVal*2) + 2) < widthLevel)
				{
					disparityBetweenImagesDevice[yVal*widthLevel + (xVal*2 - checkerboardPartAdjustment) + 2] = bestDisparity2;
				}
			}
			else
			{
				disparityBetweenImagesDevice[yVal * widthLevel + (xVal * 2 - checkerboardPartAdjustment)] =
						0;
				if (((xVal * 2) + 2) < widthLevel)
				{
					disparityBetweenImagesDevice[yVal * widthLevel + (xVal * 2 - checkerboardPartAdjustment)
							+ 2] = 0;
				}
			}
		}
		else //pixel from part 2 of checkerboard
		{
			int	checkerboardPartAdjustment = ((yVal + 1) % 2);
			half* messageLPrevStereoCheckerboard1Half = (half*)messageLPrevStereoCheckerboard1;
			half* messageRPrevStereoCheckerboard1Half = (half*)messageRPrevStereoCheckerboard1;

			if ((xVal >= 1) && (xVal < (widthLevel - 1)) && (yVal >= 1) && (yVal < (heightLevel - 1)))
			{
				// keep track of "best" disparity for current pixel
				int bestDisparity1 = 0;
				int bestDisparity2 = 0;
				float best_val1 = INF_BP;
				float best_val2 = INF_BP;
				for (int currentDisparity = 0; currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES; currentDisparity++)
				{
					half2 val =
							__hadd2(
									messageUPrevStereoCheckerboard1[retrieveIndexInDataAndMessage(
											xValInCheckerboardPart, (yVal + 1),
											widthCheckerboard, heightLevel,
											currentDisparity,
											NUM_POSSIBLE_DISPARITY_VALUES)],
											messageDPrevStereoCheckerboard1[retrieveIndexInDataAndMessage(
													xValInCheckerboardPart,
													(yVal - 1),
													widthCheckerboard,
													heightLevel,
													currentDisparity,
													NUM_POSSIBLE_DISPARITY_VALUES)]);
					val =
							__hadd2(val,
									__halves2half2(
											messageLPrevStereoCheckerboard1Half[retrieveIndexInDataAndMessage(
													((xValInCheckerboardPart * 2)
															+ checkerboardPartAdjustment),
													yVal, widthCheckerboard * 2,
													heightLevel,
													currentDisparity,
													NUM_POSSIBLE_DISPARITY_VALUES)],
									messageLPrevStereoCheckerboard1Half[retrieveIndexInDataAndMessage(
											((xValInCheckerboardPart * 2 + 1)
													+ checkerboardPartAdjustment),
											yVal, widthCheckerboard * 2,
											heightLevel, currentDisparity,
											NUM_POSSIBLE_DISPARITY_VALUES)]));
					val =
							__hadd2(val,
									__halves2half2(
											messageRPrevStereoCheckerboard1Half[retrieveIndexInDataAndMessage(
													((xValInCheckerboardPart * 2)
															- 1
															+ checkerboardPartAdjustment),
													yVal, widthCheckerboard * 2,
													heightLevel,
													currentDisparity,
													NUM_POSSIBLE_DISPARITY_VALUES)],
									messageRPrevStereoCheckerboard1Half[retrieveIndexInDataAndMessage(
											((xValInCheckerboardPart * 2 + 1)
													- 1
													+ checkerboardPartAdjustment),
											yVal, widthCheckerboard * 2,
											heightLevel, currentDisparity,
											NUM_POSSIBLE_DISPARITY_VALUES)]));

					val =
							__hadd2(val,
									dataCostStereoCheckerboard2[retrieveIndexInDataAndMessage(
											xValInCheckerboardPart, yVal,
											widthCheckerboard, heightLevel,
											currentDisparity,
											NUM_POSSIBLE_DISPARITY_VALUES)]);

					float val1 = __low2float(val);
					float val2 = __high2float(val);
					if (val1 < best_val1) {
						best_val1 = val1;
						bestDisparity1 = currentDisparity;
					}
					if (val2 < best_val2) {
						best_val2 = val2;
						bestDisparity2 = currentDisparity;
					}
				}

				disparityBetweenImagesDevice[yVal * widthLevel + (xVal * 2 - checkerboardPartAdjustment)] =
						bestDisparity1;
				if (((xVal * 2) + 2) < widthLevel) {
					disparityBetweenImagesDevice[yVal * widthLevel + (xVal * 2 - checkerboardPartAdjustment)
							+ 2] = bestDisparity2;
				}
			}
			else
			{
				disparityBetweenImagesDevice[yVal * widthLevel + (xVal * 2 - checkerboardPartAdjustment)] =
						0;
				if (((xVal * 2) + 2) < widthLevel) {
					disparityBetweenImagesDevice[yVal * widthLevel + (xVal * 2 - checkerboardPartAdjustment)
							+ 2] = 0;
				}
			}
		}
	}
}
*/

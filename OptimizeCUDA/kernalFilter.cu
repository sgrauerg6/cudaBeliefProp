#include "hip/hip_runtime.h"
/*
Copyright (C) 2009 Scott Grauer-Gray, Chandra Kambhamettu, and Kannappan Palaniappan

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307 USA
*/

//This kernal is used to filter the image with the given filter in the vertical and horizontal directions


#include "kernalFilterHeader.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "bpStereoCudaParameters.h"

#define PROCESSING_ON_GPU
#include "../SharedFuncts/SharedSmoothImageFuncts.h"
#undef PROCESSING_ON_GPU

//checks if the current point is within the image bounds
__device__ bool withinImageBoundsFilter(int xVal, int yVal, int width, int height)
{
	return ((xVal >= 0) && (xVal < width) && (yVal >= 0) && (yVal < height));
}


//kernal to convert the unsigned int pixels to float pixels in an image when
//smoothing is not desired but the pixels need to be converted to floats
//the input image is stored as unsigned ints in the texture imagePixelsUnsignedIntToFilterTexture
//output filtered image stored in floatImagePixels
__global__ void convertUnsignedIntImageToFloat(unsigned int* imagePixelsUnsignedIntToFilter, float* floatImagePixels, int widthImages, int heightImages)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_FILTER_IMAGES + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_FILTER_IMAGES + ty;

	//make sure that (xVal, yVal) is within image bounds
	if (withinImageBoundsFilter(xVal, yVal, widthImages, heightImages))
	{
		//retrieve the float-value of the unsigned int pixel value at the current location
		float floatPixelVal = 1.0f * imagePixelsUnsignedIntToFilter[yVal*widthImages + xVal];

		floatImagePixels[yVal*widthImages + xVal] = floatPixelVal;
	}
}


//kernal to apply a horizontal filter on each pixel of the image in parallel
//input image stored in texture imagePixelsFloatToFilterTexture
//output filtered image stored in filteredImagePixels
template<typename T>
__global__ void filterImageAcross(T* imagePixelsToFilter, float* filteredImagePixels, int widthImages, int heightImages, float* imageFilter, int sizeFilter)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_FILTER_IMAGES + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_FILTER_IMAGES + ty;

	//make sure that (xVal, yVal) is within image bounds
	if (withinImageBoundsFilter(xVal, yVal, widthImages, heightImages))
	{
		filterImageAcrossProcessPixel<T>(xVal, yVal,
				imagePixelsToFilter,
				filteredImagePixels, widthImages, heightImages,
				imageFilter, sizeFilter);
	}
}


//kernal to apply a vertical filter on each pixel of the image in parallel
//input image stored in texture imagePixelsFloatToFilterTexture
//output filtered image stored in filteredImagePixels
template<typename T>
__global__ void filterImageVertical(T* imagePixelsToFilter, float* filteredImagePixels, int widthImages, int heightImages, float* imageFilter, int sizeFilter)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_FILTER_IMAGES + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_FILTER_IMAGES + ty;

	//make sure that (xVal, yVal) is within image bounds
	if (withinImageBoundsFilter(xVal, yVal, widthImages, heightImages))
	{
		filterImageVerticalProcessPixel<T>(xVal, yVal,
				imagePixelsToFilter,
				filteredImagePixels, widthImages, heightImages,
				imageFilter, sizeFilter);
	}
}

#include "hip/hip_runtime.h"
//code for using shared memory in the belief prop function; seems to work but is generally slower than not using shared memory,
//so currently not using except for testing

#define PROCESSING_ON_GPU
#include "../../SharedFuncts/SharedBPProcessingFuncts.h"
#include "../../bpStereoCudaParameters.h"
#undef PROCESSING_ON_GPU


//function retrieve the minimum value at each 1-d disparity value in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
template<typename T>
__device__ inline void dtStereoSharedMemory(T* dstShared) {
	T prev;
	unsigned int startIndexDstShared = threadIdx.y * BLOCK_SIZE_WIDTH_BP + threadIdx.x;
	unsigned int indexIndexDstShared = startIndexDstShared;

//#pragma unroll 64
	for (unsigned int currentDisparity = 1;
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++) {
		prev = dstShared[indexIndexDstShared] + (T) 1.0;
		indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
		if (prev < dstShared[indexIndexDstShared]) {
			dstShared[indexIndexDstShared] = prev;
		}
	}

//#pragma unroll 64
	for (unsigned int currentDisparity = NUM_POSSIBLE_DISPARITY_VALUES - 2;
			currentDisparity >= 0; currentDisparity--) {
		prev = dstShared[indexIndexDstShared] + (T) 1.0;
		indexIndexDstShared -= BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
		if (prev < dstShared[indexIndexDstShared]) {
			dstShared[indexIndexDstShared] = prev;
		}
	}
}

template<>
__device__ inline void dtStereoSharedMemory<half>(half* dstShared) {

	int halfIndexSharedVals[2] = {1, (2*BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP)-1};
	int indexIntervalNextHalfIndexSharedVals = 0;

	half prev;
	unsigned int startIndexDstShared = 2*(threadIdx.y * BLOCK_SIZE_WIDTH_BP + threadIdx.x);
	unsigned int indexIndexDstShared = startIndexDstShared;

//#pragma unroll 64
	for (unsigned int currentDisparity = 1;
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++) {
		prev = dstShared[indexIndexDstShared] + (half) 1.0;
		indexIndexDstShared += halfIndexSharedVals[indexIntervalNextHalfIndexSharedVals];
		indexIntervalNextHalfIndexSharedVals = !indexIntervalNextHalfIndexSharedVals;
		//indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
		if (prev < dstShared[indexIndexDstShared]) {
			dstShared[indexIndexDstShared] = prev;
		}
	}

//#pragma unroll 64
	for (unsigned int currentDisparity = NUM_POSSIBLE_DISPARITY_VALUES - 2;
			currentDisparity >= 0; currentDisparity--) {
		prev = dstShared[indexIndexDstShared] + (half) 1.0;
		indexIntervalNextHalfIndexSharedVals = !indexIntervalNextHalfIndexSharedVals;
		indexIndexDstShared -= halfIndexSharedVals[indexIntervalNextHalfIndexSharedVals];
		//indexIndexDstShared -= BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
		if (prev < dstShared[indexIndexDstShared]) {
			dstShared[indexIndexDstShared] = prev;
		}
	}
}

template<typename T>
__device__ inline void dtStereoSharedAndRegLocalMemory(T* dstShared, T* dst) {
	T prev;
	T lastVal;

#if DISP_INDEX_START_REG_LOCAL_MEM > 0
	unsigned int startIndexDstShared = threadIdx.y * BLOCK_SIZE_WIDTH_BP + threadIdx.x;
	unsigned int indexIndexDstShared = startIndexDstShared;
	lastVal = dstShared[indexIndexDstShared];
#else
	lastVal= dst[0];
#endif
#if DISP_INDEX_START_REG_LOCAL_MEM > 0
//#pragma unroll 64
	for (unsigned int currentDisparity = 1;
			currentDisparity < DISP_INDEX_START_REG_LOCAL_MEM;
			currentDisparity++) {
		prev = lastVal + (T) 1.0;
		indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
		if (prev < dstShared[indexIndexDstShared]) {
			dstShared[indexIndexDstShared] = prev;
		}
		lastVal = dstShared[indexIndexDstShared];
	}
#endif
#pragma unroll
	for (unsigned int currentDisparity = getMax(1, DISP_INDEX_START_REG_LOCAL_MEM);
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++) {
		prev = lastVal + (T) 1.0;
		if (prev < dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM]) {
			dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM] = prev;
		}
		lastVal = dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM];
	}

//#pragma unroll 64
	for (unsigned int currentDisparity = NUM_POSSIBLE_DISPARITY_VALUES - 2;
			currentDisparity >= DISP_INDEX_START_REG_LOCAL_MEM;
			currentDisparity--) {
		prev = lastVal + (T) 1.0;
		if (prev < dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM]) {
			dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM] = prev;
		}
		lastVal = dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM];
	}
#if DISP_INDEX_START_REG_LOCAL_MEM > 0
//#pragma unroll 64
	for (unsigned int currentDisparity = getMin(NUM_POSSIBLE_DISPARITY_VALUES - 2,
			DISP_INDEX_START_REG_LOCAL_MEM - 1); currentDisparity >= 0;
			currentDisparity--) {
		prev = lastVal + (T) 1.0;
		if (prev < dstShared[indexIndexDstShared]) {
			dstShared[indexIndexDstShared] = prev;
		}
		lastVal = dstShared[indexIndexDstShared];
		indexIndexDstShared -= BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
	}
#endif
}

template<>
__device__ inline void dtStereoSharedAndRegLocalMemory<half>(half* dstShared, half* dst)
{
	int halfIndexSharedVals[2] = {1, (2*BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP)-1};
	int indexIntervalNextHalfIndexSharedVals = 0;

	half prev;
	half lastVal;

#if DISP_INDEX_START_REG_LOCAL_MEM > 0
	unsigned int startIndexDstShared = 2*(threadIdx.y * BLOCK_SIZE_WIDTH_BP + threadIdx.x);
	unsigned int indexIndexDstShared = startIndexDstShared;
	lastVal = dstShared[indexIndexDstShared];
#else
	lastVal= dst[0];
#endif
#if DISP_INDEX_START_REG_LOCAL_MEM > 0
//#pragma unroll 64
	for (unsigned int currentDisparity = 1;
			currentDisparity < DISP_INDEX_START_REG_LOCAL_MEM;
			currentDisparity++) {
		prev = lastVal + (half) 1.0;
		indexIndexDstShared += halfIndexSharedVals[indexIntervalNextHalfIndexSharedVals];
		indexIntervalNextHalfIndexSharedVals = !indexIntervalNextHalfIndexSharedVals;
		//indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
		if (prev < dstShared[indexIndexDstShared]) {
			dstShared[indexIndexDstShared] = prev;
		}
		lastVal = dstShared[indexIndexDstShared];
	}
#endif
#pragma unroll
	for (unsigned int currentDisparity = getMax(1, DISP_INDEX_START_REG_LOCAL_MEM);
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++) {
		prev = lastVal + (half) 1.0;
		if (prev < dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM]) {
			dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM] = prev;
		}
		lastVal = dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM];
	}

//#pragma unroll 64
	for (unsigned int currentDisparity = NUM_POSSIBLE_DISPARITY_VALUES - 2;
			currentDisparity >= DISP_INDEX_START_REG_LOCAL_MEM;
			currentDisparity--) {
		prev = lastVal + (half) 1.0;
		if (prev < dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM]) {
			dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM] = prev;
		}
		lastVal = dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM];
	}
#if DISP_INDEX_START_REG_LOCAL_MEM > 0
//#pragma unroll 64
	for (unsigned int currentDisparity = getMin(NUM_POSSIBLE_DISPARITY_VALUES - 2,
			DISP_INDEX_START_REG_LOCAL_MEM - 1); currentDisparity >= 0;
			currentDisparity--) {
		prev = lastVal + (half) 1.0;
		if (prev < dstShared[indexIndexDstShared]) {
			dstShared[indexIndexDstShared] = prev;
		}
		lastVal = dstShared[indexIndexDstShared];
		indexIntervalNextHalfIndexSharedVals = !indexIntervalNextHalfIndexSharedVals;
		indexIndexDstShared -= halfIndexSharedVals[indexIntervalNextHalfIndexSharedVals];
		//indexIndexDstShared -= BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
	}
#endif
}


template<>
__device__ inline void msgStereo<float, float>(int xVal, int yVal,
		beliefprop::levelProperties& currentLevelProperties,
		float messageValsNeighbor1[NUM_POSSIBLE_DISPARITY_VALUES],
		float messageValsNeighbor2[NUM_POSSIBLE_DISPARITY_VALUES],
		float messageValsNeighbor3[NUM_POSSIBLE_DISPARITY_VALUES],
		float dataCosts[NUM_POSSIBLE_DISPARITY_VALUES], float* dstMessageArray,
		float disc_k_bp, bool dataAligned)
{
	//printf("USED SHARED MEMORY\n");
	// aggregate and find min
	float minimum = INF_BP;

#if DISP_INDEX_START_REG_LOCAL_MEM == 0
	float* dstSharedMem = nullptr;
#else
	__shared__ float dstSharedMem[BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP
			* (NUM_POSSIBLE_DISPARITY_VALUES - (NUM_POSSIBLE_DISPARITY_VALUES - DISP_INDEX_START_REG_LOCAL_MEM))];
#endif

	unsigned int startIndexDstShared = threadIdx.y * BLOCK_SIZE_WIDTH_BP + threadIdx.x;
	unsigned int indexIndexDstShared = startIndexDstShared;

#if DISP_INDEX_START_REG_LOCAL_MEM >= NUM_POSSIBLE_DISPARITY_VALUES
//#pragma unroll 64
	for (unsigned int currentDisparity = 0;
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++)
	{
		dstSharedMem[indexIndexDstShared] =
		messageValsNeighbor1[currentDisparity]
		+ messageValsNeighbor2[currentDisparity]
		+ messageValsNeighbor3[currentDisparity]
		+ dataCosts[currentDisparity];
		if (dstSharedMem[indexIndexDstShared] < minimum)
		{
			minimum = dstSharedMem[indexIndexDstShared];
		}
		indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
	}

	indexIndexDstShared = startIndexDstShared;

//#pragma unroll 64
	for (unsigned int currentDisparity = 0;
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++)
	{
		dstSharedMem[indexIndexDstShared] =
		messageValsNeighbor1[currentDisparity]
		+ messageValsNeighbor2[currentDisparity]
		+ messageValsNeighbor3[currentDisparity]
		+ dataCosts[currentDisparity];
		if (dstSharedMem[indexIndexDstShared] < minimum)
		{
			minimum = dstSharedMem[indexIndexDstShared];
		}
		indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
//#if (NUM_POSSIBLE_DISPARITY_VALUES - 1) <= DISPARITY_START_SHARED_MEM //no shared memory used
//	dtStereo<float>(dst);
//#else
	dtStereoSharedMemory<float>(dstSharedMem);
//#endif

	// truncate
	minimum += disc_k_bp;

	// normalize
	float valToNormalize = 0.0f;
	indexIndexDstShared = startIndexDstShared;

//#pragma unroll 64
	for (unsigned int currentDisparity = 0;
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++)
	{
		if (minimum < dstSharedMem[indexIndexDstShared])
		{
			dstSharedMem[indexIndexDstShared] = minimum;
		}

		valToNormalize += dstSharedMem[indexIndexDstShared];
		indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
	}

	valToNormalize /= ((float) NUM_POSSIBLE_DISPARITY_VALUES);

	unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
			currentLevelProperties.paddedWidthCheckerboardLevel_,
			currentLevelProperties.heightLevel_, 0,
			NUM_POSSIBLE_DISPARITY_VALUES);

	indexIndexDstShared = startIndexDstShared;
//#pragma unroll 64
	for (unsigned int currentDisparity = 0;
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++)
	{
		dstSharedMem[indexIndexDstShared] -= valToNormalize;
		dstMessageArray[destMessageArrayIndex] = dstSharedMem[indexIndexDstShared];
		indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;

#if beliefprop::OPTIMIZED_INDEXING_SETTING == 1
		destMessageArrayIndex +=
		currentLevelProperties.paddedWidthCheckerboardLevel_;
#else
		destMessageArrayIndex++;
#endif //beliefprop::OPTIMIZED_INDEXING_SETTING == 1
	}
#else
	float dst[NUM_POSSIBLE_DISPARITY_VALUES - DISP_INDEX_START_REG_LOCAL_MEM];

#if DISP_INDEX_START_REG_LOCAL_MEM > 0
//#pragma unroll 64
	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_INDEX_START_REG_LOCAL_MEM;
			currentDisparity++) {
		dstSharedMem[indexIndexDstShared] =
				messageValsNeighbor1[currentDisparity]
						+ messageValsNeighbor2[currentDisparity]
						+ messageValsNeighbor3[currentDisparity]
						+ dataCosts[currentDisparity];
		if (dstSharedMem[indexIndexDstShared] < minimum) {
			minimum = dstSharedMem[indexIndexDstShared];
		}
		indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
	}
#endif

//#pragma unroll 64
	for (unsigned int currentDisparity = DISP_INDEX_START_REG_LOCAL_MEM;
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++) {
		dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM] =
				messageValsNeighbor1[currentDisparity]
						+ messageValsNeighbor2[currentDisparity]
						+ messageValsNeighbor3[currentDisparity]
						+ dataCosts[currentDisparity];
		if (dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM] < minimum) {
			minimum = dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM];
		}
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
//#if (NUM_POSSIBLE_DISPARITY_VALUES - 1) <= DISPARITY_START_SHARED_MEM //no shared memory used
//	dtStereo<float>(dst);
//#else
	dtStereoSharedAndRegLocalMemory<float>(dstSharedMem, dst);
//#endif

	// truncate
	minimum += disc_k_bp;

	// normalize
	float valToNormalize = 0.0f;

#if DISP_INDEX_START_REG_LOCAL_MEM > 0
	indexIndexDstShared = startIndexDstShared;
//#pragma unroll 64
	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_INDEX_START_REG_LOCAL_MEM;
			currentDisparity++) {
		if (minimum < dstSharedMem[indexIndexDstShared]) {
			dstSharedMem[indexIndexDstShared] = minimum;
		}

		valToNormalize += dstSharedMem[indexIndexDstShared];
		indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
	}
#endif
//#pragma unroll 64
	for (unsigned int currentDisparity = DISP_INDEX_START_REG_LOCAL_MEM;
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++) {
		if (minimum < dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM]) {
			dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM] = minimum;
		}

		valToNormalize +=
				dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM];
	}

	valToNormalize /= ((float) NUM_POSSIBLE_DISPARITY_VALUES);

	unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
			currentLevelProperties.paddedWidthCheckerboardLevel_,
			currentLevelProperties.heightLevel_, 0,
			NUM_POSSIBLE_DISPARITY_VALUES);

#if DISP_INDEX_START_REG_LOCAL_MEM > 0
	indexIndexDstShared = startIndexDstShared;
//#pragma unroll 64
	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_INDEX_START_REG_LOCAL_MEM;
			currentDisparity++) {
		dstSharedMem[indexIndexDstShared] -= valToNormalize;
		dstMessageArray[destMessageArrayIndex] =
				dstSharedMem[indexIndexDstShared];
		indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;

#if beliefprop::OPTIMIZED_INDEXING_SETTING == 1
		destMessageArrayIndex +=
		currentLevelProperties.paddedWidthCheckerboardLevel_;
#else
		destMessageArrayIndex++;
#endif //beliefprop::OPTIMIZED_INDEXING_SETTING == 1
	}
#endif
//#pragma unroll 64
	for (unsigned int currentDisparity = DISP_INDEX_START_REG_LOCAL_MEM;
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++) {
		dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM] -=
				valToNormalize;
		dstMessageArray[destMessageArrayIndex] = dst[currentDisparity
				- DISP_INDEX_START_REG_LOCAL_MEM];

#if beliefprop::OPTIMIZED_INDEXING_SETTING == 1
		destMessageArrayIndex +=
		currentLevelProperties.paddedWidthCheckerboardLevel_;
#else
		destMessageArrayIndex++;
#endif //beliefprop::OPTIMIZED_INDEXING_SETTING == 1
	}

#endif
}

template<>
__device__ inline void msgStereo<half, half>(int xVal, int yVal,
		beliefprop::levelProperties& currentLevelProperties,
		half messageValsNeighbor1[NUM_POSSIBLE_DISPARITY_VALUES],
		half messageValsNeighbor2[NUM_POSSIBLE_DISPARITY_VALUES],
		half messageValsNeighbor3[NUM_POSSIBLE_DISPARITY_VALUES],
		half dataCosts[NUM_POSSIBLE_DISPARITY_VALUES], half* dstMessageArray,
		half disc_k_bp, bool dataAligned)
{
	int halfIndexSharedVals[2] = {1, (2*BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP)-1};
	int indexIntervalNextHalfIndexSharedVals = 0;
	//printf("USED SHARED MEMORY\n");
	// aggregate and find min
	half minimum = INF_BP;

#if DISP_INDEX_START_REG_LOCAL_MEM == 0
	half* dstSharedMem = nullptr;
#else
	__shared__ half dstSharedMem[BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP
			* (NUM_POSSIBLE_DISPARITY_VALUES - (NUM_POSSIBLE_DISPARITY_VALUES - DISP_INDEX_START_REG_LOCAL_MEM) + ((NUM_POSSIBLE_DISPARITY_VALUES - DISP_INDEX_START_REG_LOCAL_MEM) % 2))];
#endif

	unsigned int startIndexDstShared = 2*(threadIdx.y * BLOCK_SIZE_WIDTH_BP + threadIdx.x);
	unsigned int indexIndexDstShared = startIndexDstShared;

#if DISP_INDEX_START_REG_LOCAL_MEM >= NUM_POSSIBLE_DISPARITY_VALUES
//#pragma unroll 64
	for (unsigned int currentDisparity = 0;
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++)
	{
		dstSharedMem[indexIndexDstShared] =
		messageValsNeighbor1[currentDisparity]
		+ messageValsNeighbor2[currentDisparity]
		+ messageValsNeighbor3[currentDisparity]
		+ dataCosts[currentDisparity];
		if (dstSharedMem[indexIndexDstShared] < minimum)
		{
			minimum = dstSharedMem[indexIndexDstShared];
		}
		indexIndexDstShared += halfIndexSharedVals[indexIntervalNextHalfIndexSharedVals];
		indexIntervalNextHalfIndexSharedVals = (indexIntervalNextHalfIndexSharedVals + 1) % 2;
		//indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
	}

	indexIndexDstShared = startIndexDstShared;
	indexIntervalNextHalfIndexSharedVals = 0;

//#pragma unroll 64
	for (unsigned int currentDisparity = 0;
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++)
	{
		dstSharedMem[indexIndexDstShared] =
		messageValsNeighbor1[currentDisparity]
		+ messageValsNeighbor2[currentDisparity]
		+ messageValsNeighbor3[currentDisparity]
		+ dataCosts[currentDisparity];
		if (dstSharedMem[indexIndexDstShared] < minimum)
		{
			minimum = dstSharedMem[indexIndexDstShared];
		}
		indexIndexDstShared += halfIndexSharedVals[indexIntervalNextHalfIndexSharedVals];
		indexIntervalNextHalfIndexSharedVals = !indexIntervalNextHalfIndexSharedVals;
		//indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
//#if (NUM_POSSIBLE_DISPARITY_VALUES - 1) <= DISPARITY_START_SHARED_MEM //no shared memory used
//	dtStereo<float>(dst);
//#else
	dtStereoSharedMemory<half>(dstSharedMem);
//#endif

	// truncate
	minimum += disc_k_bp;

	// normalize
	half valToNormalize = (half)0.0f;
	indexIndexDstShared = startIndexDstShared;
	indexIntervalNextHalfIndexSharedVals = 0;

//#pragma unroll 64
	for (unsigned int currentDisparity = 0;
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++)
	{
		if (minimum < dstSharedMem[indexIndexDstShared])
		{
			dstSharedMem[indexIndexDstShared] = minimum;
		}

		valToNormalize += dstSharedMem[indexIndexDstShared];
		indexIndexDstShared += halfIndexSharedVals[indexIntervalNextHalfIndexSharedVals];
		indexIntervalNextHalfIndexSharedVals = !indexIntervalNextHalfIndexSharedVals;
		//indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
	}

	if (__hisnan(valToNormalize) || ((__hisinf(valToNormalize)) != 0))
	{
		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
						currentLevelProperties.paddedWidthCheckerboardLevel_,
						currentLevelProperties.heightLevel_, 0,
						NUM_POSSIBLE_DISPARITY_VALUES);

		for (unsigned int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++) {
			dstMessageArray[destMessageArrayIndex] = (half) 0.0;
#if beliefprop::OPTIMIZED_INDEXING_SETTING == 1
			destMessageArrayIndex +=
			currentLevelProperties.paddedWidthCheckerboardLevel_;
#else
			destMessageArrayIndex++;
#endif //beliefprop::OPTIMIZED_INDEXING_SETTING == 1
		}
	}
	else
	{
		valToNormalize /= ((half) NUM_POSSIBLE_DISPARITY_VALUES);

		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				NUM_POSSIBLE_DISPARITY_VALUES);

		indexIndexDstShared = startIndexDstShared;
//#pragma unroll 64
		for (unsigned int currentDisparity = 0;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++)
		{
			dstSharedMem[indexIndexDstShared] -= valToNormalize;
			dstMessageArray[destMessageArrayIndex] = dstSharedMem[indexIndexDstShared];
			indexIndexDstShared += halfIndexSharedVals[indexIntervalNextHalfIndexSharedVals];
			indexIntervalNextHalfIndexSharedVals = !indexIntervalNextHalfIndexSharedVals;
			//indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;

#if beliefprop::OPTIMIZED_INDEXING_SETTING == 1
			destMessageArrayIndex +=
			currentLevelProperties.paddedWidthCheckerboardLevel_;
#else
			destMessageArrayIndex++;
#endif //beliefprop::OPTIMIZED_INDEXING_SETTING == 1
		}
	}
#else
	half dst[NUM_POSSIBLE_DISPARITY_VALUES - DISP_INDEX_START_REG_LOCAL_MEM];

#if DISP_INDEX_START_REG_LOCAL_MEM > 0
//#pragma unroll 64
	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_INDEX_START_REG_LOCAL_MEM;
			currentDisparity++) {
		dstSharedMem[indexIndexDstShared] =
				messageValsNeighbor1[currentDisparity]
						+ messageValsNeighbor2[currentDisparity]
						+ messageValsNeighbor3[currentDisparity]
						+ dataCosts[currentDisparity];
		if (dstSharedMem[indexIndexDstShared] < minimum) {
			minimum = dstSharedMem[indexIndexDstShared];
		}
		indexIndexDstShared += halfIndexSharedVals[indexIntervalNextHalfIndexSharedVals];
		indexIntervalNextHalfIndexSharedVals = !indexIntervalNextHalfIndexSharedVals;
		//indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
	}
#endif

	indexIntervalNextHalfIndexSharedVals = 0;
//#pragma unroll 64
	for (unsigned int currentDisparity = DISP_INDEX_START_REG_LOCAL_MEM;
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++) {
		dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM] =
				messageValsNeighbor1[currentDisparity]
						+ messageValsNeighbor2[currentDisparity]
						+ messageValsNeighbor3[currentDisparity]
						+ dataCosts[currentDisparity];
		if (dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM] < minimum) {
			minimum = dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM];
		}
	}

	//retrieve the minimum value at each disparity in O(n) time using Felzenszwalb's method (see "Efficient Belief Propagation for Early Vision")
//#if (NUM_POSSIBLE_DISPARITY_VALUES - 1) <= DISPARITY_START_SHARED_MEM //no shared memory used
//	dtStereo<float>(dst);
//#else
	dtStereoSharedAndRegLocalMemory<half>(dstSharedMem, dst);
//#endif

	// truncate
	minimum += disc_k_bp;

	// normalize
	half valToNormalize = (half)0.0f;

#if DISP_INDEX_START_REG_LOCAL_MEM > 0
	indexIndexDstShared = startIndexDstShared;
	indexIntervalNextHalfIndexSharedVals = 0;
//#pragma unroll 64
	for (unsigned int currentDisparity = 0;
			currentDisparity < DISP_INDEX_START_REG_LOCAL_MEM;
			currentDisparity++) {
		if (minimum < dstSharedMem[indexIndexDstShared]) {
			dstSharedMem[indexIndexDstShared] = minimum;
		}

		valToNormalize += dstSharedMem[indexIndexDstShared];
		indexIndexDstShared += halfIndexSharedVals[indexIntervalNextHalfIndexSharedVals];
		indexIntervalNextHalfIndexSharedVals = !indexIntervalNextHalfIndexSharedVals;
		//indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;
	}
#endif
//#pragma unroll 64
	for (unsigned int currentDisparity = DISP_INDEX_START_REG_LOCAL_MEM;
			currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
			currentDisparity++) {
		if (minimum < dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM]) {
			dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM] = minimum;
		}

		valToNormalize +=
				dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM];
	}
	if (__hisnan(valToNormalize) || ((__hisinf(valToNormalize)) != 0))
		{
			unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
							currentLevelProperties.paddedWidthCheckerboardLevel_,
							currentLevelProperties.heightLevel_, 0,
							NUM_POSSIBLE_DISPARITY_VALUES);

			for (unsigned int currentDisparity = 0;
					currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
					currentDisparity++) {
				dstMessageArray[destMessageArrayIndex] = (half) 0.0;
	#if beliefprop::OPTIMIZED_INDEXING_SETTING == 1
				destMessageArrayIndex +=
				currentLevelProperties.paddedWidthCheckerboardLevel_;
	#else
				destMessageArrayIndex++;
	#endif //beliefprop::OPTIMIZED_INDEXING_SETTING == 1
			}
		} else {

		valToNormalize /= ((half) NUM_POSSIBLE_DISPARITY_VALUES);

		unsigned int destMessageArrayIndex = retrieveIndexInDataAndMessage(xVal, yVal,
				currentLevelProperties.paddedWidthCheckerboardLevel_,
				currentLevelProperties.heightLevel_, 0,
				NUM_POSSIBLE_DISPARITY_VALUES);

#if DISP_INDEX_START_REG_LOCAL_MEM > 0
		indexIndexDstShared = startIndexDstShared;
		indexIntervalNextHalfIndexSharedVals = 0;
//#pragma unroll 64
		for (unsigned int currentDisparity = 0;
				currentDisparity < DISP_INDEX_START_REG_LOCAL_MEM;
				currentDisparity++) {
			dstSharedMem[indexIndexDstShared] -= valToNormalize;
			dstMessageArray[destMessageArrayIndex] =
					dstSharedMem[indexIndexDstShared];
			indexIndexDstShared += halfIndexSharedVals[indexIntervalNextHalfIndexSharedVals];
			indexIntervalNextHalfIndexSharedVals = !indexIntervalNextHalfIndexSharedVals;
			//indexIndexDstShared += BLOCK_SIZE_WIDTH_BP * BLOCK_SIZE_HEIGHT_BP;

#if beliefprop::OPTIMIZED_INDEXING_SETTING == 1
			destMessageArrayIndex +=
					currentLevelProperties.paddedWidthCheckerboardLevel_;
#else
			destMessageArrayIndex++;
#endif //beliefprop::OPTIMIZED_INDEXING_SETTING == 1
		}
#endif
//#pragma unroll 64
		for (unsigned int currentDisparity = DISP_INDEX_START_REG_LOCAL_MEM;
				currentDisparity < NUM_POSSIBLE_DISPARITY_VALUES;
				currentDisparity++) {
			dst[currentDisparity - DISP_INDEX_START_REG_LOCAL_MEM] -=
					valToNormalize;
			dstMessageArray[destMessageArrayIndex] = dst[currentDisparity
					- DISP_INDEX_START_REG_LOCAL_MEM];

#if beliefprop::OPTIMIZED_INDEXING_SETTING == 1
			destMessageArrayIndex +=
					currentLevelProperties.paddedWidthCheckerboardLevel_;
#else
			destMessageArrayIndex++;
#endif //beliefprop::OPTIMIZED_INDEXING_SETTING == 1
		}
	}

#endif
}



